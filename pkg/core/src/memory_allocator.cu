// **************************************************************************
// File       [ memory_allocator.cu ]
// Author     [ littleshamoo ]
// Synopsis   [ Allocate GPU memory ]
// History    [ Version 1.0 2012/03/13 ]
// **************************************************************************

#include <iostream>
#include <fstream>
#include <vector>
#include <stdlib.h>
#include <stdio.h>
#include <algorithm>
#include <set>
#include <list>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include <cutil.h>
//#include <cutil_inline.h>
#include <iomanip>
//#include "hip/hip_runtime_api.h"
#include "pattern.h"
#include "fault.h"
#include "logic.h"

#include "memory_allocator.h"
using namespace CoreNs;
using namespace std;
void MemoryAllocator::alloc(int nFaults) {
    nTotalFaults_ = nFaults;
    transform();
    hipSetDevice(0);
    hipDeviceReset();

    // allocate only
    hipMalloc((void**)&gpuFaultGate_, sizeof(int)*(nTotalFaults_));
    hipMalloc((void**)&gpuFaultDetect_, sizeof(int)*(nTotalFaults_));
    hipMalloc((void**)&gpuFaultType_, sizeof(int)*(nTotalFaults_));
    hipMalloc((void**)&gpuFaultLine_, sizeof(int)*(nTotalFaults_));
    hipMalloc((void**)&gpuValues_,    sizeof(unsigned long)*(nValues_));

    // allocate and copy
    hipMalloc((void**)&gpuSplit_,     sizeof(unsigned long)*(nSplits_));
    hipMalloc((void**)&gpuGateType_,  sizeof(int)*(nGateTypes_));
    hipMalloc((void**)&gpuGateSplit_, sizeof(int)*(nGateSplits_));
    hipMalloc((void**)&gpuFanins_,    sizeof(int)*(nFanins_));
    hipMalloc((void**)&gpuNLevels_,   sizeof(int)*(nLevels_));
    hipMalloc((void**)&gpuNInputs_,   sizeof(int));
    hipMemcpy(gpuSplit_,     cpuSplit_,     sizeof(unsigned long)*(nSplits_), hipMemcpyHostToDevice);
    hipMemcpy(gpuGateType_,  cpuGateType_,  sizeof(int)*(nGateTypes_),        hipMemcpyHostToDevice);
    hipMemcpy(gpuGateSplit_, cpuGateSplit_, sizeof(int)*(nGateSplits_),       hipMemcpyHostToDevice);
    hipMemcpy(gpuFanins_,    cpuFanin_,     sizeof(int)*(nFanins_),           hipMemcpyHostToDevice);
    hipMemcpy(gpuNLevels_,   cpuNLevels_,   sizeof(int)*(nLevels_),           hipMemcpyHostToDevice);
    hipMemcpy(gpuNInputs_,   cpuNInputs_,   sizeof(int),                      hipMemcpyHostToDevice);
}

void MemoryAllocator::transform() {

    // allocate split vector pool on GPU
    // 5 pools of vectors with different zero-one ratios as in memry_allocator.h
    // each pool contains 1000 vectors
    nSplits_ = 5 * 1000;
    cpuSplit_ = new unsigned long[nSplits_];
    vector<ParaValue> sp11 = genSpVec(1.0);
    vector<ParaValue> sp31 = genSpVec(3.0);
    vector<ParaValue> sp71 = genSpVec(7.0);
    vector<ParaValue> sp13 = genSpVec(0.333);
    vector<ParaValue> sp17 = genSpVec(0.143);
    for (size_t i = 0; i < 1000; ++i)
        cpuSplit_[i] = sp11[i];
    for (size_t i = 0; i < 1000; ++i)
        cpuSplit_[1000 + i] = sp31[i];
    for (size_t i = 0; i < 1000; ++i)
        cpuSplit_[2000 + i] = sp71[i];
    for (size_t i = 0; i < 1000; ++i)
        cpuSplit_[3000 + i] = sp13[i];
    for (size_t i = 0; i < 1000; ++i)
        cpuSplit_[4000+ i] = sp17[i];

    // Transform circuit. Number of gates within each level will be equal
    // to number of threads. Record number of gates in original level and
    // mapping of orginal level to new level.
    vector<int> nGatesInOrgLevel;
    vector<int> orgLevelToNewLevel;
    orgLevelToNewLevel.push_back(0); // level 0 in both org and new are the same
    int prevOrgLevel = 0;
    int gateCount = 0;
    for (size_t i = 0; i < cir_->nGates(); ++i) {
        Gate* g = cir_->getGate(i);
        int currOrgLevel = g->getLvl();
        if (currOrgLevel != prevOrgLevel) {
            nGatesInOrgLevel.push_back(gateCount);
            int prevNewLevel = orgLevelToNewLevel[orgLevelToNewLevel.size() - 1];
            int nAddedLevels = gateCount / nThreads_;
            if (gateCount % nThreads_ != 0)
                nAddedLevels++;
            orgLevelToNewLevel.push_back(prevNewLevel + nAddedLevels);

            // find next new level
            gateCount = 0;
            prevOrgLevel = currOrgLevel;
        }
        gateCount++;
    }
    nLevels_ = 1;

    // map original ID to new ID
    int levelStartOrgId = 0;
    prevOrgLevel = 0;
    for (size_t i = 0; i < cir_->nGates(); ++i) {
        Gate* g = cir_->getGate(i);
        int currOrgLevel = g->getLvl();
        if (currOrgLevel != prevOrgLevel) {
            levelStartOrgId = g->getId();
            prevOrgLevel = currOrgLevel;
        }
        int newLevel = orgLevelToNewLevel[currOrgLevel];
        int newId = (g->getId() - levelStartOrgId) + newLevel * nThreads_;
        cpuIdToGpuId_.push_back(newId);
    }
    int lastNewId = cpuIdToGpuId_[cpuIdToGpuId_.size() - 1];
    int nTotalNewLevels = lastNewId / nThreads_;
    if (nTotalNewLevels % nThreads_ != 0)
        nTotalNewLevels++;
    cpuNLevels_ = new int[nLevels_];
    *cpuNLevels_ = nTotalNewLevels;
    cout<<"nTotalNewLevels:"<<nTotalNewLevels<<endl;
    size_t nGatesPerFrame = *cpuNLevels_ * nThreads_;
    size_t nTotalGates = nGatesPerFrame * cir_->nFrames();

    // transform gate type vector. Types are the same as those in gate.h
    nGateTypes_ = nTotalGates;
    cpuGateType_ = new int[nGateTypes_];
    for (size_t i = 0; i < nGateTypes_; ++i)
        cpuGateType_[i] = -1; // initialize to -1 to indicate NA gates
    for (size_t i = 0; i < cir_->nFrames(); ++i) {
        for (size_t j = 0; j < cir_->nGates(); ++j) {
            Gate *g = cir_->getGate(j);
            int newId = cpuIdToGpuId_[j];
            cpuGateType_[i * nGatesPerFrame + newId] = g->getType();
        }
    }

    // use weighted split vector if timing-aware mode is on.
    nGateSplits_ = nTotalGates;
    cpuGateSplit_ = new int[nGateSplits_];
    if (!taMode_) { // all gates use split vector pool 0 (zero:one = 1:1)
        for (size_t i = 0; i < nGateSplits_; ++i)
            cpuGateSplit_[i] = 0;
    }

    else { // choose split vector pool based on arrival time
        for (size_t i = 0; i < cir_->nFrames(); ++i) {
            for (size_t j = 0; j < cir_->nGates(); ++j) {
                Gate* g = cir_->getGate(j);
                int spVecPool = 0; // default
                // PI, PPI, PO, and PPO do not need weighted split
                if (g->getType() == Gate::PI || g->getType() == Gate::PPI
                    || g->getType() == Gate::PO || g->getType() == Gate::PPO
                    || g->getType() == Gate::INV || g->getType() == Gate::BUF)
                    ;
                else {
                    float atRatio = g->getFi(0)->getArrivalTime() / g->getFi(1)->getArrivalTime();
                    if (atRatio < 0.6) // fanin 0 shorter >40%
                        spVecPool = 2; // n0:n1 = 7:1
                    else if (atRatio < 0.85) // 15% < fanin 0 shorter < 40%
                        spVecPool = 1; // n0:n1 = 3:1
                    else if (atRatio > 1.4) // fanin 0 longer > 40%
                        spVecPool = 4; // n0:n1 = 1:7
                    else if (atRatio > 1.15) // 15% < fanin 0 longer < 40%
                        spVecPool = 3; // n0:n1 = 1:3
                }
                int newId = cpuIdToGpuId_[j];
                cpuGateSplit_[i * nGatesPerFrame + newId] = spVecPool;
            }
        }
    }

    // transform gate fanin vector
    nFanins_ = nTotalGates * 4; // maximum 4 fanins
    cpuFanin_ = new int[nFanins_];
    map<int,int> FFmapping;
    for (size_t i = 0; i < nFanins_; ++i)
        cpuFanin_[i] = -1; // initialized to -1 to indicate not connected
    for (size_t i = 0; i < cir_->nFrames(); ++i) {
        for (size_t j = 0; j < cir_->nGates(); ++j) {
            Gate* g = cir_->getGate(j);
            for (size_t k = 0; k < g->nFis(); ++k) {
                if(i==0 && g->getType()==2){//when FF in time frame 1
                    FFmapping[g->getFi(0)->getId()] = g->getId();// add to FFmapping, process when fransform gate fanout
                    continue;//no need to add fanin value
                }
                int newId = cpuIdToGpuId_[j];
                size_t index = i * nThreads_ * (*cpuNLevels_) * 4 + newId * 4 + k;
                size_t fiId = cpuIdToGpuId_[g->getFi(k)->getId()] + i * nGatesPerFrame;
                if(g->getType() == 2) fiId = cpuIdToGpuId_[g->getFi(k)->getId()] + 0 * nGatesPerFrame;
                cpuFanin_[index] = fiId;
            }
        }
    }

    // transform gate fanout vector
    /*nFanouts_ = nTotalGates * 8; // maximum 8 fanouts
    cpuFanout_ = new int[nFanouts_];
    for (size_t i = 0; i < nFanouts_; ++i)
        cpuFanout_[i] = -1; // initialized to -1 to indicate not connected
    map<int,int>::iterator it=FFmapping.begin();
    for(;it!=FFmapping.end();it++){
        int newId = cpuIdToGpuId_[it->first];
        size_t index = 0 * nThreads_ * (*cpuNLevels_) * 8 + newId * 8 + 0;
        size_t FoNewId = 1 * nThreads_ * (*cpuNLevels_) + cpuIdToGpuId_[it->second];
        cpuFanout_[index] = FoNewId;
    }
    for (size_t i = 0; i < cir_->nFrames(); ++i) {
        for (size_t j = 0; j < cir_->nGates(); ++j) {
            Gate* g = cir_->getGate(j);
            for (size_t k = 0; k < g->nFos(); ++k) {
                int newId = cpuIdToGpuId_[j];
                size_t index = i * nThreads_ * (*cpuNLevels_) * 8 + newId * 8 + k;// index = i * nGatesPerFrame + newId * 8 + k
                size_t foId = cpuIdToGpuId_[g->getFo(k)->getId()] + i * nGatesPerFrame;
                cpuFanout_[index] = foId;
            }
        }
    }*/

    // number of values denpends total number of gates and number of blocks.
    // Each gate has 8 values
    nValues_ = nTotalGates * nBlocks_ * 8;


    // littleshamoo
    cpuNInputs_ = new int;
    *cpuNInputs_ = cir_->nPis() + cir_->nSeqs();
}

vector<ParaValue> MemoryAllocator::genSpVec(float ratio) {
    int       count   = 0;
    size_t    zeroNum = 0;
    size_t    oneNum  = 0;
    ParaValue mask    = 0;
    ParaValue sp      = 0;
    vector<ParaValue> spVec;

    float percentage = ratio / (ratio + 1.0);
    size_t reqZeroNum = (size_t)(32.0 * percentage);
    for (size_t i = 0; i < 1000; ++i) {
        zeroNum = 0;
        oneNum  = 0;
        mask    = 1;
        sp      = 0;
        count   = 0;

        while(count < 32 - reqZeroNum){
            int nBit = rand()%32;
            if( (sp>>nBit) & 0x01 == 1)continue;
            else{
                sp |= 0x01<<nBit;
                count++;
            }
        }

        spVec.push_back(sp & 0xFFFFFFFF);
    }
    return spVec;
}
