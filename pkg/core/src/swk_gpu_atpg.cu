#include "hip/hip_runtime.h"
// **************************************************************************
// File       [ swk_gpu_atpg.cpp ]
// Author     [ littleshamoo ]
// Synopsis   [ ]
// History    [ Version 1.0 2010/03/10 ]
// **************************************************************************

#include "swk_gpu_atpg.h"
#include "hiprand/hiprand_kernel.h"
#include "pattern.h"
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <cutil_inline.h>
#include <iomanip>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include "hip/hip_runtime_api.h"
using namespace CoreNs;
using namespace std;

// Function Name : void __checkCudaErrors
// Functionality : use to check whether kernel has any error or not
// Usage         : CCE(hipDeviceSynchronize());
template< typename T >
inline void __checkCudaErrors(T result, char const *const func, const char *const file, int const line)
{
	hipError_t err = hipGetLastError();

	if (hipSuccess != err)
	{
		fprintf(stderr, "%s:%i : checkCudaErrors() CUDA error (#%d): %s.\n",
				file, line, (int)err, hipGetErrorString(err));
		exit(-1);
	}
}
#define CCE(val) __checkCudaErrors( (val), #val, __FILE__, __LINE__ )


// Function Name : void faultSim
// Functionality : the kernel of SWK fault simulation
__device__ void faultSim (
    int*             numRmnFaults
    , unsigned long* split
    , int*           faultGate
    , int*           faultType
    , int*           faultLine
    , int*           faultDetect
    , int*           gateType
    , int*           fanin
    , int*           nLevels
    , unsigned long* value
    , int*           nDetect
    , unsigned long* pattern
);

__device__ void randFill(
    int*             gateType
    , int            nLevels
    , unsigned long* value
    , unsigned long* split
);

__device__ void goodEval(
    int*             gateType
    , int*           fanin
    , int            nLevels
    , unsigned long* value
);

__device__ void faultEval(
    int*             gateType
    , int*           fanin
    , int            nLevels
    , unsigned long* value
    , int            numRmnFaults
    , int*           faultGate
    , int*           faultType
    , int*           faultLine
    , int*           faultDetect
    , int            nDetect
    , unsigned long* pattern
);

__device__ void resetValuesToGood(
    int              startLevel
    , int*           gateType
    , int            nLevels
    , unsigned long* value
);

// Function Name : void gpuGen
// Functionality : the kernel of SWK algorithm
__global__ void gpuGen (
    // circuit information
    int*             numRmnFaults
    , int*           faultGate
    , int*           faultType
    , int*           faultLine
    , int*           faultDetect
    , unsigned long* split
    , int*           gateType
    , int*           gateSplit
    , int*           fanin
    , int*           nLevels
    , unsigned long* value
    // backtrack
    , int*           bkStackSize
    , int*           bkValue
    , int*           bkPtr
    , int*           bkGate
    , bool*          bkFlipped
    // atpg parameters
    , int*           nDetect
    , int*           abLimit
    , int*           bkLimit
    , bool*          taMode
    , bool*          getMode
    // zero copy
    , int*           nInputs
    , unsigned long* pattern
);


// Function Name : void initBkParam
// Functionality : initialize the backtrack parameter bkPtr to -1
// Usage         : deadClones(32bits), bit = 1 means the clone need initialize
__device__ void initBkParam(
    int*  bkPtr,
    unsigned long & deadClones
);


// Function Name : void initialObjective
// Functionality : initialize the initial obkective
// Usage         : deadMask(32bits), bit = 1 means the clone need initial objective
// Usage         : only 4 threads needed for this function due to 4 total faults in 32 clones
__device__ void initialObjectives (
    int nFaults
    , int*           faultGate
    , int*           faultType
    , int*           faultLine
    , int            nLevels
    , int*           fanin
    , unsigned long* value
    , int            nDetect
    , bool           getMode
    , unsigned long  deadMask
);


// Function Name : void propagation
// Functionality : do propagation, -1 < level < nLevels * 2, 2 due to two timeframes
// Usage         : propagatable(32bits), bit = 1 means there is p at any PO or PPO in this clone
// Usage         : detected(32bits), bit = 1 means there is d|b at any PO or PPO in this clone
__device__ void propagation(
    int              currentLevel
    , int            nFaults
    , unsigned long& propagatable
    , unsigned long& detected
    , int*           gateType
    , int*           gateSplit
    , int*           fanin
    //, int*           fanout
    , int            nLevels
    , unsigned long* values
    , int* faultGate
    , int* faultType
    , int* faultLine
);


// Function Name : void backtrack
// Functionality : do backtrack
// Usage         : deadClones(32bits), bit = 1 means the clone is dead and the stack is not empty -->needs backtrack
// Usage         : StackEmpty(32bits), bit = 1 means the cloes's stack is empty, no need for backtrack anymore
__device__ void backtrack(
    unsigned long    deadClones
    , int            nLevels
    , unsigned long* value
    , int            bkStackSize
    , int*           bkValue
    , int*           bkPtr
    , int*           bkGate
    , bool*          bkFlipped
    , unsigned long* StackEmpty
);


// Function Name : void backtrace
// Functionality : do propagation, nLevels * 2 > level > -1, 2 due to two timeframes
// Usage         : obj(32bits), bit = 1 means the clone has objective, no o-generation can be performed
__device__ void backtrace(
    int              currentLevel
    , unsigned long& obj
    , unsigned long* split
    , int*           gateType
    , int*           gateSplit
    , int*           fanin
    //, int*           fanout
    , int            nLevels
    , unsigned long* value
    // for backtrack
    , int            bkStackSize
    , int*           bkValue
    , int*           bkPtr
    , int*           bkGate
    , bool*          bkFlipped
);


// Function Name : void trackAssignment
// Functionality : record the assignment of each clone in the stack after each backtrace loop
// Usage         : assignedId(int * 128), the gateId in the corresponding backtrace loop
// Usage         : assignedV0(32bits * 128), the new l value in the corresponding backtrace loop  (after assignment)
// Usage         : assignedV1(32bits * 128), the new h value in the corresponding backtrace loop  (after assignment)
// Usage         : prevV0(32bits * 128), the previous l value in the corresponding backtrace loop (before assignment)
// Usage         : prevV1(32bits * 128), the previous h value in the corresponding backtrace loop (before assignment)
__device__ void trackAssignment(
    int*             assignedId
    , unsigned long* assignedV0
    , unsigned long* assignedV1
    , unsigned long* prevV0
    , unsigned long* prevV1
    , int            bkStackSize
    , int*           bkValue
    , int*           bkPtr
    , int*           bkGate
    , bool*          bkFlipped
);


// Function Name : void zeroCopy
// Functionality : copy pattern in GPU memory to CPU memory
__device__ void zeroCopy(
    int*             gateType
    , unsigned long* values
    , unsigned long* pattern
    , int            nLevels
    , int            nInputs
);


// Function Name : void printValue
// Functionality : print the value according to v0 and v1
__device__ void printValue(
    int             gateId
    , unsigned long v0
    , unsigned long v1
);


// Function Name : void clearValue
// Functionality : clear the value in "values"
// Usage         : clearClones(32bits), bit = 1 means the values(l,h,d,b,btl,bth,p) of this clone need to be clean
// Usage         : pp flag = 1 means clear all the values <<<instead of PI in timeframe1,2 and PPI in timeframe1>>>
// Usage         : bt flag = 1 means clear all the backtrace values(btl,bth)
__device__ void clearValue(unsigned long *values
    , int* gateType
    , int nLevels
    , unsigned long clearClones
    , bool pp
    , bool bt
);


// Function Name : void NumberOfSetBits
// Functionality : return how many set bits in a 32-bits value
int SwkGpuAtpg::NumberOfSetBits(unsigned long i)
{
     i = i - ((i >> 1) & 0x55555555);
     i = (i & 0x33333333) + ((i >> 2) & 0x33333333);
     return (int)(((i + (i >> 4)) & 0x0F0F0F0F) * 0x01010101) >> 24;
}


// Function Name : void WritePattern
// Functionality : build pCol(add 32 pattern) after each kernel
// Usage         : TotalBlocks(int), shows the used number of block in this kernal
void SwkGpuAtpg::WritePattern(PatternColl *pCol, int TotalBlocks)
{
    unsigned long v0;
    unsigned long v1;

    // trace all the blocks in this kernel
    for(int block = 0; block < TotalBlocks; ++block){

        // detect information has been saved in the begin of the array cpuPattern_
        //unsigned long Detected = cpuPattern_[block];
        unsigned long Detected = ~(0x0);
        int nDetectedPattern = NumberOfSetBits(Detected);
        printf(" | block %2d ----> detected = %8lx\n",block,Detected);

        // create number of detected pattern class and update clock information
        Pattern* Pat[nDetectedPattern];
        for(int i = 0; i < nDetectedPattern; ++i){
            Pat[i] = new Pattern( cir_->nPis() + 3, cir_->nPos(), cir_->nFrames(), cir_->nSeqs() );// CK...

            // set CAPT in timeframe1,2
            Pattern::Clk clk = Pattern::CAPT;
            Value v = L;
            size_t frame = 0;
            Pat[i]->setClk(clk, frame);
            Pat[i]->setPi(v, 0, frame);// CK
            Pat[i]->setPi(v, 1, frame);// test_si
            Pat[i]->setPi(v, 2, frame);// test_se
            frame = 1;
            Pat[i]->setClk(clk, frame);
            Pat[i]->setPi(v, 0, frame);// CK
            Pat[i]->setPi(v, 1, frame);// test_se
            Pat[i]->setPi(v, 2, frame);// test_si
        }

        // set input value
        // trace all the PI PPI in two timeframes
        for(int k = 0; k < nInputs_ * 2; ++k){
            // k is input offset
            // block * nInputs_ * 2 is block offset(*2 due to two timeframe)
            // *2 outside is due to two value per PI or PPI
            // TotalBlocks is the offset for detection information
            v0 = cpuPattern_[(k + block * nInputs_ * 2) * 2 + 0 + TotalBlocks];
            v1 = cpuPattern_[(k + block * nInputs_ * 2) * 2 + 1 + TotalBlocks];

            Value v;

            // skip if the input is PPI and the input is in timeframe2 (in __CAPT__ mode)
            if(k >= cir_->nPis() * 2 && k%2 != 0)continue;

            // idx = k / 2                if the input is PI
            // idx = k / 2 - cir_->nPis() if the input is PPI (remove the offset of number of PI)
            size_t idx   = k < cir_->nPis() * 2 ? k / 2 : k / 2 - cir_->nPis();

            // frame = k % 2                if the input is PI
            // frame = 0                    if the input is PPI (if the input is PPI, the timeframe is always 1)
            size_t frame = k < cir_->nPis() * 2 ? k % 2 : 0 ;

            int AddedPattern = 0;
            // trace all the clone
            for (size_t i = 0; i < 32; ++i) {
                // skip if not detect in this clone
                int skip = (Detected >> i) & 0x01;
                if(skip == 0)continue;

                // get value according to the v0 and v1
                unsigned long v0bit = (v0 >> i) & (unsigned long)0x01;
                unsigned long v1bit = (v1 >> i) & (unsigned long)0x01;

                // X--0--1--? in each condition
                if(v0bit == 0 && v1bit == 0)
                    v = rand()%2 == 0 ? L : H;//random fill
                else if (v0bit == 1 && v1bit == 0)
                    v = L;
                else if (v0bit == 0 && v1bit == 1)
                    v = H;
                else
                    v = rand()%2 == 0 ? L : H;//random fill

                // add value into pattern
                if(k < cir_->nPis() * 2)
                    Pat[AddedPattern]->setPi(v, idx + 3, frame);// CK...
                else
                    Pat[AddedPattern]->setPpi(v, idx);
                AddedPattern++;
            }
        }

        // push back all the pattern objects to pCol
        for(int i = 0; i < nDetectedPattern; ++i)
            pCol->addPattern(Pat[i]);
    }
}


// Function Name : void allocFaults
// Functionality : allocate memory for remaining faults on GPU
void SwkGpuAtpg::allocFaults(FaultList& rmnFault) {
    (*cpuNumRmnFaults_) = rmnFault.size();
    FaultListIter iter = rmnFault.begin();
    int idx = 0;
    while (iter != rmnFault.end()) {
        Fault* f = *iter;
        cpuFaultGate_[idx] = memAlloc_->gpuId(f->getGate()->getId());
        cpuFaultDetect_[idx] = f->getDet();
        cpuFaultType_[idx] = f->getType();
        cpuFaultLine_[idx] = f->getLine();
        ++idx;
        ++iter;
    }

    // copy memory to GPU
    hipMemcpy(gpuNumRmnFaults_, cpuNumRmnFaults_, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(memAlloc_->faultGateGpuPtr(), cpuFaultGate_, sizeof(int) * (*cpuNumRmnFaults_), hipMemcpyHostToDevice);
    hipMemcpy(memAlloc_->faultDetectGpuPtr(), cpuFaultDetect_, sizeof(int) * (*cpuNumRmnFaults_), hipMemcpyHostToDevice);
    hipMemcpy(memAlloc_->faultTypeGpuPtr(), cpuFaultType_, sizeof(int) * (*cpuNumRmnFaults_), hipMemcpyHostToDevice);
    hipMemcpy(memAlloc_->faultLineGpuPtr(), cpuFaultLine_, sizeof(int) * (*cpuNumRmnFaults_), hipMemcpyHostToDevice);
}


// Function Name : void zeroCopyPreparation
// Functionality : prepare for zerocopy
void SwkGpuAtpg::zeroCopyPreparation() {

    // number of inputs is number of PI add number of PPI
    nInputs_ = cir_->nPis() + cir_->nSeqs();

    // number of pattern is number of inputs * 2 timeframe * 2 values(l,h)
    int nValues = nInputs_ * cir_->nFrames() * 2;

    // number of pattern array is number of pattern + total blocks in this kernel(use to record the detected information)
    nPatterns_ = nValues * nBlocks_ + nBlocks_;
    cpuPattern_ = (unsigned long*)malloc(sizeof(unsigned long) * nPatterns_);
    hipMalloc((void**)&gpuPattern_, sizeof(unsigned long) * nPatterns_);
}

void SwkGpuAtpg::gen(PatternColl *pCol, FaultColl *fCol) {

    //sddcudaSim->cudaDataTrans();
    cout << " [Correct] Simulator Alloaction finished" << endl;

    cout << " ==========================================================" << endl;
    cout << " =                  Start SWK ATPG                        =" << endl;
    cout << " ==========================================================" << endl;
    unsigned int timer = 0;
    (cutCreateTimer(&timer));
    (cutStartTimer(timer));

    Fault     *f;

    // allocate ATPG parameters memory on GPU
    allocAtpgParameters();

    // prepare zero copy generated test patterns
    zeroCopyPreparation();

    // put all the faults in the circuit into the remain fault list
    FaultList rmnGenFault;
    FaultList rmnSimFault;
    for (size_t i = 0; i < fCol->getFaults()->size(); ++i) {
        f = fCol->getFaults()->at(i);
        if(f->getState() == Fault::UD || f->getState() == Fault::AB)
            rmnGenFault.push_back(fCol->getFaults()->at(i));
    }

    // allocate space to store faults
    int nTotalFaults = fCol->getFaults()->size();
    cpuNumRmnFaults_ = new int;
    cpuNumPrimaryFaults_ = new int;
    cpuFaultGate_ = new int[nTotalFaults];
    cpuFaultDetect_ = new int[nTotalFaults];
    cpuFaultType_ = new int[nTotalFaults];
    cpuFaultLine_ = new int[nTotalFaults];
    hipMalloc((void**)&gpuNumRmnFaults_, sizeof(int));
    (*cpuNumPrimaryFaults_) = nTotalFaults;
    cout << " [Correct] Fault Allocation finished" << endl;

    // report Memory Usage
    size_t free_byte;
    size_t total_byte;
    if(hipMemGetInfo(&free_byte,&total_byte) != hipSuccess){
        printf(" [Error]: Memory Get Info Fail!!\n");
        return;
    }
    else
        printf(" [Success]: Memory Get Info success!!\n");
    cout<<" ------------------------------------------\n";
    cout<<" | GPU memory free = "<<setw(12)<<(float)(free_byte)/1024.0/1024.0<<" MB      |\n";
    cout<<" | GPU memory used = "<<setw(12)<<(float)(total_byte - free_byte)/1024.0/1024.0<<" MB      |\n";
    cout<<" ------------------------------------------\n";
    cout<<" | Mem Setup Time  = "<<setw(12)<< cutGetTimerValue(timer) <<" ms      |\n";
    cout<<" ------------------------------------------\n";


    cout << " ==========================================================" << endl;
    cout << " =                  Start Generation                      =" << endl;
    cout << " ==========================================================" << endl;
    int iter = 0;
    while (rmnGenFault.size() > 0) {
        // allocate corresponding faults on GPU
        allocFaults(rmnGenFault);

        cout << " ----------------------------------------------------------" << endl;
        cout << " | Clock  = " << setw(12) << cutGetTimerValue(timer) <<" ms" << endl;
        cout << " | Iter   = " << iter << endl;
        cout << " | Generate " << nBlocks_ * 4 << " faults... " << *cpuNumPrimaryFaults_ << " remains" << endl;

        // test generation kernel
        gpuGen <<< nBlocks_, nThreads_ >>> (
            // circuit information
            gpuNumRmnFaults_
            , memAlloc_->faultGateGpuPtr()
            , memAlloc_->faultTypeGpuPtr()
            , memAlloc_->faultLineGpuPtr()
            , memAlloc_->faultDetectGpuPtr()
            , memAlloc_->splitGpuPtr()
            , memAlloc_->gateTypeGpuPtr()
            , memAlloc_->gateSplitGpuPtr()
            , memAlloc_->faninsGpuPtr()
            , memAlloc_->nLevelsGpuPtr()
            , memAlloc_->valuesGpuPtr()
            // backtrack parameters
            , gpuBkStackSize_
            , gpuBkValue_
            , gpuBkPtr_
            , gpuBkGate_
            , gpuBkFlipped_
            // atpg parameters
            , gpuNDetect_
            , gpuDcLimit_
            , gpuBkLimit_
            , gpuTaMode_
            , gpuGetMode_
            // zero copy
            , memAlloc_->nInputsGpuPtr()
            , gpuPattern_
        );
        hipDeviceSynchronize();

        // zero copy patterns back to CPU
        CCE(hipMemcpy(cpuPattern_, gpuPattern_, sizeof(unsigned long)*nPatterns_ , hipMemcpyDeviceToHost));

        WritePattern(pCol, nBlocks_);
        //pCol->print();

        dropFaults(rmnGenFault, rmnSimFault, pCol);
        iter++;
    }
}

void SwkGpuAtpg::dropFaults(FaultList& rmnGenFault
    , FaultList& rmnSimFault
    , PatternColl* pCol)
{
    // put generated faults into remaining sim list
    int nPrevGenFaults = nBlocks_ * 4;
    FaultListIter it = rmnGenFault.begin();
    int count = 0;
    while (it != rmnGenFault.end()) {
        Fault* f = (*it);
        it = rmnGenFault.erase(it);
        rmnSimFault.insert(rmnSimFault.begin(), f);
        ++count;
        if (count == nPrevGenFaults)
            break;
        ++it;
    }

    cout << " ==========================================================" << endl;
    cout << " =                  Start Simulation                      =" << endl;
    cout << " ==========================================================" << endl;
    // perform fault simulation on two fault lists
    int nPrevGenPatterns = nBlocks_ * 32;
    sim_->simulate(pCol, rmnSimFault, nPrevGenPatterns);
    sim_->simulate(pCol, rmnGenFault, nPrevGenPatterns);
    cout << " Done " << endl;
    cout << " Remaining number of generation faults: " << rmnGenFault.size() << endl;
    cout << " Remaining number of simulation faults: " << rmnSimFault.size() << endl;
}

__device__ void faultSim (
    int*             numRmnFaults
    , unsigned long* split
    , int*           faultGate
    , int*           faultType
    , int*           faultLine
    , int*           faultDetect
    , int*           gateType
    , int*           fanin
    , int*           nLevels
    , unsigned long* value
    , int*           nDetect
    , unsigned long* pattern
) {
    // DEBUG
    clock_t timerStart;
    clock_t timerStop;
    double totaltime;
    if (blockIdx.x == 1 && threadIdx.x == 0)
        timerStart = clock();
    __syncthreads();

    randFill(gateType, (*nLevels), value, split);

    if (blockIdx.x == 1 && threadIdx.x == 0) {
        timerStop = clock64();
        totaltime = (double)(timerStop - timerStart)/CLOCKS_PER_SEC / 1000;
        printf("  + Random fill %lf s\n", totaltime);
        timerStart = clock64();
    }
    __syncthreads();

    goodEval(gateType, fanin, (*nLevels), value);

    if (blockIdx.x == 1 && threadIdx.x == 0) {
        timerStop = clock64();
        totaltime = (double)(timerStop - timerStart)/CLOCKS_PER_SEC / 1000;
        printf("  + Good evaluation %lf s\n", totaltime);
        timerStart = clock64();
    }
    __syncthreads();

    faultEval(
        gateType
        , fanin
        , (*nLevels)
        , value
        , (*numRmnFaults)
        , faultGate
        , faultType
        , faultLine
        , faultDetect
        , (*nDetect)
        , pattern
    );

    if (blockIdx.x == 1 && threadIdx.x == 0) {
        timerStop = clock64();
        totaltime = (double)(timerStop - timerStart)/CLOCKS_PER_SEC / 1000;
        printf("  + Faulty evaluation %lf s\n", totaltime);
        timerStart = clock64();
    }
    __syncthreads();

}

__device__ void faultEval(
    int*             gateType
    , int*           fanin
    , int            nLevels
    , unsigned long* value
    , int            numRmnFaults
    , int*           faultGate
    , int*           faultType
    , int*           faultLine
    , int*           faultDetect
    , int            nDetect
    , unsigned long* pattern
) {
    int thId = threadIdx.x;
    int blId = blockIdx.x;

    __shared__ int nGatesPerFrame;      // for finding corresponding gates in second time frame
    __shared__ unsigned long activated; // monitor activated bits at fault site
    __shared__ unsigned long detected;  // monitor detected bits at PO and PPO
    //__shared__ unsigned long changed;   // monitor faulty value change in each level

    clock_t timerStart;
    clock_t timerStop;
    clock_t timerStart2;
    clock_t timerStop2;
    double totaltime;

    if (thId == 0)
        nGatesPerFrame = blockDim.x * nLevels;
    __syncthreads();
    
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        timerStart2 = clock64();
    }

    int startIdx = blockIdx.x * (numRmnFaults / gridDim.x);
    for (int i = 0; i < numRmnFaults ; ++i) {
        bool debug = false;
        if (blockIdx.x == 1 && threadIdx.x == 0) {
            if (i % 10000 == 0 ) {//|| i % 10000 == 1) {
                timerStop2 = clock64();
                //totaltime = ((double)(timerStop2) - (double)(timerStart2))/CLOCKS_PER_SEC;
                //printf("    + total i = %d\n", i);
                //printf("    + total time = %f\n", totaltime);
                debug = true;
                printf("    + i = %d\n", i);
                printf("    + timeStop  = %lu s\n", timerStop2);
                printf("    + timeStart = %lu s\n", timerStart2);
                //printf("    + clock per sec = %lu s\n", CLOCKS_PER_SEC);
                //printf("    + time      = %lf s\n", (double)(timerStop2 - timerStart2)/CLOCKS_PER_SEC);
                timerStart = clock64();
            }
        }

        // each block start from different fault
        int faultIdx = i + startIdx;
        if (faultIdx >= numRmnFaults)  // loop to the beginning
            faultIdx -= numRmnFaults;

        /*if (blockIdx.x == 0 && threadIdx.x == 0) {

            if (i == 1 || i == 3 || i == 5) {
                printf("    + i = %d\n", i);
                debug = true;
                timerStart = clock();
            }
        }*/
        __syncthreads();

        // check previous detection
        if (faultDetect[faultIdx] >= nDetect) {
            /*if (threadIdx.x == 0) {
                printf("block %d skip\n",blockIdx.x);
            }*/
            __syncthreads();
            continue;
        }
        int gate = faultGate[faultIdx];
        int type = faultType[faultIdx];
        int line = faultLine[faultIdx];
        if (line > 0)
            gate = fanin[gate * 4 + line - 1];
        int gateT2 = gate + nGatesPerFrame;

        // thread 0 check activation
        if (thId == 0) {
            activated = 0x0;
            detected = 0x0;
            //changed = 0x0;

            // faultGate value, used for inject fault
            int faultgate = faultGate[faultIdx];
            int faultgateT2 = faultGate[faultIdx] + nGatesPerFrame;
            int gatetype = gateType[faultgate];

            // falutGate's input value, used for check activation
            int faultgate_a = fanin[faultgate * 4 + 0]; 
            int faultgate_b = fanin[faultgate * 4 + 1];
            int faultgate_aT2 = fanin[faultgateT2 * 4 + 0];
            int faultgate_bT2 = fanin[faultgateT2 * 4 + 1];

            // find values in the two time frames
            unsigned long t1gl = value[blId * nGatesPerFrame * 8 * 2 + gate * 8 + 0];
            unsigned long t1gh = value[blId * nGatesPerFrame * 8 * 2 + gate * 8 + 1];
            unsigned long t2gl = value[blId * nGatesPerFrame * 8 * 2 + gateT2 * 8 + 0];
            unsigned long t2gh = value[blId * nGatesPerFrame * 8 * 2 + gateT2 * 8 + 1];

            unsigned long t1a0 = value[blId * nGatesPerFrame * 8 * 2 + faultgate_a * 8 + 0];
            unsigned long t1a1 = value[blId * nGatesPerFrame * 8 * 2 + faultgate_a * 8 + 1];
            unsigned long t1b0 = value[blId * nGatesPerFrame * 8 * 2 + faultgate_b * 8 + 0];
            unsigned long t1b1 = value[blId * nGatesPerFrame * 8 * 2 + faultgate_b * 8 + 1];
            unsigned long t2a0 = value[blId * nGatesPerFrame * 8 * 2 + faultgate_aT2 * 8 + 0];
            unsigned long t2a1 = value[blId * nGatesPerFrame * 8 * 2 + faultgate_aT2 * 8 + 1];
            unsigned long t2b0 = value[blId * nGatesPerFrame * 8 * 2 + faultgate_bT2 * 8 + 0];
            unsigned long t2b1 = value[blId * nGatesPerFrame * 8 * 2 + faultgate_bT2 * 8 + 1];

            // determine activation based on fault types
            // if line==0 : check faultGate's output
            // if line!=0 : check faultGate's input
            if (type == 2) // slow-to-rise
                activated = t1gl & t2gh;
            else if (type == 3) // slow-to-fall
                activated = t1gh & t2gl;

            // update activated value (check side input value) if line!=0

            if (line == 1){
                if (gatetype == 6 || gatetype == 7) // AND / NAND  side input not zero
                    if(type == 2)
                        activated &= ~t2b0;
                    else if(type == 3)
                        activated &= ~t1b0;
                else if (gatetype == 8 || gatetype == 9) // OR / NOR side input not 1
                    if(type == 2)
                        activated &= ~t2b1;
                    else if(type == 3)
                        activated &= ~t1b1;
            }
            else if (line == 2){
                if (gatetype == 6 || gatetype == 7) // AND / NAND  side input not zero
                    if(type == 2)
                        activated &= ~t2a0;
                    else if(type == 3)
                        activated &= ~t1a0;
                else if (gatetype == 8 || gatetype == 9) // OR / NOR side input not 1
                    if(type == 2)
                        activated &= ~t2a1;
                    else if(type == 3)
                        activated &= ~t1a1;
            }

        }

        if (debug) {
            timerStop = clock64();
            totaltime = (double)(timerStop - timerStart)/CLOCKS_PER_SEC / 1000;
            printf("    + Check activation %lf s\n", totaltime);
            timerStart = clock64();
        }

        __syncthreads();
        if (activated == 0x0) // find next activated fault
            continue;

        // update gate after checking activation
        gate = faultGate[faultIdx];
        gateT2 = gate + nGatesPerFrame;

        // thread 0 inject faults at gate output
        if (thId == 0) {
            int gatetype = gateType[gate];
            if (type == 2) { // slow-to-rise
                if((line == 0) || (gatetype != 4 && gatetype != 7 && gatetype != 9)){// INV NAND NOR
                    value[blId * nGatesPerFrame * 8 * 2 + gateT2 * 8 + 2] = ~(0x0); // faulty low in second time frame
                    value[blId * nGatesPerFrame * 8 * 2 + gateT2 * 8 + 3] = 0x0;    // faulty high in second time frame
                }
                else{
                    value[blId * nGatesPerFrame * 8 * 2 + gateT2 * 8 + 2] = 0x0; // faulty low in second time frame
                    value[blId * nGatesPerFrame * 8 * 2 + gateT2 * 8 + 3] = ~(0x0);    // faulty high in second time frame
                }
            }
            else if (type == 3) { // slow-to-fall
                if((line == 0) || (gatetype != 4 && gatetype != 7 && gatetype != 9)){
                    value[blId * nGatesPerFrame * 8 * 2 + gateT2 * 8 + 2] = 0x0;    // faulty low in second time frame
                    value[blId * nGatesPerFrame * 8 * 2 + gateT2 * 8 + 3] = ~(0x0); // faulty high in second time frame
                }
                else{
                    value[blId * nGatesPerFrame * 8 * 2 + gateT2 * 8 + 2] = ~(0x0);    // faulty low in second time frame
                    value[blId * nGatesPerFrame * 8 * 2 + gateT2 * 8 + 3] = 0x0; // faulty high in second time frame
                }
            }
        }

        if (debug) {
            timerStop = clock64();
            totaltime = (double)(timerStop - timerStart)/CLOCKS_PER_SEC / 1000;
            printf("    + Inject fault %lf s\n", totaltime);
            timerStart = clock64();
        }

        __syncthreads();

        // perform faulty evaluation for each gate starting from the level of the faulty gate
        int startLevel = gateT2 / blockDim.x;
        for (int currLevel = startLevel + 1; currLevel < nLevels * 2; ++currLevel) {
            // check for events using the changed flag
            //if (thId == 0)
            //    changed = 0x0;
            //__syncthreads();

            int blockOffset = (nLevels * 2) * blockDim.x * blockIdx.x;
            int levelOffset = currLevel * blockDim.x;
            int valueStart = (blockOffset + levelOffset) * 8 + threadIdx.x * 8;

            // find gate and gate type
            int gate = levelOffset + threadIdx.x;
            int type = gateType[gate];

            unsigned long gl = value[valueStart + 0]; // good low
            unsigned long gh = value[valueStart + 1]; // good high
            unsigned long fl = value[valueStart + 2]; // faulty low
            unsigned long fh = value[valueStart + 3]; // faulty high

            // find fanin values
            int aStart= -1;
            int bStart= -1;
            if(fanin[gate * 4] != -1)
                aStart = blockOffset * 8 + fanin[gate * 4] * 8;     // fanin 0
            if(fanin[gate * 4 + 1] != -1)
                bStart = blockOffset * 8 + fanin[gate * 4 + 1] * 8; // fanin 1

            unsigned long afl = 0x0;
            unsigned long afh = 0x0;
            unsigned long bfl = 0x0;
            unsigned long bfh = 0x0;
            if (aStart != -1) {
                afl = value[aStart + 2];
                afh = value[aStart + 3];
            }
            if (bStart != -1) {
                bfl = value[bStart + 2];
                bfh = value[bStart + 3];
            }
            
            // perform good simulation for different gate types
            //type in here is gate type
            if (type == 0) // PI do not need simulation
                ;
            else if (type == 2 && currLevel >= nLevels) { // PPI act as BUF
                fl = afl;
                fh = afh;
            }
            else if (type == 1 || type == 3) { // PO, PPO
                // special case when faultGate is PO
                // no propagation is needed because fault effect has already at PO
                fl = afl;
                fh = afh;
                //atomicOr((unsigned int *)&detected, (gl ^ fh) & (gh ^ fl));
                //atomicOr((unsigned int *)&detected, (gl ^ fl) & (gh ^ fh));
            }
            else if (type == 4) { // INV
                fl = afh;
                fh = afl;
            }
            else if (type == 5) { // BUF
                fl = afl;
                fh = afh;
            }
            else if (type == 6) { // AND
                fl = afl | bfl;
                fh = afh & bfh;
            }
            else if (type == 7) { // NAND
                fh = afl | bfl;
                fl = afh & bfh;
            }
            else if (type == 8) { // OR
                fl = afl & bfl;
                fh = afh | bfh;
            }
            else if (type == 9) { // NOR
                fl = afh | bfh;
                fh = afl & bfl;
            }

            //printf("thread %d, level %d type %d\n", threadIdx.x, currLevel, type);
            if(type != -1){
                //printValue(gate, gl, gh);
                //printValue(gate, fl, fh);
            }
            // assign simulated faulty values back to global memory
            value[valueStart + 2] = fl;
            value[valueStart + 3] = fh;
            __syncthreads();
        }

        if (debug) {
            timerStop = clock64();
            totaltime = (double)(timerStop - timerStart)/CLOCKS_PER_SEC / 1000;
            printf("    + Evaluation %lf s\n", totaltime);
            timerStart = clock64();
        }

        // update detected info
        for (int currLevel = nLevels; currLevel < nLevels * 2; ++currLevel) {

            int blockOffset = (nLevels * 2) * blockDim.x * blockIdx.x;
            int levelOffset = currLevel * blockDim.x;
            int valueStart = (blockOffset + levelOffset) * 8 + threadIdx.x * 8;

            // find gate and gate type
            int gate = levelOffset + threadIdx.x;
            int type = gateType[gate];

            if(type == 1 || type == 3){
                unsigned long gl = value[valueStart + 0]; // good low
                unsigned long gh = value[valueStart + 1]; // good high
                unsigned long fl = value[valueStart + 2]; // faulty low
                unsigned long fh = value[valueStart + 3]; // faulty high
                atomicOr((unsigned int *)&detected, (gl ^ fl) & (gh ^ fh));
            }
            //__syncthreads();
        }

            __syncthreads();

        if(thId == 0){
            detected &= activated;
            // update detection data
            //pattern[blId] |= detected;// JKY @ 20150415
        }

        if (debug) {
            timerStop = clock64();
            totaltime = (double)(timerStop - timerStart)/CLOCKS_PER_SEC / 1000;
            printf("    + Update detected %lf s\n", totaltime);
            timerStart = clock64();
        }

        __syncthreads();

        // count number of detection
        if (thId == 0 && detected != 0x0) {
            unsigned long mask = 0x1;
            for (int j = 0; j < 32; ++j) {
                unsigned long bitDetect = mask & detected;
                if (bitDetect != 0x0){
                    atomicAdd((unsigned int*)&faultDetect[faultIdx], 1);
                    pattern[blId] |= bitDetect;

                    if(faultDetect[faultIdx] >= nDetect)
                        break;
                }
                mask <<= 1;
            }
        }

        if (debug) {
            timerStop = clock64();
            totaltime = (double)(timerStop - timerStart)/CLOCKS_PER_SEC / 1000;
            printf("    + Count detected %lf s\n", totaltime);
            timerStart = clock64();
        }

        __syncthreads();

        // reset circuit to good evaluation values
        resetValuesToGood(startLevel, gateType, nLevels, value);

        if (debug) {
            timerStop = clock64();
            totaltime = (double)(timerStop - timerStart)/CLOCKS_PER_SEC / 1000;
            printf("    + Reset to good %lf s\n", totaltime);
            timerStart = clock64();
        }

        __syncthreads();
    }
}

__device__ void resetValuesToGood(
    int              startLevel
    , int*           gateType
    , int            nLevels
    , unsigned long* value
) {
    for (int currLevel = startLevel; currLevel < nLevels * 2; ++currLevel) {
        int blockOffset = (nLevels * 2) * blockDim.x * blockIdx.x;
        int levelOffset = currLevel * blockDim.x;
        int valueStart = (blockOffset + levelOffset) * 8 + threadIdx.x * 8;

        // find gate and gate type
        value[valueStart + 2] = value[valueStart + 0]; // set faulty low to good low
        value[valueStart + 3] = value[valueStart + 1]; // set faulty high to good high
    }
}

__device__ void goodEval(
    int*             gateType
    , int*           fanin
    , int            nLevels
    , unsigned long* value
) {
    for (int level = 0; level < nLevels * 2; ++level) {
        int blockOffset = (nLevels * 2) * blockDim.x * blockIdx.x;
        int levelOffset = level * blockDim.x;
        int valueStart = (blockOffset + levelOffset) * 8 + threadIdx.x * 8;

        // find gate and gate type
        int gate = levelOffset + threadIdx.x;
        int type = gateType[gate];

        unsigned long gl = value[valueStart + 0]; // good low
        unsigned long gh = value[valueStart + 1]; // good high

        // find fanin values
        int aStart = -1;
        int bStart = -1;
        if(fanin[gate * 4] != -1)
            aStart = blockOffset * 8 + fanin[gate * 4] * 8;     // fanin 0
        if(fanin[gate * 4 + 1] != -1)
            bStart = blockOffset * 8 + fanin[gate * 4 + 1] * 8; // fanin 1

        unsigned long agl   = 0x0;
        unsigned long agh   = 0x0;
        unsigned long bgl   = 0x0;
        unsigned long bgh   = 0x0;
        if (aStart != -1) {
            agl = value[aStart + 0];
            agh = value[aStart + 1];
        }
        if (bStart != -1) {
            bgl = value[bStart + 0];
            bgh = value[bStart + 1];
        }

        // perform good simulation for different gate types
        if (type == 0) // PI do not need simulation
            ;
        else if (type == 2 && level >= nLevels) { // PPI act as BUF
            gl = agl;
            gh = agh;
        }
        else if (type == 1 || type == 3 || type == 5) { // PO, PPO, and BUF
            gl = agl;
            gh = agh;
        }
        else if (type == 4) { // INV
            gl = agh;
            gh = agl;
        }
        else if (type == 6) { // AND
            gl = agl | bgl;
            gh = agh & bgh;
        }
        else if (type == 7) { // NAND
            gh = agl | bgl;
            gl = agh & bgh;
        }
        else if (type == 8) { // OR
            gl = agl & bgl;
            gh = agh | bgh;
        }
        else if (type == 9) { // NOR
            gl = agh | bgh;
            gh = agl & bgl;
        }

        // assign simulated good values back to global memory
        // faulty values and the same as good values
        value[valueStart + 0] = gl;
        value[valueStart + 1] = gh;
        value[valueStart + 2] = gl;
        value[valueStart + 3] = gh;
        __syncthreads();
    }
}

__device__ void randFill(
    int*             gateType
    , int            nLevels
    , unsigned long* value
    , unsigned long* split
) {

    int thId = threadIdx.x;
    int blId = blockIdx.x;

    // initialize random vector
    hiprandState rand_s;
    unsigned int seed = (unsigned int) clock64() * (threadIdx.x + 2);
    if (threadIdx.x == 0)
        hiprand_init(seed, blockIdx.x * threadIdx.x, 0, &rand_s);
    __syncthreads();

    for (int level = 0; level < nLevels; ++level) {
        int blockOffset = (nLevels * 2) * blockDim.x * blockIdx.x;
        int levelOffset = level * blockDim.x;
        int valueStart = (blockOffset + levelOffset) * 8 + threadIdx.x * 8;

        // find gate and gate type
        int gate = levelOffset + threadIdx.x;
        int type = gateType[gate];


        // PIs in both time frames and PPIs in the first time frame
        if (type == 0 || (type == 2 && level < nLevels)) {
            unsigned long gl = value[valueStart + 0] | value[valueStart + 3]; // low or D bar
            unsigned long gh = value[valueStart + 1] | value[valueStart + 2]; // high or D

            // fill dont care bits and random fill
            // unsigned long rand_int = (unsigned long)(hiprand_uniform(&rand_s) * (4294967295 + 0.999999));
            // JKY @ 20150416 choose random values in pool
            int rand_int = (int)(hiprand_uniform(&rand_s) * 100000);
            unsigned long evenS = split[(rand_int+thId) % 1000];
            unsigned long dc = ~(gl ^ gh); // find dont care bits
            gl = ((~evenS) & dc) | gl;
            gh = (evenS & dc) | gh;

            // write filled values back to global memory
            value[valueStart + 0] = gl & 0xFFFFFFFF;
            value[valueStart + 1] = gh & 0xFFFFFFFF;
            value[valueStart + 2] = gl & 0xFFFFFFFF; // faulty low same as good low
            value[valueStart + 3] = gh & 0xFFFFFFFF; // faulty high same as good high
            //printf("random %d\n", rand_int);
            //printValue(gate, gl, gh);

            if (type == 0) { // fill dont cares of the corresponding PI in time frame 2
                int levelOffset = (level + nLevels) * blockDim.x;
                int valueStart = (blockOffset + levelOffset) * 8 + threadIdx.x * 8;

                unsigned long gl = value[valueStart + 0] | value[valueStart + 3]; // low or D bar
                unsigned long gh = value[valueStart + 1] | value[valueStart + 2]; // high or D

                // fill dont care bits and random fill
                // unsigned long rand_int = (unsigned long)(hiprand_uniform(&rand_s) * 100000);
                // JKY @ 20150416 choose random values in pool
                int rand_int = (int)(hiprand_uniform(&rand_s) * 100000);
                unsigned long evenS = split[(rand_int+thId) % 1000];
                unsigned long dc = ~(gl ^ gh); // find dont care bits
                gl = ((~evenS) & dc) | gl;
                gh = (evenS & dc) | gh;

                // write filled values back to global memory
                value[valueStart + 0] = gl & 0xFFFFFFFF;
                value[valueStart + 1] = gh & 0xFFFFFFFF;
                value[valueStart + 2] = gl & 0xFFFFFFFF; // faulty low same as good low
                value[valueStart + 3] = gh & 0xFFFFFFFF; // faulty high same as good high
            }
        }
        else // break for all other gate types
            break;
    }
    __syncthreads();
}

__global__ void gpuGen (
    // circuit information
    int*             numRmnFaults
    , int*           faultGate
    , int*           faultType
    , int*           faultLine
    , int*           faultDetect
    , unsigned long* split
    , int*           gateType
    , int*           gateSplit
    , int*           fanin
    , int*           nLevels
    , unsigned long* value
    // backtrack
    , int*           bkStackSize
    , int*           bkValue
    , int*           bkPtr
    , int*           bkGate
    , bool*          bkFlipped
    // atpg parameters
    , int*           nDetect
    , int*           abLimit
    , int*           bkLimit
    , bool*          taMode
    , bool*          getMode
    // zero copy
    , int*           nInputs
    , unsigned long* pattern
) {
    int thId = threadIdx.x;
    int blId = blockIdx.x;

    // DEBUG timer
    clock_t timerStart;
    clock_t timerStop;
    double totaltime;
    /*if (blId == 0 && thId == 0) {
        timerStart = clock();
    }*/
    __syncthreads();



    // calculate how many faults in a GPU word
    int nFaults = 32 / (*nDetect);

    //clear all the value in memory
    clearValue(value, gateType, *nLevels, ~(0x0), 0 ,0);
    __syncthreads();

    // initialize objectives for 4 faults with 4 corresponding threads
    if (thId < nFaults)
        initialObjectives(nFaults // nFaults = 4 because of 32/8 (8-detects)
                , faultGate
                , faultType
                , faultLine
                , *nLevels
                , fanin
                , value
                , *nDetect
                , *getMode
                , ~(0x0) // initial all the clone (deadMask: 1 means need to be initial)
        );
    __syncthreads();// wait for initialize objectives to finish

    __shared__ int currentLevel;           // current execution level for all clones
    __shared__ unsigned long detected;     // detection status for each clone
    __shared__ unsigned long propagatable; // propagation value at output for each clone
    __shared__ unsigned long obj;          // objective flag for each clone
    __shared__ unsigned long deadClones;   // cehck whether there is any deadclone or not
    __shared__ int nIters;                 // abort limit counter


    // thread 0 update atpg level and status
    if (thId == 0) {
        currentLevel = (*nLevels) * 2 - 1;
        detected     = 0x0;
        obj          = 0x0;
        nIters       = 0;
        deadClones   = 0xFFFFFFFF;
    }
    __syncthreads();

    initBkParam(bkPtr, deadClones);
    __syncthreads();

    if (thId == 0) {
        deadClones   = 0;
    }
    __syncthreads();


    // test generation loop
    //                          |---<---backtrack limit---<----
    //                          |                             |
    // backtrace--->propagation--->(backtrack--->propagation)--->re-initial objective----
    //    |                                                                             |
    //    --------------<-----------------abort limit---------------<--------------------
    while (true) {
        if (blId == 1 && thId == 0) {
            timerStart = clock();
        }

        if(thId==0){
            nIters++;
            obj = 0x0;
        }
        __syncthreads();

        // backtrace
        backtrace(
            (*nLevels) * 2 - 1
            , obj
            , split
            , gateType
            , gateSplit
            , fanin
            //, fanout
            , *nLevels
            , value
            // for backtrack
            , *bkStackSize
            , bkValue
            , bkPtr
            , bkGate
            , bkFlipped
        );
        __syncthreads();

        //clear all the value of btl bth
        clearValue(value, gateType, *nLevels, ~(0x0), 0, 1);
        __syncthreads();

        if(thId == 0){
            propagatable = 0x0;
            detected     = 0x0;
        }
        __syncthreads();

        // propagation
        propagation(0
            , nFaults
            , propagatable
            , detected
            , gateType
            , gateSplit
            , fanin
            , *nLevels
            , value
            , faultGate
            , faultType
            , faultLine
        );
        __syncthreads();

        if(thId == 0)
            deadClones = ~detected & ~propagatable & 0xFFFFFFFF;
        __syncthreads();

        __shared__ unsigned long StackEmpty;
        __shared__ unsigned long NeedBK;
        __shared__ int bkIters;
        if (thId == 0) {
            StackEmpty = 0x0;
            bkIters    = 0;
            NeedBK     = ~(propagatable | detected | StackEmpty) & 0xFFFFFFFF;
        }
        __syncthreads();


        // perform backtrack
        if (NeedBK != 0x0) {
            backtrack(
                deadClones
                , *nLevels
                , value
                , *bkStackSize
                , bkValue
                , bkPtr
                , bkGate
                , bkFlipped
                , &StackEmpty
            );
            if(thId == 0){
                propagatable = 0x0;
                detected     = 0x0;
            }
            __syncthreads();
            propagation(0
                , nFaults
                , propagatable
                , detected
                , gateType
                , gateSplit
                , fanin
                , *nLevels
                , value
                , faultGate
                , faultType
                , faultLine
            );

            if(thId == 0)
                deadClones = ~detected & ~propagatable & 0xFFFFFFFF;
            __syncthreads();

            // reinitialize dead clones primary fault objectives
            if (thId < nFaults)
                initialObjectives(nFaults
                    , faultGate
                    , faultType
                    , faultLine
                    , *nLevels
                    , fanin
                    , value
                    , *nDetect
                    , *getMode
                    , deadClones
                );
        }

        // check for termination condition
        if ((detected == 0xFFFFFFFF) || nIters >= *abLimit){
            if(thId == 0)
                pattern[blId] = detected;
                //pattern[blId] = 0x0;
            break;
        }
        __syncthreads();

        // initBkParam(bkPtr, deadClones);
        // __syncthreads();
    }
    //__syncthreads();

    // DEBUG timer
    //if (blId == 0 && thId == 0) {
    //    timerStop = clock();
    //    printf("+ ATPG runtime %d s\n", (timerStop - timerStart)/CLOCKS_PER_SEC);
    //    timerStart = clock();
    //}
    //__syncthreads();

    // do fault simulation after test generation
    //faultSim (
    //    numRmnFaults
    //    , split
    //    , faultGate
    //    , faultType
    //    , faultLine
    //    , faultDetect
    //    , gateType
    //    , fanin
    //    , nLevels
    //   , value
    //    , nDetect
    //    , pattern
    //);

    // DEBUG timer
    //if (blId == 0 && thId == 0) {
    //    timerStop = clock();
    //    printf("+ Simulation runtime %d s\n", (timerStop - timerStart)/CLOCKS_PER_SEC);
    //    timerStart = clock();
    //}
    randFill(gateType, (*nLevels), value, split);
    __syncthreads();


    // pattern zero copy
    zeroCopy(gateType, value, pattern, *nLevels, *nInputs);

    // DEBUG timer
    //if (blId == 0 && thId == 0) {
    //    timerStop = clock();
    //    printf("+ Zero copy runtime %d s\n", (timerStop - timerStart)/CLOCKS_PER_SEC);
    //}
    __syncthreads();
}

// clear corresponding clones' values if bit of clearClones is 1
__device__ void clearValue(unsigned long *values
    , int* gateType
    , int nLevels
    , unsigned long clearClones
    , bool pp // if pp = 1, clear all the values exclude PI in timeframe1,2 and PPI in timeframe1
    , bool bt // if bt = 1, only clear backtrace value(btl,bth)
) {

    int thId = threadIdx.x;
    //int blId = blockIdx.x;
    int level = 0;

    while (level != nLevels * 2) {
        int blockOffset = ( nLevels * 2 ) * blockDim.x * blockIdx.x; //total gate in block
        int levelOffset = level * blockDim.x; //gate number per level
        int valueStart = (blockOffset + levelOffset) * 8 + threadIdx.x * 8; //the start point of gate

        if(bt == 0){
            //continue if in pp mode and the gate is PI or PPI(time frame 1)
            if(pp == 1 && gateType[levelOffset + thId] == 0){level++;continue;}
            if(pp == 1 && gateType[levelOffset + thId] == 2 && level < nLevels){level++;continue;}
            values[valueStart+0] &= ~clearClones;
            values[valueStart+1] &= ~clearClones;
            values[valueStart+2] &= ~clearClones;
            values[valueStart+3] &= ~clearClones;
            values[valueStart+4] &= ~clearClones;
            values[valueStart+5] &= ~clearClones;
            values[valueStart+6] &= ~clearClones;
            values[valueStart+7] &= ~clearClones;
        }
        else if(bt == 1){
            values[valueStart+4] &= ~clearClones;
            values[valueStart+5] &= ~clearClones;
        }

        level++;
    }
}

__device__ void zeroCopy(
    int*             gateType
    , unsigned long* values
    , unsigned long* pattern
    , int            nLevels
    , int            nInputs
) {
    // #######################################################################
    //   pattern sequence
    //
    //   start-- block0--    detected   [ unsigned long 32 bits            ]
    //        |  block1--    detected   [ unsigned long 32 bits            ]
    //        ...
    //        ...
    //   start-- block0-- PI0-- t0-- v0 [ unsigned long 32 bits            ]
    //        |        |     \    `- v1 [ 00100010010111010100101010101000 ]
    //        |        |      - t1-- v0 [                                  ]
    //        |        |          `- v1 [                                  ]
    //        |        +- PI1-- t0-- v0 [                                  ]
    //        |        |     \    `- v1 [                                  ]
    //        |        |      - t1-- v0 [                                  ]
    //        |        |          `- v1 [                                  ]
    //        |
    //        |        ...
    //        |
    //        |        `- PIn-- t0-- v0 [                                  ]
    //        |              \    `- v1 [                                  ]
    //        |               - t1-- v0 [                                  ]
    //        |                   `- v1 [                                  ]
    //
    //        ...
    //
    //        `
    //         -  block1-- PI0-- t0-- v0 [                                 ]
    //
    // #######################################################################

    //int thId = threadIdx.x;
    //int blId = blockIdx.x;
    int level = 0;
    int timeframe = 0;
    while (level != nLevels * 2) {

        // find gate type
        int gateLevelOffset = level * blockDim.x; //gate's number per level 
        int gateId = gateLevelOffset + threadIdx.x; 
        int type = gateType[gateId];

        if (type != 0 && type != 2) {
            timeframe = 1;
            level++;
            continue;
        }

        // find value starting point
        int gateBlockOffset = ( nLevels * 2 ) * blockDim.x * blockIdx.x;
            //gate's number per block(the total circuit gate's number * 2 timeframes ) * blockId
        int valueStart = (gateBlockOffset + gateLevelOffset + threadIdx.x) * 8;

        // find pattern starting point
        int patBlockOffset = nInputs * blockIdx.x; 
        int patPiOffset = gateId - timeframe * nLevels * blockDim.x;
        int patStart = (patBlockOffset + patPiOffset) * 4 + timeframe * 2;

        // write pattern
        pattern[patStart + 0 + gridDim.x] = values[valueStart + 0];
        pattern[patStart + 1 + gridDim.x] = values[valueStart + 1];

        // DEBUG littleshamoo
        //printValue(gateId, pattern[patStart + 0 + gridDim.x], pattern[patStart + 1 + gridDim.x]);

        level++;
    }
}

__device__ void printValue(int gateId, unsigned long v0, unsigned long v1)
{
    char vStr[33];
    vStr[32] = '\0';
    for (size_t i = 0; i < 32; ++i) {
        unsigned long v0bit = (v0 >> (31 - i)) & (unsigned long)0x01;
        unsigned long v1bit = (v1 >> (31 - i)) & (unsigned long)0x01;
        if (v0bit == 0 && v1bit == 0)
            vStr[i] = 'X';
        else if (v0bit == 1 && v1bit == 0)
            vStr[i] = '0';
        else if (v0bit == 0 && v1bit == 1)
            vStr[i] = '1';
        else
            vStr[i] = '?';
    }
    printf("Gate %d, value %s\n", gateId, vStr);
}

__device__ void propagation(
    int              currentLevel
    , int            nFaults
    , unsigned long& propagatable
    , unsigned long& detected
    , int*           gateType
    , int*           gateSplit
    , int*           fanin
    , int            nLevels
    , unsigned long* values
    , int*           faultGate
    , int*           faultType
    , int*           faultLine
) {
    int thId = threadIdx.x;
    int blId = blockIdx.x;

    __shared__ int SfaultGate[4];
    __shared__ int SfaultType[4];
    __shared__ int SfaultLine[4];
    __shared__ unsigned long Sactive[4];
    __shared__ unsigned long Spp;
    __shared__ unsigned long Sdetect;

    // update "propagatable" and "detected" without race condition
    if(thId < 4){
        SfaultGate[thId] = faultGate[blId * nFaults + thId]; //nFaults = 4
        SfaultType[thId] = faultType[blId * nFaults + thId];
        SfaultLine[thId] = faultLine[blId * nFaults + thId];
        Sactive[thId]    = 0x0;
        Spp              = 0x0;
        Sdetect          = 0x0;
    }

    //clear all the values( instead of PI(1,2) and PPI(1) ) before propagation
    clearValue(values, gateType, nLevels, ~(0x0), 1, 0);
    __syncthreads();

    for(int level=currentLevel; level != nLevels * 2; ++level){

        // determine offset first
        int blockOffset = ( nLevels * 2 ) * blockDim.x * blockIdx.x;
        int levelOffset = level * blockDim.x;

        // find gate and gate type
        int gate = levelOffset + threadIdx.x;
        int type = gateType[gate];
        if (type != -1){ // do operation if not empty gate
            int valueStart = (blockOffset + levelOffset) * 8 + threadIdx.x * 8;

            int aStart=-1, bStart=-1;
            if(fanin[gate * 4] != -1)
                aStart = blockOffset * 8 + fanin[gate * 4] * 8;     // fanin 0
            if(fanin[gate * 4 + 1] != -1)
                bStart = blockOffset * 8 + fanin[gate * 4 + 1] * 8; // fanin 1


            unsigned long l   = values[valueStart + 0];
            unsigned long h   = values[valueStart + 1];
            unsigned long d   = values[valueStart + 2];
            unsigned long b   = values[valueStart + 3];
            unsigned long btl = values[valueStart + 4];
            unsigned long bth = values[valueStart + 5];
            unsigned long pp  = values[valueStart + 6];
            unsigned long x   = ~(l | h | d | b);

            unsigned long al   = 0x0;
            unsigned long ah   = 0x0;
            unsigned long ad   = 0x0;
            unsigned long ab   = 0x0;
            unsigned long abtl = 0x0;
            unsigned long abth = 0x0;
            unsigned long app  = 0x0;
            unsigned long ax   = 0x0;
            unsigned long bl   = 0x0;
            unsigned long bh   = 0x0;
            unsigned long bd   = 0x0;
            unsigned long bb   = 0x0;
            unsigned long bbtl = 0x0;
            unsigned long bbth = 0x0;
            unsigned long bpp  = 0x0;
            unsigned long bx   = 0x0;

            if(aStart != -1){
                al   = values[aStart + 0];
                ah   = values[aStart + 1];
                ad   = values[aStart + 2];
                ab   = values[aStart + 3];
                abtl = values[aStart + 4];
                abth = values[aStart + 5];
                app  = values[aStart + 6];
                ax   = ~(al | ah | ad | ab);
            }

            if(bStart != -1){
                bl   = values[bStart + 0];
                bh   = values[bStart + 1];
                bd   = values[bStart + 2];
                bb   = values[bStart + 3];
                bbtl = values[bStart + 4];
                bbth = values[bStart + 5];
                bpp  = values[bStart + 6];
                bx   = ~(bl | bh | bd | bb);
            }

            //handle D B before calculation
            for(int i = 0; i < 4; ++i){
                //D B check activation (time frame 1)
                if( level < nLevels ){
                    if(gate == SfaultGate[i] && SfaultLine[i] != 0 && SfaultType[i] == 2){//find fault gate and STR
                        Sactive[i] = (SfaultLine[i]==1 ? al : bl)>>(i*8) & 0xFF;
                    }
                    else if(gate == SfaultGate[i] && SfaultLine[i] != 0 && SfaultType[i] == 3){//find fault gate and STF
                        Sactive[i] = (SfaultLine[i]==1 ? ah : bh)>>(i*8) & 0xFF;
                    }
                }
                //D B generation (time frame 2)
                else{
                    if(gate-nLevels*blockDim.x == SfaultGate[i] && SfaultLine[i] != 0 && SfaultType[i] == 2){//find fault gate and STR
                        if(SfaultLine[i] == 1){
                            ad |= Sactive[i]<<(i*8) & ah & 0xFF<<(i*8); //if fault be activated successful
                            ah = ( ah & ~(0xFF<<(i*8)) ) | ( ~ad & ah & 0xFF<<(i*8) ); //then h be cleaned
                        }
                        else{
                            bd |= Sactive[i]<<(i*8) & bh & 0xFF<<(i*8);
                            bh = ( bh & ~(0xFF<<(i*8)) ) | ( ~bd & bh & 0xFF<<(i*8) );
                        }
                    }
                    else if(gate-nLevels*blockDim.x == SfaultGate[i] && SfaultLine[i] != 0 && SfaultType[i] == 3){//find fault gate and STF
                        if(SfaultLine[i] == 1){
                            ab |= Sactive[i]<<(i*8) & al & 0xFF<<(i*8);
                            al = ( al & ~(0xFF<<(i*8)) ) | ( ~ab & al & 0xFF<<(i*8) );
                        }
                        else{
                            bb |= Sactive[i]<<(i*8) & bl & 0xFF<<(i*8);
                            bl = ( bl & ~(0xFF<<(i*8)) ) | ( ~bb & bl & 0xFF<<(i*8) );
                        }
                    }
                }
            }

            if (type == 0) // PI do not need propagation
                ;
            else if (type == 2 && level >= nLevels) { // PPI act as BUF
                l = al;
                h = ah;
                d = ad;
                b = ab;
                pp  = app;
            }
            // PO and PPO also check propagation and detection
            else if (type == 1 || type == 3) {
                l = al;
                h = ah;
                d = ad;
                b = ab;
                pp  = app;
                atomicOr((unsigned int*) &Spp ,pp);
                atomicOr((unsigned int*) &Sdetect ,d | b);
            }
            else if (type == 5) { // BUF
                l = al;
                h = ah;
                d = ad;
                b = ab;
                pp  = app;
            }
            else if (type == 4) { // INV
                l = ah;
                h = al;
                d = ab;
                b = ad;
                btl = abth;
                bth = abtl;
                pp  = app;
            }
            else if (type == 6) { // AND
                l = al | bl | ad & bb | ab & bd;
                h = ah & bh;
                d = ad & bd | ad & bh | ah & bd;
                b = ab & bb | ab & bh | ah & bb;
                pp = ax & (bd | bb) | bx & (ad | ab);
                pp |= ~al & bpp & x;
                pp |= ~bl & app & x;
            }
            else if (type == 7) { // NAND
                h = al | bl | ad & bb | ab & bd;
                l = ah & bh;
                b = ad & bd | ad & bh | ah & bd;
                d = ab & bb | ab & bh | ah & bb;
                pp = ax & (bd | bb) | bx & (ad | ab);
                pp |= ~al & bpp & x;
                pp |= ~bl & app & x;
            }
            else if (type == 8) { // OR
                l = al & bl;
                h = ah | bh | ad & bb | ab & bd;
                d = ad & bd | ad & bl | al & bd;
                b = ab & bb | ab & bl | al & bb;
                pp = ax & (bd | bb) | bx & (ad | ab);
                pp |= ~ah & bpp & x;
                pp |= ~bh & app & x;
            }
            else if (type == 9) { // NOR
                l = ah | bh | ad & bb | ab & bd;
                h = al & bl;
                d = ab & bb | ab & bl | al & bb;
                b = ad & bd | ad & bl | al & bd;
                pp = ax & (bd | bb) | bx & (ad | ab);
                pp |= ~ah & bpp & x;
                pp |= ~bh & app & x;
            }

            //handle D B after calculation

            for(int i = 0; i < 4; ++i){
                //D B check activation (time frame 1)
                if( level < nLevels ){
                    if(gate == SfaultGate[i] && SfaultLine[i] == 0 && SfaultType[i] == 2){//find fault gate and STR
                        Sactive[i] = l>>(i*8) & 0xFF;
                    }
                    else if(gate == SfaultGate[i] && SfaultLine[i] == 0 && SfaultType[i] == 3){//find fault gate and STF
                        Sactive[i] = h>>(i*8) & 0xFF;
                    }
                }
                //D B generation (time frame 2)
                else{
                    if(gate-nLevels*blockDim.x == SfaultGate[i] && SfaultLine[i] == 0 && SfaultType[i] == 2){//find fault gate and STR
                        d |= Sactive[i]<<(i*8) & h & 0xFF<<(i*8);
                        h = ( h & ~(0xFF<<(i*8)) ) | ( ~d & h & 0xFF<<(i*8) ) & 0xFFFFFFFF;
                    }
                    else if(gate-nLevels*blockDim.x == SfaultGate[i] && SfaultLine[i] == 0 && SfaultType[i] == 3){//find fault gate and STF
                        b |= Sactive[i]<<(i*8) & l & 0xFF<<(i*8);
                        l = ( l & ~(0xFF<<(i*8)) ) | ( ~b & l & 0xFF<<(i*8) ) & 0xFFFFFFFF;
                    }
                }
            }

            // l and h update above
            values[valueStart + 0] = l;
            values[valueStart + 1] = h;
            atomicOr((unsigned int*) &values[valueStart + 2] ,d);
            atomicOr((unsigned int*) &values[valueStart + 3] ,b);
            atomicOr((unsigned int*) &values[valueStart + 4] ,btl);
            atomicOr((unsigned int*) &values[valueStart + 5] ,bth);
            atomicOr((unsigned int*) &values[valueStart + 6] ,pp);

            if(aStart != -1){
                atomicOr((unsigned int*) &values[aStart + 0] ,al);
                atomicOr((unsigned int*) &values[aStart + 1] ,ah);
                atomicOr((unsigned int*) &values[aStart + 2] ,ad);
                atomicOr((unsigned int*) &values[aStart + 3] ,ab);
                atomicOr((unsigned int*) &values[aStart + 6] ,app);
            }

            if(bStart != -1){
                atomicOr((unsigned int*) &values[bStart + 0] ,bl);
                atomicOr((unsigned int*) &values[bStart + 1] ,bh);
                atomicOr((unsigned int*) &values[bStart + 2] ,bd);
                atomicOr((unsigned int*) &values[bStart + 3] ,bb);
                atomicOr((unsigned int*) &values[bStart + 6] ,bpp);
            }
        }
        __syncthreads();
        if(thId == 0){
            propagatable |= Spp;
            detected |= Sdetect;
        }
        __syncthreads();
    }
}

__device__ void backtrace(
    int              currentLevel
    , unsigned long& obj
    , unsigned long* split
    , int*           gateType
    , int*           gateSplit
    , int*           fanin
    //, int*           fanout
    , int            nLevels
    , unsigned long* value
    // for backtrack
    , int            bkStackSize
    , int*           bkValue
    , int*           bkPtr
    , int*           bkGate
    , bool*          bkFlipped
) {
    int thId = threadIdx.x;
    int blId = blockIdx.x;
    // use share memory to track PI assignments
    __shared__ int           assignedId[128]; //gate Id will be assigned value
    __shared__ unsigned long assignedV0[128]; //current assign V0
    __shared__ unsigned long assignedV1[128]; //current assign V1
    __shared__ unsigned long prevV0[128]; //previous V0 value
    __shared__ unsigned long prevV1[128]; //previous V1 value

    for(int level=currentLevel; level != -1; level--){

        // clear assigned gates
        assignedId[thId] = -1;
        assignedV0[thId] = 0x0;
        assignedV1[thId] = 0x0;
        prevV0[thId]     = 0x0;
        prevV1[thId]     = 0x0;

        // determine offset
        int blockOffset = ( nLevels * 2) * blockDim.x * blockIdx.x; //#gate per block
        int levelOffset = level * blockDim.x; //#gate per level

        // find gate and gate type
        int gate = levelOffset + threadIdx.x; //gateId
        if(gate < 0)break;// JKY @ 20150113 level may not updated when checking != -1
        int type = gateType[gate];

        if (type != -1){ // empty gate. cannot use continue because traceAssignment need all the thread

            //rand() is not available, use this instead.
            __shared__ hiprandState rand_s;
            unsigned int seed = (unsigned int) clock64() * (thId+2);
            if(thId==0)hiprand_init(seed, blId*thId, 0, &rand_s);

            // find split vector
            int rand_int = (int)(hiprand_uniform(&rand_s) * 100000);
            unsigned long evenS = split[rand_int % 1000];

            rand_int = (int)(hiprand_uniform(&rand_s) * 1000000);
            unsigned long s = split[gateSplit[gate] * 1000 + rand_int % 1000];

            // stem backtrace for all types of gates
            int valueStart = (blockOffset + levelOffset) * 8 + threadIdx.x * 8;
            unsigned long stemBtl = value[valueStart + 4]; //btl
            unsigned long stemBth = value[valueStart + 5]; //bth
            unsigned long tempBtl = stemBtl;
            unsigned long tempBth = stemBth;

            //Btl & Bth no conflict or random choose
            stemBtl = (tempBtl & ~tempBth) | (tempBtl & tempBth & evenS);  
            stemBth = (tempBth & ~tempBtl) | (tempBtl & tempBth & ~evenS);
            value[valueStart + 4] = stemBtl;
            value[valueStart + 5] = stemBth;

            int aStart=-1, bStart=-1;
            if(fanin[gate * 4] != -1)
                aStart = blockOffset * 8 + fanin[gate * 4] * 8;     // fanin 0
            if(fanin[gate * 4 + 1] != -1)
                bStart = blockOffset * 8 + fanin[gate * 4 + 1] * 8; // fanin 1

            unsigned long l   = value[valueStart + 0];
            unsigned long h   = value[valueStart + 1];
            unsigned long d   = value[valueStart + 2];
            unsigned long b   = value[valueStart + 3];
            unsigned long btl = value[valueStart + 4];
            unsigned long bth = value[valueStart + 5];
            unsigned long pp  = value[valueStart + 6];
            unsigned long al  = 0x0;
            unsigned long ah  = 0x0;
            unsigned long ad  = 0x0;
            unsigned long ab  = 0x0;
            unsigned long abtl= 0x0;
            unsigned long abth= 0x0;
            unsigned long app = 0x0;
            unsigned long ax  = 0x0;
            unsigned long bl  = 0x0;
            unsigned long bh  = 0x0;
            unsigned long bd  = 0x0;
            unsigned long bb  = 0x0;
            unsigned long bbtl= 0x0;
            unsigned long bbth= 0x0;
            unsigned long bpp = 0x0;
            unsigned long bx  = 0x0;
            unsigned long x   = ~(l | h | d | b);
            if(aStart != -1){
                al   = value[aStart + 0];
                ah   = value[aStart + 1];
                ad   = value[aStart + 2];
                ab   = value[aStart + 3];
                abtl = value[aStart + 4];
                abth = value[aStart + 5];
                app  = value[aStart + 6];
                ax   = ~(al | ah | ad | ab);
            }

            if(bStart != -1){
                bl   = value[bStart + 0];
                bh   = value[bStart + 1];
                bd   = value[bStart + 2];
                bb   = value[bStart + 3];
                bbtl = value[bStart + 4];
                bbth = value[bStart + 5];
                bpp  = value[bStart + 6];
                bx   = ~(bl | bh | bd | bb);
            }
            // backtrace equations for all types of gates
            if (type == 0 || (type == 2 && level < nLevels)) { // PI and PPI do assignment instead
                // need to check both l and b, h and d
                unsigned long Low       = l | b;
                unsigned long High      = h | d;
                unsigned long LowCheck  = (Low ^ btl) & x; //check is Low and btl conflict or not
                unsigned long HighCheck = (High ^ bth) & x;
                //!=0x0 means x = 11...11(can assign value) and "Low=0, btl=1" or "Low=1,btl=0"
                if (LowCheck != 0x0 || HighCheck != 0x0) {
                    assignedId[threadIdx.x] = gate;
                    assignedV0[threadIdx.x] = (btl & x) | Low; //assign 0
                    assignedV1[threadIdx.x] = (bth & x) | High; //assign 1
                    prevV0[threadIdx.x]     = Low; //previous Low value
                    prevV1[threadIdx.x]     = High; //previous High value
                    value[valueStart + 0]   |= btl & x; //update value
                    value[valueStart + 1]   |= bth & x;
                }
            }
            else if (type == 2 && level >= nLevels) {//PPI in time frame 2 do backtrace as BUF
                abtl = btl;
                abth = bth;
                app  = pp;
            }
            else if (type == 1 || type == 3 || type == 5) { // PO, PPO, and BUF
                abtl = btl;
                abth = bth;
                app  = pp;
            }
            else if (type == 4) { // INV
                abtl = bth;
                abth = btl;
                app  = pp;
            }
            else if (type == 6) { // AND
                abth  = ax &  (bd | bb) & pp & ~obj;
                abth |= ax & ~(bl | bb) & bth;
                abth |= ax &  (bd | bb) & bth;
                abtl  = ax &  bx  & btl & s;
                abtl |= ax &  (bh | bd | bb) & btl;
                bbth  = bx &  (ad | ab) & pp & ~obj;
                bbth |= bx & ~(al | ab) & bth;
                bbth |= bx &  (ad | ab) & bth & s;
                bbtl  = bx &  ax  & btl & ~s;
                bbtl |= bx &  (ah | ad | ab) & btl;
                atomicOr((unsigned int*) &obj ,abth | abtl | bbth | bbtl);

                app   = app & ~bpp & pp & ~obj;
                app  |= app & bpp  & pp & ~obj & s;
                bpp   = bpp & ~app & pp & ~obj;
                bpp  |= app & bpp  & pp & ~obj & ~s;
            }
            else if (type == 7) { // NAND
                abth  = ax &  (bd | bb) & pp & ~obj;
                abth |= ax & ~(bl | bb) & btl;
                abth |= ax &  (bd | bb) & btl;
                abtl  = ax &  bx  & bth & s;
                abtl |= ax &  (bh | bd | bb) & bth;
                bbth  = bx &  (ad | ab) & pp & ~obj;
                bbth |= bx & ~(al | ab) & btl;
                bbth |= bx &  (ad | ab) & btl & s;
                bbtl  = bx &  ax  & bth & ~s;
                bbtl |= bx &  (ah | ad | ab) & bth;
                atomicOr((unsigned int*) &obj ,abth | abtl | bbth | bbtl);

                app   = app & ~bpp & pp & ~obj;
                app  |= app & bpp  & pp & ~obj & s;
                bpp   = bpp & ~app & pp & ~obj;
                bpp  |= app & bpp  & pp & ~obj & ~s;
            }
            else if (type == 8) { // OR
                abtl  = ax &  (bd | bb) & pp & ~obj;
                abtl |= ax & ~(bh | bd) & btl;
                abtl |= ax &  (bd | bb) & btl;
                abth  = ax &  bx  & bth & s;
                abth |= ax &  (bl | bd | bb) & bth;
                bbtl  = bx &  (ad | ab) & pp & ~obj;
                bbtl |= bx & ~(ah | ad) & btl;
                bbtl |= bx &  (ad | ab) & btl;
                bbth  = bx &  ax  & bth & ~s;
                bbth |= bx &  (al | ad | ab) & bth;
                atomicOr((unsigned int*) &obj ,abth | abtl | bbth | bbtl);

                app   = app & ~bpp & pp & ~obj;
                app  |= app & bpp  & pp & ~obj & s;
                bpp   = bpp & ~app & pp & ~obj;
                bpp  |= app & bpp  & pp & ~obj & ~s;
            }
            else if (type == 9) { // NOR
                abtl  = ax &  (bd | bb) & pp & ~obj;
                abtl |= ax & ~(bh | bd) & bth;
                abtl |= ax &  (bd | bb) & bth;
                abth  = ax &  bx  & btl & s;
                abth |= ax &  (bl | bd | bb) & btl;
                bbtl  = bx &  (ad | ab) & pp & ~obj;
                bbtl |= bx & ~(ah | ad) & bth;
                bbtl |= bx &  (ad | ab) & bth;
                bbth  = bx &  ax  & btl & ~s;
                bbth |= bx &  (al | ad | ab) & btl;
                atomicOr((unsigned int*) &obj ,abth | abtl | bbth | bbtl);

                app   = app & ~bpp & pp & ~obj;
                app  |= app & bpp  & pp & ~obj & s;
                bpp   = bpp & ~app & pp & ~obj;
                bpp  |= app & bpp  & pp & ~obj & ~s;
            }

            // update fanin value // JKY @ 20141104
            if(aStart != -1){
                atomicOr((unsigned int*) &value[aStart + 4] ,abtl);
                atomicOr((unsigned int*) &value[aStart + 5] ,abth);
                atomicOr((unsigned int*) &value[aStart + 6] ,app);
            }
            if(bStart != -1){
                atomicOr((unsigned int*) &value[bStart + 4] ,bbtl);
                atomicOr((unsigned int*) &value[bStart + 5] ,bbth);
                atomicOr((unsigned int*) &value[bStart + 6] ,bpp);
            }
        }
        __syncthreads();

        // track assignments for backtrack
        trackAssignment(
            assignedId
            , assignedV0
            , assignedV1
            , prevV0
            , prevV1
            , bkStackSize
            , bkValue
            , bkPtr
            , bkGate
            , bkFlipped
        );
        __syncthreads();
    }
}


__device__ void trackAssignment(
    int*             assignedId
    , unsigned long* assignedV0
    , unsigned long* assignedV1
    , unsigned long* prevV0
    , unsigned long* prevV1
    , int            bkStackSize
    , int*           bkValue
    , int*           bkPtr
    , int*           bkGate
    , bool*          bkFlipped
) {
    int thId = threadIdx.x;
    int blId = blockIdx.x;

    // handle 128 gates separately
    for (int i = 0; i < blockDim.x; ++i) {

        // skip if no assignments
        int skip = false;
        if (assignedId[i] == -1) {
            skip = true;
        }
        if (skip)
            continue;

        int gateId = assignedId[i];
        unsigned long pV0 = prevV0[i];
        unsigned long pV1 = prevV1[i];
        unsigned long aV0 = assignedV0[i];
        unsigned long aV1 = assignedV1[i];
        if (thId < 32) { // only need 32 threads to handle 32 clones
            // shift bits to find the clone of interest
            int vpV0 = (int)((pV0 >> (thId)) & 0x01);
            int vpV1 = (int)((pV1 >> (thId)) & 0x01);
            int vaV0 = (int)((aV0 >> (thId)) & 0x01);
            int vaV1 = (int)((aV1 >> (thId)) & 0x01);
            if (vpV0 == vaV0 && vpV1 == vaV1) //assign value like previous, do nothing
                ;
            else {
                // find assignment value
                int va = 0;
                if (vaV1 != 0) //vaV1 = 1, va = 1
                    va = 1;
                // put assignment into the stack
                int offset = (blId * 32 + thId) * bkStackSize; //bkValue offset
                int ptr = bkPtr[blId * 32 + thId]; //back track ptr
                bkPtr[blId * 32 + thId]++; //put new one into stack, Ptr++
                ptr++;
                bkValue[offset + ptr] = va;
                bkGate[offset + ptr] = gateId;
                bkFlipped[offset + ptr] = false;
            }
        }
        __syncthreads();
    }
}


__device__ void backtrack(
    unsigned long    deadClones
    , int            nLevels
    , unsigned long* value
    , int            bkStackSize
    , int*           bkValue
    , int*           bkPtr
    , int*           bkGate
    , bool*          bkFlipped
    , unsigned long* StackEmpty
) {
    int thId = threadIdx.x;
    int blId = blockIdx.x;
    int dead;

    if(thId < 32)
        dead = (int)(deadClones >> thId) & 0x01; //check this clone is dead or not

    __syncthreads();// JKY @ 20150113

    if (thId >= 32)
        ;
    // clone not dead, no need for backtrack
    else if( dead == 0 )
        ;
    else {
        int offset = (blId * 32 + thId) * bkStackSize;
        int ptr = bkPtr[blId * 32 + thId];
        if (ptr == -1 && ptr < bkStackSize) // stack empty
            atomicOr( (unsigned int*) StackEmpty ,(unsigned int)(0x01 << thId));
        else {
            // find unflipped assignment
            while (ptr > -1 && ptr < bkStackSize) {
                int va = bkValue[offset + ptr];
                int gateId  = bkGate[offset + ptr];
                bool flipped = bkFlipped[offset + ptr];
                int blockOffset = (nLevels * 2) * blockDim.x * blockIdx.x;
                int valueStart = (blockOffset + gateId) * 8;
                if (flipped) { // reset gate value to X //flipped = true means has been flipped
                    atomicAnd( (unsigned int*) &value[valueStart + 0] ,(unsigned int)(~(0x01 << thId)) );
                    atomicAnd( (unsigned int*) &value[valueStart + 1] ,(unsigned int)(~(0x01 << thId)) );
                    bkPtr[blId * 32 + thId]--;
                }
                else {
                    // determine backtrack value
                    if (va == 0){
                        atomicAnd( (unsigned int*) &value[valueStart + 0] ,(unsigned int)(~(0x01 << thId)) );
                        atomicOr ( (unsigned int*) &value[valueStart + 1] ,(unsigned int)( (0x01 << thId)) );
                    }
                    else{
                        atomicOr ( (unsigned int*) &value[valueStart + 0] ,(unsigned int)( (0x01 << thId)) );
                        atomicAnd( (unsigned int*) &value[valueStart + 1] ,(unsigned int)(~(0x01 << thId)) );
                    }
                    bkFlipped[offset + ptr] = 1;
                    break;
                }
                ptr = bkPtr[blId * 32 + thId];
            }
            //__syncthreads();
        }
    }
    __syncthreads();
}

__device__ void initBkParam(int* bkPtr, unsigned long & deadClones)
{
    // first 32 threads initialize backtrack parameters
    // since we have 32 clones. If number of threads is
    // less than 32, we loop available threads.
    int nClones = 32;
    if(threadIdx.x < 32)
        if(deadClones & (0x01<<threadIdx.x) != 0x0)
            bkPtr[blockIdx.x * nClones + threadIdx.x] = -1;
}

__device__ void initialObjectives (
    int              nFaults
    , int*           faultGate
    , int*           faultType
    , int*           faultLine
    , int            nLevels
    , int*           fanin
    , unsigned long* value
    , int            nDetect
    , bool           getMode
    , unsigned long  deadMask
) {

    int thId = threadIdx.x;
    int blId = blockIdx.x;

    // generate mask that has number of ones equal to nDetect
    unsigned long mask = 0x0;
    for (int i = 0; i < nDetect; ++i) {
        mask <<= 1;
        mask |= 0x1;
    }
    // shift to the position that is used by this thread
    mask <<= nDetect * thId;// JKY @ 20141101 thId

    int gate = faultGate[blId * nFaults + thId];
    int type = faultType[blId * nFaults + thId];
    int line = faultLine[blId * nFaults + thId];
    unsigned long t1btl = 0x0;
    unsigned long t1bth = 0x0;
    unsigned long t2btl = 0x0;
    unsigned long t2bth = 0x0;
    if (type == 2) { // slow-to-rise
        t1btl = ~(0x0);
        t2bth = ~(0x0);
    }
    else if (type == 3) { // slow-to-fall
        t1bth = ~(0x0);
        t2btl = ~(0x0);
    }

    int btGate = gate; // find backtrace gate. Default output of the faulty gate
    if (line > 0) //find fanin gate (fault on input)
        btGate = fanin[gate * 4 + line - 1];

    // find btGate in time frame 2
    int nGates = blockDim.x * nLevels; //total gate's number  of t1 circuit(#gate/level * total level)
    int nGatesPerFrame = nGates;
    int btGateT2 = btGate + nGatesPerFrame;

    // set backtrace value
    // value index: 0: L, 1: H, 2: D, 3: B, 4: btl, 5: bth, 6: pp, 7: x
    // deadMask are used to give second chance to dead clones
    // If the mask is 0, it means the clone is dead. Else it's alive and no
    // values will be changed

    atomicOr ( (unsigned int*) &value[blId * nGates * 8 * 2 + btGate * 8 + 4] ,t1btl & mask & deadMask);
    atomicOr ( (unsigned int*) &value[blId * nGates * 8 * 2 + btGate * 8 + 5] ,t1bth & mask & deadMask);
    atomicOr ( (unsigned int*) &value[blId * nGates * 8 * 2 + btGateT2 * 8 + 4] ,t2btl & mask & deadMask);
    atomicOr ( (unsigned int*) &value[blId * nGates * 8 * 2 + btGateT2 * 8 + 5] ,t2bth & mask & deadMask);
}

void SwkGpuAtpg::allocAtpgParameters() {
    hipMalloc((void**)&gpuNDetect_, sizeof(int));
    hipMalloc((void**)&gpuDcLimit_, sizeof(int));
    hipMalloc((void**)&gpuBkLimit_, sizeof(int));
    hipMalloc((void**)&gpuTaMode_,  sizeof(bool));
    hipMalloc((void**)&gpuGetMode_, sizeof(bool));

    hipMemcpy(gpuNDetect_, &nDetect_, sizeof(int),  hipMemcpyHostToDevice);
    hipMemcpy(gpuDcLimit_, &abLimit_, sizeof(int),  hipMemcpyHostToDevice);
    hipMemcpy(gpuBkLimit_, &bkLimit_, sizeof(int),  hipMemcpyHostToDevice);
    hipMemcpy(gpuTaMode_,  &taMode_,  sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(gpuGetMode_, &getMode_, sizeof(bool), hipMemcpyHostToDevice);

    // for backtrack memory allocation. By littleshamoo
    //
    //                                               bkStackSize
    //                                      ______________/\_____________
    //                                     /                             \
    //   gpuBkValue_ -- block0 --  clone0  [ | | | | | | | | | | ... | | ]
    //               |         +-  clone1  [ | | | | | | | | | | ... | | ]
    //               |         +-  clone2  [ | | | | | | | | | | ... | | ]
    //               ...       ...
    //               |         `-  clone31 [ | | | | | | | | | | ... | | ]
    //               |
    //               `- block1 --  clone0  [ | | | | | | | | | | ... | | ]
    //
    int nClones = nBlocks_ * 32;
    hipMalloc((void**)&gpuBkStackSize_, sizeof(int));
    hipMalloc((void**)&gpuBkValue_,     sizeof(int)  * nClones * bkStackSize_);
    hipMalloc((void**)&gpuBkPtr_,       sizeof(int)  * nClones);
    hipMalloc((void**)&gpuBkGate_,      sizeof(int)  * nClones * bkStackSize_);
    hipMalloc((void**)&gpuBkFlipped_,   sizeof(bool) * nClones * bkStackSize_);
    hipMemcpy(gpuBkStackSize_, &bkStackSize_, sizeof(int), hipMemcpyHostToDevice);
}
