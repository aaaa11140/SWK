#include "hip/hip_runtime.h"
#include "SddCudaSim.h"
#include "ArgSim.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <cutil_inline.h>
#include <iomanip>
#include <stdlib.h>
#include <stdio.h>
#define  SBA_thNum      512 // SBA = Static Bound Analysis
#define  SBA_bkNum      2
#define  LS_thNum       512 // Logic simulation
#define  LS_bkNum       1
#define  DBA_thNum      256 // DBA = Dynamic Bound Analysis
#define  DBA_bkNum      128
#define  EVB_thNum      128 // EVB = Evaluation Bound
#define  EVB_bkNum     256
#define  UFS_bkNum      64 // useless
#define  TFS_thGroup    1
#define  FC_thNum       1024    // number of threads to Compact faults in a block
#define  SFD_thNum      256 // SFD build thread Number
#define  SFD_bkNum      256

//#define delta 0.6
using namespace std;
using namespace IntfNs;

char getBitValue(PatValue& l_,PatValue& h_, const int& bitIdx);
// print function
void PrintBinaryValue(PatValue Val0,PatValue Val1);
void PrintDict(char* partialDict_d,int i, CircuitInfo* cirInfo);
void PrintSFD(char* partialDict_d,int* SFD,int* SFD_d,vector<unsigned int>& RmnfaultList,int patLoop,int patNum, CircuitInfo* cirInfo);

texture<unsigned int, 1, hipReadModeElementType> gDum2Ori_t;
texture<unsigned int, 1, hipReadModeElementType> cirInfo_t;
__global__ void staticBoundAnalysis(unsigned int* gTypeOri_d,float* dList_d,unsigned int* foArrayOri_d,unsigned int* foIdxArrayOri_d,unsigned int* foOffsetOri_d,float* PT_UBLB_d,float* Ttc_d,
                          unsigned int* gFiOri_d,unsigned int* gStrOnLvlOri_d,float* ATUB_d);
__global__ void logicSim(unsigned int* gTypeOri_d,unsigned int* gFiOri_d,unsigned int* gStrOnLvlOri_d,PatValue* pat_t0_d,PatValue* pat_t1_d,PatValue* pat_t0_z,PatValue* pat_t1_z,
                          PatValue* val_d,unsigned int* Rmnfault_d,unsigned int* Rmnfault_z,unsigned int* RmnfNum_d,unsigned int RmnfNum,unsigned int RmnpNum,int patLoop);
__global__ void dynamicBoundAnalysis(unsigned int* gTypeOri_d,unsigned int* gFiOri_d,float* at_d,unsigned int* gStrOnLvlOri_d,PatValue* val_d,float* dList_d,int currLvl);

__global__ void evalLBCriteria(unsigned int* gTypeOri_d,unsigned int* gFiOri_d,float* at_d,float* ATUB_d,float* PT_UBLB_d,float* Ttc_d,float* dList_d,unsigned int* gStrOnLvlOri_d,
                               PatValue* val_d,unsigned int* fList_d,char* partialDict_d,unsigned int* Rmnfault_d,unsigned int* RmnfNum_d,bool* fRdn_d,unsigned int iterNum,float delta);
__global__ void fCompact1(unsigned int* fMask_d,unsigned int* fSum_d, unsigned int* bSum_d,unsigned int* RmnfNum_d);
__global__ void fCompact2(unsigned int* fSum_d, unsigned int* bSum_d,unsigned int FC_bkNum);
__global__ void uniformAdd(unsigned int* fMask_d,unsigned int* fSum_d, unsigned int* bSum_d,
                           unsigned int* Rmnfault_d,unsigned int* RmnfNum_d,unsigned int* LBRmnfault_d,unsigned int* LBRmnfNum_d,bool afterLB);
__global__ void untimedFaultSim(unsigned int* gTypeDum_d,unsigned int* gFiDum_d,unsigned int* foArrayDum_d,unsigned int* foIdxArrayDum_d,unsigned int* foOffsetDum_d,
                                unsigned int* gStrOnLvlDum_d,PatValue* val_d,unsigned int* fList_d,char* partialDict_d,unsigned int* Rmnfault_d,unsigned int* RmnfNum_d,
                                PatValue* twoLvlval_d,unsigned int* twoLvlfG_d,unsigned int* EventList_d,unsigned int* fLvl_d);
__global__ void evalUBCriteria(unsigned int* gTypeOri_d,unsigned int* gFiOri_d,float* at_d,float* ATUB_d,float* PT_UBLB_d,float* Ttc_d,float* dList_d,
                               unsigned int* gStrOnLvlOri_d,unsigned int* fList_d,char* partialDict_d,unsigned int* Rmnfault_d,unsigned int* RmnfNum_d,float delta);
__global__ void actualPathDelayCal(unsigned int* gTypeDum_d,unsigned int* gFiDum_d,unsigned int* foArrayDum_d,unsigned int* foIdxArrayDum_d,unsigned int* foOffsetDum_d,
                                   float* at_d,float* Ttc_d,float* dList_d,unsigned int* gStrOnLvlDum_d,PatValue* val_d,unsigned int* fList_d,unsigned int* Rmnfault_d,
                                   unsigned int* RmnfNum_d,char* partialDict_d,PatValue* twoLvlval_d,float* twoLvlat_d,unsigned int* twoLvlfG_d,unsigned int* EventList_d,
                                   unsigned int* fLvl_d,float* ATUB_d,float* PT_UBLB_d,float delta);

__global__ void SFDRdnPatBuild(char* partialDict_d,int* SFD_d,int* SFD_z,unsigned int* LBRmnfault_d,unsigned int* LBRmnfNum_d,
                                    bool* pRdn_d,unsigned int RmnpNum,unsigned int* fMask_d,int patLoop,int mode);
__global__ void SFDAnalysis(char* partialDict_d,int* SFD_d,int* SFD_z,unsigned int* fMask_d,unsigned int* Rmnfault_d,unsigned int* RmnfNum_d,
                            bool* pRdn_d,unsigned int RmnpNum,int patLoop,int iterNum,int mode);
__device__ bool isInv(unsigned int gType);
__device__ void evalGate(unsigned int gateId,unsigned int gType,unsigned int gFiOri0,unsigned int gFiOri1,unsigned int gFiOri2,unsigned int gFiOri3,
                         PatValue* val_d,PatValue hold_capture,int timeframe);
__device__ void evalGate(unsigned int gType,PatValue fi0l_,PatValue fi0h_,PatValue fi1l_,PatValue fi1h_,PatValue fi2l_,PatValue fi2h_,PatValue fi3l_,PatValue fi3h_,PatValue* twoLvlval_d);
__device__ char getTrans(PatValue t0_l_,PatValue t0_h_,PatValue t1_l_,PatValue t1_h_,int bitIdx); // 0 = Rising ; 1 = Falling ; 2 = static
__device__ char getFiNum(unsigned int gType);
__device__ char transType(unsigned int gType, char trans);
__device__ bool getCtrl(unsigned int gType, char trans);
__device__ char getBV(PatValue pv,int bitIdx);
__device__ char getV(PatValue fl_,PatValue fh_,PatValue gl_, PatValue gh_,unsigned int bitIdx);
//{{{ void cudaSimulation()
void SddCudaSim::cudaSimulation(){

    cout << " ==========================================================" << endl;
    cout << " =                  Start Cuda Sim                        =" << endl;
    cout << " ==========================================================" << endl;
    fout << " ==========================================================" << endl;
    fout << " =                  Start Cuda Sim                        =" << endl;
    fout << " ==========================================================" << endl;
    unsigned int timer = 0;
    (cutCreateTimer(&timer));
    (cutStartTimer(timer));
    int idev = 0;
    hipSetDevice(idev);
    hipDeviceReset();
    // Memory setup
    // _d means pointer on device
    // _s means pointer on device share memory
    CircuitInfo*    cirInfo_d;
    unsigned int*   gDum2Ori_d;
    // Ori Circuit
    unsigned int*   gTypeOri_d;
    unsigned int*   gFiOri_d;
    unsigned int*   gStrOnLvlOri_d;
    // Dum Circuit
    unsigned int*   gTypeDum_d;
    unsigned int*   gFiDum_d;
    unsigned int*   gStrOnLvlDum_d;
    // Fanout
    unsigned int*   foArrayOri_d;
    unsigned int*   foIdxArrayOri_d;
    unsigned int*   foOffsetOri_d;

    unsigned int*   foArrayDum_d;
    unsigned int*   foIdxArrayDum_d;
    unsigned int*   foOffsetDum_d;
    // delay and path delay
    float*          dList_d;
    float*          PT_UBLB_d;
    float*          ATUB_d;
    float*          Ttc_d;
    float*          at_d;
    // value
    PatValue*       val_d;
    // dictionary
    char*           partialDict_d;
    // fault
    unsigned int*   fList_d;
    // Redundant fault is fault that not detect by anything (U faults without any pattern ID)
    // This kind of faults won't be calculated after second iteration
    // In second iteration Redudant Fault are faults don't use bond Analysis
    bool*           fRdn_d;  // to see if fault is redundant
    // for pdSim
    unsigned int*   fLvl_d;
    PatValue*       twoLvlval_d;
    float*          twoLvlat_d;
    unsigned int*   twoLvlfG_d;
    unsigned int*   EventList_d;
    unsigned int*   RmnfNum_d;
    unsigned int*   LBRmnfNum_d;
    // for fault compaction
    int             FC_bkNum = (cirInfo->fNum - 1)/FC_thNum + 1;
    unsigned int*   fSum_d;     // fault number accumulate
    unsigned int*   fMask_d;     // fault should remained
    unsigned int*   bSum_d;
    //{{{ hipMalloc & cpy
    hipMalloc((void**)&cirInfo_d   ,sizeof(CircuitInfo));
    hipMalloc((void**)&gDum2Ori_d  ,sizeof(unsigned int)*(cirInfo->DumgateNum));
    // Ori Circuit
    hipMalloc((void**)&gTypeOri_d     ,sizeof(unsigned int)*(cirInfo->OrigateNum));
    hipMalloc((void**)&gFiOri_d       ,sizeof(unsigned int)*(cirInfo->OrigateNum)*4);
    hipMalloc((void**)&gStrOnLvlOri_d  ,sizeof(unsigned int)*(cirInfo->cirlvl + 1));
    // Dum Circuit
    hipMalloc((void**)&gTypeDum_d     ,sizeof(unsigned int)*(cirInfo->DumgateNum));
    hipMalloc((void**)&gFiDum_d       ,sizeof(unsigned int)*(cirInfo->DumgateNum)*4);
    hipMalloc((void**)&gStrOnLvlDum_d ,sizeof(unsigned int)*(cirInfo->cirlvl + 1));
    // fanout
    hipMalloc((void**)&foArrayOri_d      ,sizeof(unsigned int)*(foOffsetOri[cirInfo->OrigateNum]));
    hipMalloc((void**)&foIdxArrayOri_d   ,sizeof(unsigned int)*(foOffsetOri[cirInfo->OrigateNum]));
    hipMalloc((void**)&foOffsetOri_d     ,sizeof(unsigned int)*(cirInfo->OrigateNum + 1));

    hipMalloc((void**)&foArrayDum_d      ,sizeof(unsigned int)*(foOffsetDum[cirInfo->DumgateNum]));
    hipMalloc((void**)&foIdxArrayDum_d   ,sizeof(unsigned int)*(foOffsetDum[cirInfo->DumgateNum]));
    hipMalloc((void**)&foOffsetDum_d     ,sizeof(unsigned int)*(cirInfo->DumgateNum + 1));
    // delay and path calculation
    hipMalloc((void**)&dList_d      ,sizeof(float)*(cirInfo->OrigateNum)*8);
    hipMalloc((void**)&PT_UBLB_d    ,sizeof(float)*(cirInfo->OrigateNum)*4);
    hipMalloc((void**)&Ttc_d        ,sizeof(float));
    hipMalloc((void**)&ATUB_d      ,sizeof(float)*(cirInfo->OrigateNum)*2);
    hipMalloc((void**)&at_d         ,sizeof(float)*(cirInfo->OrigateNum)*paraPatNum);
    // fault
    hipMalloc((void**)&fList_d      ,sizeof(float)*(cirInfo->fNum)*3);
    hipMalloc((void**)&fRdn_d       ,sizeof(bool)*(cirInfo->fNum));
    // dictionary
    hipMalloc((void**)&partialDict_d      ,sizeof(char)*(cirInfo->fNum)*paraPatNum);
    // value
    hipMalloc((void**)&val_d       ,sizeof(PatValue)*(cirInfo->OrigateNum)*4*LS_bkNum);
    // for pdSim
    hipMalloc((void**)&fLvl_d       ,sizeof(unsigned int)*(cirInfo->fNum));

    hipMalloc((void**)&twoLvlval_d  ,sizeof(unsigned int)*2*(cirInfo->gatesPerLvl)*(paraPatNum+1)*arg->APD_bkNum*2*2*TFS_thGroup);       // two for val l_/h_ and two for pre/curr val
    hipMalloc((void**)&twoLvlfG_d   ,sizeof(unsigned int)*(cirInfo->gatesPerLvl)*arg->APD_bkNum*(1+paraPatNum)*TFS_thGroup);       // two for event List
    hipMalloc((void**)&EventList_d  ,sizeof(unsigned int)*(cirInfo->gatesPerLvl)*arg->APD_bkNum*(1+paraPatNum)*2*TFS_thGroup);     // two for event List
    hipMalloc((void**)&RmnfNum_d    ,sizeof(unsigned int)*2);
    hipMalloc((void**)&LBRmnfNum_d  ,sizeof(unsigned int));
    // fault Compaction
    hipMalloc((void**)&fSum_d       ,sizeof(unsigned int)*(cirInfo->fNum));
    hipMalloc((void**)&fMask_d       ,sizeof(unsigned int)*(cirInfo->fNum));
    hipMalloc((void**)&bSum_d        ,sizeof(unsigned int)*FC_bkNum);
    cout <<" [Correct]: Finish Malloc...\n";
    // Memcpy
    hipMemcpy(cirInfo_d       ,cirInfo     ,sizeof(CircuitInfo)        ,hipMemcpyHostToDevice);
    hipMemcpy(gDum2Ori_d      ,gDum2Ori     ,sizeof(unsigned int)*(cirInfo->DumgateNum)                        ,hipMemcpyHostToDevice);
    // Ori Circuit
    hipMemcpy(gTypeOri_d      ,gTypeOri       ,sizeof(unsigned int)*(cirInfo->OrigateNum)    ,hipMemcpyHostToDevice);
    hipMemcpy(gFiOri_d        ,gFiOri         ,sizeof(unsigned int)*(cirInfo->OrigateNum)*4  ,hipMemcpyHostToDevice);
    hipMemcpy(gStrOnLvlOri_d  ,gStrOnLvlOri ,sizeof(unsigned int)*(cirInfo->cirlvl + 1)                        ,hipMemcpyHostToDevice);
    // Dum Circuit
    hipMemcpy(gTypeDum_d      ,gTypeDum       ,sizeof(unsigned int)*(cirInfo->DumgateNum)    ,hipMemcpyHostToDevice);
    hipMemcpy(gFiDum_d        ,gFiDum         ,sizeof(unsigned int)*(cirInfo->DumgateNum)*4  ,hipMemcpyHostToDevice);
    hipMemcpy(gStrOnLvlDum_d  ,gStrOnLvlDum   ,sizeof(unsigned int)*(cirInfo->cirlvl + 1)                        ,hipMemcpyHostToDevice);
    // fanout
    hipMemcpy(foArrayOri_d       ,foArrayOri      ,sizeof(unsigned int)*(foOffsetOri[cirInfo->OrigateNum])          ,hipMemcpyHostToDevice);
    hipMemcpy(foIdxArrayOri_d    ,foIdxArrayOri   ,sizeof(unsigned int)*(foOffsetOri[cirInfo->OrigateNum])          ,hipMemcpyHostToDevice);
    hipMemcpy(foOffsetOri_d      ,foOffsetOri     ,sizeof(unsigned int)*(cirInfo->OrigateNum+1)                        ,hipMemcpyHostToDevice);

    hipMemcpy(foArrayDum_d       ,foArrayDum      ,sizeof(unsigned int)*(foOffsetDum[cirInfo->DumgateNum])          ,hipMemcpyHostToDevice);
    hipMemcpy(foIdxArrayDum_d    ,foIdxArrayDum   ,sizeof(unsigned int)*(foOffsetDum[cirInfo->DumgateNum])          ,hipMemcpyHostToDevice);
    hipMemcpy(foOffsetDum_d      ,foOffsetDum     ,sizeof(unsigned int)*(cirInfo->DumgateNum+1)                        ,hipMemcpyHostToDevice);
    // delay
    hipMemcpy(dList_d         ,dList       ,sizeof(float)*(cirInfo->OrigateNum)*8         ,hipMemcpyHostToDevice);
    // fault
    hipMemcpy(fList_d         ,fList       ,sizeof(unsigned int)*(cirInfo->fNum)*3     ,hipMemcpyHostToDevice);
    // pdSim
    hipMemcpy(fLvl_d          ,fLvl         ,sizeof(unsigned int)*(cirInfo->fNum)                              ,hipMemcpyHostToDevice);
    cout <<" [Correct]: Finish Memcpy...\n";
    //}}}
    //******* Prepare for zero copy ******
    hipSetDeviceFlags(hipDeviceMapHost);
    // pattern
    int patDataSizeGPU = sizeof(PatValue)*(cirInfo->piNum+cirInfo->ppiNum)*2*LS_bkNum*2;
    int patDataSizeCPU = sizeof(PatValue)*(cirInfo->piNum+cirInfo->ppiNum)*2*((cirInfo->patNum -1)/paraPatNum + 1);
    // _z is temp for zero copy
    // _c is temp on CPU memory
    PatValue*       pat_t0_d;
    PatValue*       pat_t1_d;
    PatValue*       pat_t0_z;   // zero copy memory place
    PatValue*       pat_t1_z;
    PatValue*       pat_t0_c = (PatValue*)malloc(patDataSizeCPU); // memory on CPU for zero copy
    PatValue*       pat_t1_c = (PatValue*)malloc(patDataSizeCPU);
    // pattern redundant
    bool*           pRdn_d;  // to see if pattern is redundant

    hipMalloc((void**)&pat_t0_d    ,patDataSizeGPU);
    hipMalloc((void**)&pat_t1_d    ,patDataSizeGPU);
    hipMalloc((void**)&pRdn_d      ,sizeof(bool)*paraPatNum);
    hipHostAlloc(&pat_t0_c,patDataSizeCPU,hipHostMallocMapped);
    hipHostAlloc(&pat_t1_c,patDataSizeCPU,hipHostMallocMapped);
    hipHostGetDevicePointer(&pat_t0_z,pat_t0_c,idev);
    hipHostGetDevicePointer(&pat_t1_z,pat_t1_c,idev);
    cout <<" [Correct]: Finish Zero Copy...\n";
    // Remain fault array after fault drop
    unsigned int*   Rmnfault = (unsigned int*)malloc(sizeof(unsigned int)*cirInfo->fNum);
    unsigned int*   Rmnfault_d;
    unsigned int*   Rmnfault_z;
    cout<<" > Finish Rmnfault alloc\n";

    hipMalloc((void**)&Rmnfault_d   ,sizeof(unsigned int)*(cirInfo->fNum));
    hipHostAlloc(&Rmnfault,sizeof(int)*cirInfo->fNum,hipHostMallocMapped);
    hipHostGetDevicePointer(&Rmnfault_z,Rmnfault,idev);
    // LB Remain fault Array
    // This array record faults that is remained after DSMLB criteria
    // Check partial dictionary of these faults to get SFD
    unsigned int* LBRmnfault_d;
    hipMalloc((void**)&LBRmnfault_d   ,sizeof(unsigned int)*(cirInfo->fNum));
    // simple fault dictionary
    //**************************************
    int*            SFD = (int*)malloc(sizeof(int)*(cirInfo->fNum));        // simple fault dictionary
    int*            SFD_d;
    int*            SFD_z;
    hipMalloc((void**)&SFD_d,sizeof(int)*cirInfo->fNum);
    hipHostAlloc(&SFD,sizeof(int)*cirInfo->fNum,hipHostMallocMapped);
    hipHostGetDevicePointer(&SFD_z,SFD,idev);
    cout<<" > Finish SFD alloc\n";
    //************************************
    //******** Texture Momory *************
    hipBindTexture(0,gDum2Ori_t,gDum2Ori_d);
    hipBindTexture(0,cirInfo_t,cirInfo_d);
    //*************************************
    // ******* Report Memory Usage ********
    size_t free_byte;
    size_t total_byte;
    if(hipMemGetInfo(&free_byte,&total_byte) != hipSuccess){
        printf(" [Error]: Memory Get Info Fail!!\n");
        fout.close();
        return;
    }
    cout<<" ------------------------------------------\n";
    cout<<" | GPU memory free = "<<setw(12)<<(float)(free_byte)/1024.0/1024.0<<" MB      |\n";
    cout<<" | GPU memory used = "<<setw(12)<<(float)(total_byte - free_byte)/1024.0/1024.0<<" MB      |\n";
    cout<<" ------------------------------------------\n";
    fout<<" ------------------------------------------\n";
    fout<<" | GPU memory free = "<<setw(12)<<(float)(free_byte)/1024.0/1024.0<<" MB      |\n";
    fout<<" | GPU memory used = "<<setw(12)<<(float)(total_byte - free_byte)/1024.0/1024.0<<" MB      |\n";
    fout<<" ------------------------------------------\n";

    cout<<" | Mem Setup Time  = "<<setw(12)<< cutGetTimerValue(timer) <<" ms      |\n";
    cout<<" ------------------------------------------\n";

    fout<<" | Mem Setup Time  = "<<setw(12)<< cutGetTimerValue(timer) <<" ms      |\n";
    fout<<" ------------------------------------------\n";


    //********** Static Bound Analysis *************
    staticBoundAnalysis<<<SBA_bkNum,SBA_thNum,sizeof(float)*SBA_thNum>>>(
                          gTypeOri_d,dList_d,foArrayOri_d,foIdxArrayOri_d,foOffsetOri_d,
                          PT_UBLB_d,Ttc_d,gFiOri_d,gStrOnLvlOri_d,ATUB_d);
    hipDeviceSynchronize();
    hipFree(foArrayOri_d);
    hipFree(foIdxArrayOri_d);
    hipFree(foOffsetOri_d);
    // *********** Output Longest path *************
    float* Ttc = new float;
    hipMemcpy(Ttc        ,Ttc_d       ,sizeof(float)   ,hipMemcpyDeviceToHost);
    cout<<" | Longest Path    = "<<setw(12)<< Ttc[0] <<" ns      |\n";
    cout<<" | GatesPerLvl     = "<<setw(12)<< cirInfo->gatesPerLvl <<"         |\n";
    cout<<" ------------------------------------------\n";
    fout<<" | Longest Path    = "<<setw(12)<< Ttc[0] <<" ns      |\n";
    fout<<" | GatesPerLvl     = "<<setw(12)<< cirInfo->gatesPerLvl <<"         |\n";
    fout<<" ------------------------------------------\n";
    bool finishComp = false;    // to see if FaultSimulation step is done
    // Initialize pattern list
    // patList is used to save pat Id, on_check and if it is redundant
    vector<Pat*> patList;
    for(unsigned int i = 0 ; i < cirInfo->patNum; ++i){
        Pat* pat = new Pat(i,0);
        patList.push_back(pat);
    }
    int iterNum = 0;
    // Initialize Reain fault List & simple fault dictionary
    vector<unsigned int>    RmnfaultList;
    vector<unsigned int>    faultList;
    vector<Pat*>             essPat; // pattern for essential faults
    // Initialize fault List
    for(unsigned int i = 0 ; i < cirInfo->fNum; ++i){
        faultList.push_back(i);
        essPat.push_back(0);
    }
    // Pattern Selection algorithm
    unsigned int oldPatNum = patList.size();
    unsigned int DSMfNum = 0;
    while(!finishComp){
        // Start a new iteration
        cout<<" ==========================================================\n";
        cout<<" |              Compact Iteration "<<setw(3)<<iterNum<<"                     |\n";
        cout<<" ----------------------------------------------------------\n";
        fout<<" ==========================================================\n";
        fout<<" |              Compact Iteration "<<setw(3)<<iterNum<<"                     |\n";
        fout<<" ----------------------------------------------------------\n";
        int patDataSize = sizeof(PatValue)*(cirInfo->piNum+cirInfo->ppiNum)*2*((patList.size()-1)/paraPatNum + 1);
        // copy first pattern
        hipMemcpy(pat_t0_d         ,pat_t0       ,sizeof(PatValue)*(cirInfo->piNum+cirInfo->ppiNum)*2         ,hipMemcpyHostToDevice);
        hipMemcpy(pat_t1_d         ,pat_t1       ,sizeof(PatValue)*(cirInfo->piNum+cirInfo->ppiNum)*2         ,hipMemcpyHostToDevice);
        // zero copy pattern
        memcpy(pat_t0_c,pat_t0,patDataSize);
        memcpy(pat_t1_c,pat_t1,patDataSize);
        // copy remain fault List
        RmnfaultList.clear();
        RmnfaultList = faultList;

        // Reset SFD every iteration
        for(unsigned int i = 0 ; i < faultList.size(); ++i){
            SFD[faultList[i]] = -4;
            essPat[faultList[i]] = 0;
        }
        // Reset patList one_check
        for(unsigned int i = 0 ; i < patList.size(); ++i)
            patList[i]->one_check = 0;
        cout<<" | Pattern Size :"<<patList.size() <<" <--- OldPattern Size: "<<oldPatNum<<endl;
        fout<<" | Pattern Size :"<<patList.size() <<" <--- OldPattern Size: "<<oldPatNum<<endl;
        cout<<" ----------------------------------------------------------\n";
        fout<<" ----------------------------------------------------------\n";
        cout<<" | Pat. Loop |         Remain Fault Size                  |\n";
        fout<<" | Pat. Loop |         Remain Fault Size                  |\n";
        cout<<" ----------------------------------------------------------\n";
        fout<<" ----------------------------------------------------------\n";

        oldPatNum = patList.size();
        // get m pattern in every pattern Loop
        for(int patLoop = 0; patLoop < (patList.size()-1)/paraPatNum + 1; patLoop+=LS_bkNum){
            // Set zero copy Remain fault
            memcpy(Rmnfault,&RmnfaultList[0],sizeof(unsigned int)*RmnfaultList.size());
            FC_bkNum = (RmnfaultList.size() - 1)/(FC_thNum*2) + 1; // Fault Compaction block Number
            cout<<" |"<<setw(10)<<patLoop<< " | ";
            cout<<setw(30)<<RmnfaultList.size()<<"             |"<<endl;
            fout<<" |"<<setw(10)<<patLoop<< " | ";
            fout<<setw(30)<<RmnfaultList.size()<<"             |"<<endl;
            if(RmnfaultList.size() == 0){ // all the faults have been detected twice
                break;
            }
            // Logic Simulation
            logicSim<<<LS_bkNum,LS_thNum>>>(gTypeOri_d,gFiOri_d,gStrOnLvlOri_d,pat_t0_d,pat_t1_d,pat_t0_z,pat_t1_z,val_d,
                                            Rmnfault_d,Rmnfault_z,RmnfNum_d,RmnfaultList.size(),patList.size(),patLoop);
            // Output logic simulation result
            //CheckLogicVal(val_d,patLoop,patList.size(),cirInfo);

            // Dynamic Bound Analysis
            for(int lvl = 0 ; lvl < cirInfo->cirlvl;++lvl){
                // perform level by level
                dynamicBoundAnalysis<<<DBA_bkNum,DBA_thNum>>>(gTypeOri_d,gFiOri_d,at_d,gStrOnLvlOri_d,val_d,dList_d,lvl);
                hipDeviceSynchronize();
            }
            //  Check dynamic Cal Result
            //CheckDynamicAT(val_d,at_d,patLoop,cirInfo);

            // evaluate DSMLB criteria (Use PDUB)
            evalLBCriteria<<<EVB_bkNum,EVB_thNum>>>(gTypeOri_d,gFiOri_d,at_d,ATUB_d,PT_UBLB_d,Ttc_d,dList_d,
                                                     gStrOnLvlOri_d,val_d,fList_d,partialDict_d,Rmnfault_d,RmnfNum_d,fRdn_d,iterNum,arg->delta);
            // SFDAnalysis prepare for fault drop and Initialize SFD and redundant pattern
            SFDAnalysis<<<SFD_bkNum,SFD_thNum>>>(partialDict_d,SFD_d,SFD_z,fMask_d,Rmnfault_d,RmnfNum_d,pRdn_d,patList.size(),patLoop,iterNum,0);
            // Perform fault dropping, left fault needs untimed fault Sim
            fCompact1<<<FC_bkNum,FC_thNum>>>(fMask_d,fSum_d,bSum_d,RmnfNum_d);
            fCompact2<<<1,FC_thNum>>>(fSum_d,bSum_d,FC_bkNum);
            uniformAdd<<<FC_bkNum,FC_thNum>>>(fMask_d,fSum_d,bSum_d,Rmnfault_d,RmnfNum_d,LBRmnfault_d,LBRmnfNum_d,true);

            // Dynamic Share Memory malloc
            unsigned int sharedSize = sizeof(PatValue)*(arg->UFS_thNum*9) +
                                      sizeof(unsigned int)*(2+arg->UFS_thNum*2);
            untimedFaultSim<<<arg->APD_bkNum*(paraPatNum+1),arg->UFS_thNum,sharedSize>>>(gTypeDum_d,gFiDum_d,foArrayDum_d,foIdxArrayDum_d,foOffsetDum_d,
                                                                   gStrOnLvlDum_d,val_d,fList_d,partialDict_d,Rmnfault_d,RmnfNum_d,
                                                                   twoLvlval_d,twoLvlfG_d,EventList_d,fLvl_d);
            hipDeviceSynchronize();
            // evaluate DSMUB criteria (Use PDLB)
            evalUBCriteria<<<EVB_bkNum,EVB_thNum>>>(gTypeOri_d,gFiOri_d,at_d,ATUB_d,PT_UBLB_d,Ttc_d,dList_d,
                                                    gStrOnLvlOri_d,fList_d,partialDict_d,Rmnfault_d,RmnfNum_d,arg->delta);
            hipDeviceSynchronize();

            SFDAnalysis<<<SFD_bkNum,SFD_thNum>>>(partialDict_d,SFD_d,SFD_z,fMask_d,Rmnfault_d,RmnfNum_d,pRdn_d,patList.size(),patLoop,iterNum,1);
            if(iterNum > 0){
                SFDRdnPatBuild<<<SFD_bkNum,SFD_thNum>>>(partialDict_d,SFD_d,SFD_z,Rmnfault_d,RmnfNum_d,pRdn_d,patList.size(),fMask_d,patLoop,1);
            }
            // Fault dropping again, left fault that needs actual path delay
            fCompact1<<<FC_bkNum,FC_thNum>>>(fMask_d,fSum_d,bSum_d,RmnfNum_d);
            fCompact2<<<1,FC_thNum>>>(fSum_d,bSum_d,FC_bkNum);
            uniformAdd<<<FC_bkNum,FC_thNum>>>(fMask_d,fSum_d,bSum_d,Rmnfault_d,RmnfNum_d,LBRmnfault_d,LBRmnfNum_d,false);
            hipDeviceSynchronize();
            // ********* Check SFD & Stream Compaction *************
            //PrintSFD(partialDict_d,SFD,SFD_d,RmnfaultList,patLoop,patList.size(),cirInfo);
            //CheckFComp(SFD,partialDict_d,Rmnfault_d,RmnfNum_d,RmnfaultList,patLoop,patList.size(),cirInfo,1);
            // ***********************************************

            // Dynamic Shared Memory malloc
            sharedSize = sizeof(PatValue)*(arg->APD_thNum*4*2) +
                         sizeof(char)*(4*paraPatNum) +
                         sizeof(unsigned int)*(3+(paraPatNum+2)*arg->APD_thNum);
            actualPathDelayCal<<<arg->APD_bkNum,arg->APD_thNum,sharedSize>>>(gTypeDum_d,gFiDum_d,foArrayDum_d,foIdxArrayDum_d,foOffsetDum_d,at_d,Ttc_d,dList_d,
                                                                           gStrOnLvlDum_d,val_d,fList_d,Rmnfault_d,RmnfNum_d,partialDict_d,twoLvlval_d,
                                                                           (float*)&twoLvlval_d[(sizeof(PatValue)/sizeof(unsigned int))*(cirInfo->gatesPerLvl)*arg->APD_bkNum*2*2*TFS_thGroup],
                                                                           twoLvlfG_d,EventList_d,fLvl_d,ATUB_d,PT_UBLB_d,arg->delta);
            if(iterNum > 0){
                SFDAnalysis<<<SFD_bkNum,SFD_thNum>>>(partialDict_d,SFD_d,SFD_z,fMask_d,LBRmnfault_d,LBRmnfNum_d,pRdn_d,patList.size(),patLoop,iterNum,2);
                hipDeviceSynchronize();
                SFDRdnPatBuild<<<SFD_bkNum,SFD_thNum>>>(partialDict_d,SFD_d,SFD_z,LBRmnfault_d,LBRmnfNum_d,pRdn_d,patList.size(),fMask_d,patLoop,2);
                SetRedundantPat(pRdn_d,patList,patLoop);
            }
            else{ // first iteration, don't have redundant pattern ID
                SFDAnalysis<<<SFD_bkNum,SFD_thNum>>>(partialDict_d,SFD_d,SFD_z,fMask_d,Rmnfault_d,RmnfNum_d,pRdn_d,patList.size(),patLoop,iterNum,2);
                hipDeviceSynchronize();
            }
            OneCheckCal(SFD,patList,RmnfaultList,essPat,patLoop);
            // End of pattern Loop
        }
        // All the pattern loop has been Sim or All the fault has been detected twice
        // Identify redundant fault
        CompactRedundantFault(SFD,fRdn_d,faultList,iterNum,DSMfNum);
        if(arg->DSM_only == false){
            // Delete Redundant Pattern & Sort Pattern
            cout<<" ----------------------------------------------------------\n";
            fout<<" ----------------------------------------------------------\n";
            SortNCompactPattern(patList);
        }
        hipDeviceSynchronize();
        //(cutStopTimer(timer));
        cout<<" ----------------------------------------------------------\n";
        cout<<" |              Compact Iteration "<<setw(3)<<iterNum<<"                     |\n";
        cout<<" | Total Cal Time So far = "<<setw(15)<<(cutGetTimerValue(timer))/1000<<" (s)            |\n";
        cout<<" | Total Cal Patt So far = "<<setw(15)<<patList.size()<<"                |\n";
        cout<<" ----------------------------------------------------------\n";
        fout<<" ----------------------------------------------------------\n";
        fout<<" |              Compact Iteration "<<setw(3)<<iterNum<<"                     |\n";
        fout<<" | Total Cal Time So far = "<<setw(15)<<(cutGetTimerValue(timer))/1000<<" (s)            |\n";
        fout<<" | Total Cal Patt So far = "<<setw(15)<<patList.size()<<"                |\n";
        fout<<" ----------------------------------------------------------\n";
        iterNum++;
        if(patList[patList.size()-1]->one_check > 0 || arg->DSM_only == true){ // no redundant pattern left
            break;
        }
    }
    if(arg->DSM_only == false)
        DumpPattern(patList);
    cout << " ==========================================================" << endl;
    cout << " =                  Finish Cuda Sim                       =" << endl;
    cout << " ==========================================================" << endl;
    fout << " ==========================================================" << endl;
    fout << " =                  Finish Cuda Sim                       =" << endl;
    fout << " ==========================================================" << endl;
    cout<<" | Circuit: "<<pCir->getModRoot()->getName()<<endl;;
    fout<<" | Circuit: "<<pCir->getModRoot()->getName()<<endl;;
    cout<<" | Total Fault: "<<cirInfo->fNum<<endl;
    fout<<" | Total Fault: "<<cirInfo->fNum<<endl;
    cout<<" | PATTERN Size: "<<cirInfo->patNum<<"  --> Test Group Size: "<<(cirInfo->patNum-1)/paraPatNum + 1<<endl;
    fout<<" | PATTERN Size: "<<cirInfo->patNum<<"  --> Test Group Size: "<<(cirInfo->patNum-1)/paraPatNum + 1<<endl;
    cout<<" | PATTERN Selected Size: "<<patList.size()<<endl;
    fout<<" | PATTERN Selected Size: "<<patList.size()<<endl;
    cout<<" | Circuit input: "<<cirInfo->piNum + cirInfo->ppiNum<<endl;
    fout<<" | Circuit input: "<<cirInfo->piNum + cirInfo->ppiNum<<endl;
    cout<<" | Circuit Size: "<<cirInfo->OrigateNum<<endl;
    fout<<" | Circuit Size: "<<cirInfo->OrigateNum<<endl;
    cout<<" | Circuit Level: "<<cirInfo->cirlvl<<endl;
    fout<<" | Circuit Level: "<<cirInfo->cirlvl<<endl;
    cout<<" | Circuit Max Gate in Level: "<<cirInfo->gatesPerLvl<<endl;
    fout<<" | Circuit Max Gate in Level: "<<cirInfo->gatesPerLvl<<endl;
    cout<<" | Compact Iteration: "<<iterNum<<endl;
    fout<<" | Compact Iteration: "<<iterNum<<endl;
    cout<<" | DSM Coverage: "<<(float)DSMfNum/cirInfo->fNum*100<<" %"<<endl;
    fout<<" | DSM Coverage: "<<(float)DSMfNum/cirInfo->fNum*100<<" %"<<endl;
    fout.close();
    (cutDeleteTimer(timer));
    //{{{ hipFree
    hipUnbindTexture(cirInfo_t);
    hipUnbindTexture(gDum2Ori_t);
    hipFree(cirInfo_d);
    hipFree(gDum2Ori_d);
    // Ori Circuit
    hipFree(gTypeOri_d);
    hipFree(gFiOri_d);
    hipFree(gStrOnLvlOri_d);
    // Dum Circuit
    hipFree(gTypeDum_d);
    hipFree(gFiDum_d);
    hipFree(gStrOnLvlDum_d);
    // Fanout
    hipFree(foArrayDum_d);
    hipFree(foIdxArrayDum_d);
    hipFree(foOffsetDum_d);
    // delay and patcudaFree(h delay
    hipFree(dList_d);
    hipFree(PT_UBLB_d);
    hipFree(ATUB_d);
    hipFree(Ttc_d);
    hipFree(at_d);
    // value
    hipFree(val_d);
    // dictionary
    hipFree(partialDict_d);
    // fault
    hipFree(fList_d);

    // for pdSim
    hipFree(fLvl_d);
    hipFree(twoLvlval_d);
    hipFree(twoLvlat_d);
    hipFree(twoLvlfG_d);
    //hipFree(Rmnfault_d);
    //}}}
}
///}}}
//{{{  __global__ void staticBoundAnalysis()
__global__ void staticBoundAnalysis(unsigned int* gTypeOri_d, float* dList_d,unsigned int* foArrayOri_d,unsigned int* foIdxArrayOri_d,unsigned int* foOffsetOri_d,float* PT_UBLB_d,float* Ttc_d,
                          unsigned int* gFiOri_d,unsigned int* gStrOnLvlOri_d,float* ATUB_d){
    extern __shared__ float longestPath_s[];
    int thId = threadIdx.x;
    int patSize;
    int loop;
    unsigned int gateId;
    if(blockIdx.x == 0){    // PTUB/PTLB Calculation
        //******* Initialize last two level ***********
        for(int i = tex1Dfetch(cirInfo_t,6)-1; i >= tex1Dfetch(cirInfo_t,6)-2; --i){
            unsigned int gStrCurrLvl = gStrOnLvlOri_d[i];
            unsigned int gStrNextLvl = gStrOnLvlOri_d[i+1];
            loop = (gStrNextLvl - gStrCurrLvl-1)/SBA_thNum + 1;
            for(int m = 0 ; m < loop; ++m){
                if(thId + m*SBA_thNum < gStrNextLvl - gStrCurrLvl ){
                    gateId = thId + m*SBA_thNum + gStrCurrLvl;
                    PT_UBLB_d[gateId*4 + 0] = 0.0;
                    PT_UBLB_d[gateId*4 + 1] = 0.0;
                    PT_UBLB_d[gateId*4 + 2] = 0.0;
                    PT_UBLB_d[gateId*4 + 3] = 0.0;
                }
            }
            __syncthreads();

        }
        // ************ calculate PT UB/LB *****************
        for(int i =tex1Dfetch(cirInfo_t,6)-3; i >= 0; --i){
            __syncthreads();
            unsigned int gStrCurrLvl = gStrOnLvlOri_d[i];
            unsigned int gStrNextLvl = gStrOnLvlOri_d[i+1];
            loop = (gStrNextLvl - gStrCurrLvl-1)/SBA_thNum + 1;
            for(int m = 0 ; m < loop; ++m){
                if(thId + m*SBA_thNum < gStrNextLvl - gStrCurrLvl ){
                    gateId = thId + m*SBA_thNum + gStrCurrLvl;
                    int foNum;
                    if(gateId+1 != tex1Dfetch(cirInfo_t,9))
                        foNum = foOffsetOri_d[gateId+1] -foOffsetOri_d[gateId];
                    else
                        foNum =tex1Dfetch(cirInfo_t,9) - foOffsetOri_d[gateId];
                    float maxRPT=0.0;
                    float minRPT=1000.0;
                    float maxFPT=0.0;
                    float minFPT=1000.0;
                    for(int j = 0 ; j < foNum; ++j){
                        int fogIdx =  foArrayOri_d[foOffsetOri_d[gateId]+j]; // fanout gate indexs
                        int fopIdx =  foIdxArrayOri_d[foOffsetOri_d[gateId]+j]; // pin number of  gateul's
                        if(isInv(gTypeOri_d[fogIdx])){
                            if(maxRPT < PT_UBLB_d[fogIdx*4 + 2] + dList_d[fogIdx*8 + 2*fopIdx +1])
                                maxRPT = PT_UBLB_d[fogIdx*4 + 2] + dList_d[fogIdx*8 + 2*fopIdx +1];
                            if(minRPT > PT_UBLB_d[fogIdx*4 + 3] + dList_d[fogIdx*8 + 2*fopIdx +1])
                                minRPT = PT_UBLB_d[fogIdx*4 + 3] + dList_d[fogIdx*8 + 2*fopIdx +1];
                            if(maxFPT < PT_UBLB_d[fogIdx*4 + 0] + dList_d[fogIdx*8 + 2*fopIdx +0])
                                maxFPT = PT_UBLB_d[fogIdx*4 + 0] + dList_d[fogIdx*8 + 2*fopIdx +0];
                            if(minFPT > PT_UBLB_d[fogIdx*4 + 1] + dList_d[fogIdx*8 + 2*fopIdx +0])
                                minFPT = PT_UBLB_d[fogIdx*4 + 1] + dList_d[fogIdx*8 + 2*fopIdx +0];
                        }
                        else{
                            if(maxRPT < PT_UBLB_d[fogIdx*4 + 0] + dList_d[fogIdx*8 + 2*fopIdx +0])
                                maxRPT = PT_UBLB_d[fogIdx*4 + 0] + dList_d[fogIdx*8 + 2*fopIdx +0];
                            if(minRPT > PT_UBLB_d[fogIdx*4 + 1] + dList_d[fogIdx*8 + 2*fopIdx +0])
                                minRPT = PT_UBLB_d[fogIdx*4 + 1] + dList_d[fogIdx*8 + 2*fopIdx +0];
                            if(maxFPT < PT_UBLB_d[fogIdx*4 + 2] + dList_d[fogIdx*8 + 2*fopIdx +1])
                                maxFPT = PT_UBLB_d[fogIdx*4 + 2] + dList_d[fogIdx*8 + 2*fopIdx +1];
                            if(minFPT > PT_UBLB_d[fogIdx*4 + 3] + dList_d[fogIdx*8 + 2*fopIdx +1])
                                minFPT = PT_UBLB_d[fogIdx*4 + 3] + dList_d[fogIdx*8 + 2*fopIdx +1];
                        }
                    }
                    PT_UBLB_d[gateId*4 + 0] = maxRPT;
                    PT_UBLB_d[gateId*4 + 1] = minRPT;
                    PT_UBLB_d[gateId*4 + 2] = maxFPT;
                    PT_UBLB_d[gateId*4 + 3] = minFPT;
                }
            }
        }
    }
    else{   // ATUB Calculation
        // ************** Eval AT ***********************
        loop = (gStrOnLvlOri_d[1] - 1)/SBA_thNum + 1;
        for(int j = 0 ; j < loop;j++){
            if(thId + j*SBA_thNum < gStrOnLvlOri_d[1] ){
                gateId = thId + j*SBA_thNum;
                ATUB_d[gateId*2 + 0] = dList_d[gateId*8 + 2*0 + 0];
                ATUB_d[gateId*2 + 1] = dList_d[gateId*8 + 2*0 + 1];
            }
        }
        float maxRAT;
        float maxFAT;
        float maxT;
        for(int i = 1 ; i < tex1Dfetch(cirInfo_t,6) - 1;++i){
            __syncthreads();
            unsigned int gStrCurrLvl = gStrOnLvlOri_d[i];
            unsigned int gStrNextLvl = gStrOnLvlOri_d[i+1];
            loop = (gStrNextLvl - gStrCurrLvl-1)/SBA_thNum + 1;
            for(int m = 0 ; m < loop; ++m){
                if(thId + m*SBA_thNum < gStrNextLvl - gStrCurrLvl  ){
                    gateId = thId + m*SBA_thNum + gStrCurrLvl;
                    char fiNum = getFiNum(gTypeOri_d[gateId]);
                    maxRAT = 0.0;
                    maxFAT = 0.0;
                    for(char j = 0 ; j < fiNum; ++j){
                        int figId = gFiOri_d[gateId*4 + j];
                        if(isInv(gTypeOri_d[gateId])){ // gate has invert
                            if(maxRAT < ATUB_d[figId*2 + 1] + dList_d[gateId*8 + j*2 + 0])
                                maxRAT = ATUB_d[figId*2 + 1] + dList_d[gateId*8 + j*2 + 0];
                            if(maxFAT < ATUB_d[figId*2 + 0] + dList_d[gateId*8 + j*2 + 1])
                                maxFAT = ATUB_d[figId*2 + 0] + dList_d[gateId*8 + j*2 + 1];
                        }
                        else{
                            if(maxRAT < ATUB_d[figId*2 + 0] + dList_d[gateId*8 + j*2 + 0])
                                maxRAT = ATUB_d[figId*2 + 0] + dList_d[gateId*8 + j*2 + 0];
                            if(maxFAT < ATUB_d[figId*2 + 1] + dList_d[gateId*8 + j*2 + 1])
                                maxFAT = ATUB_d[figId*2 + 1] + dList_d[gateId*8 + j*2 + 1];

                        }
                    }
                    ATUB_d[gateId*2 + 0] = maxRAT;
                    ATUB_d[gateId*2 + 1] = maxFAT;
                    maxT = (maxT > maxRAT ? maxT : maxRAT);
                    maxT = (maxT > maxFAT ? maxT : maxFAT);
                }
            }
        }
        // ************ find longest path *************
        longestPath_s[thId] = maxT;
        __syncthreads();
        for(unsigned int j = SBA_thNum/2 ; j >0; j>>= 1){
            if(thId < j){
                float t0 = longestPath_s[thId];
                float t1 = longestPath_s[thId+j];
                longestPath_s[thId] = t0 > t1 ? t0:t1;
            }
            __syncthreads();
        }
        // This will be transfered back to CPU
        Ttc_d[0] = longestPath_s[0];
    }
}
//}}}
//{{{ __global__ void logicSim()
__global__ void  logicSim(unsigned int* gTypeOri_d,unsigned int* gFiOri_d,unsigned int* gStrOnLvlOri_d,PatValue* pat_t0_d,PatValue* pat_t1_d,PatValue* pat_t0_z,PatValue* pat_t1_z,
                          PatValue* val_d,unsigned int* Rmnfault_d,unsigned int* Rmnfault_z,unsigned int* RmnfNum_d,unsigned int RmnfNum,unsigned int RmnpNum,int patLoop){
    unsigned int totPatNum = (RmnpNum - 1)/paraPatNum + 1;
    // Idle block just return
    if(patLoop + blockIdx.x >= totPatNum)
        return;
    // Initialize Rmnfault_d array
    int loop = (RmnfNum - 1)/(LS_bkNum*LS_thNum) + 1;
    for(int i = 0 ; i < loop ; ++i){
        if(i*LS_bkNum*LS_thNum + blockIdx.x*LS_thNum + threadIdx.x < RmnfNum){
            Rmnfault_d[i*LS_bkNum*LS_thNum + blockIdx.x*LS_thNum + threadIdx.x] = Rmnfault_z[i*LS_bkNum*SBA_thNum + blockIdx.x*LS_thNum + threadIdx.x];
        }
        if(i*LS_bkNum*LS_thNum + blockIdx.x*LS_thNum + threadIdx.x ==0){
            RmnfNum_d[0] = RmnfNum;
        }
    }
    __shared__ unsigned int gFiOri_s[LS_thNum*4];
    __shared__ unsigned int gTypeOri_s[LS_thNum];
    int thId = threadIdx.x;
    unsigned int gatesPerLvl = tex1Dfetch(cirInfo_t,7);
    // ****** zero copy m patterns **************
    int patSize = (tex1Dfetch(cirInfo_t,3) + tex1Dfetch(cirInfo_t,4))*2;
    loop = (patSize - 1)/LS_thNum + 1;
    if(patLoop + LS_bkNum + blockIdx.x < totPatNum){
        for(int i = 0 ; i < loop ; ++i){
            if(thId + i * LS_thNum < patSize){
                pat_t0_d[thId + i * LS_thNum + ((patLoop+1)%2)*patSize*LS_bkNum + blockIdx.x * patSize] = pat_t0_z[thId + i * LS_thNum + (patLoop + LS_bkNum + blockIdx.x) * patSize];
                pat_t1_d[thId + i * LS_thNum + ((patLoop+1)%2)*patSize*LS_bkNum + blockIdx.x * patSize] = pat_t1_z[thId + i * LS_thNum + (patLoop + LS_bkNum + blockIdx.x) * patSize];
            }
        }
    }
    // *******************************************
    loop = (gStrOnLvlOri_d[1] - 1)/LS_thNum + 1;
    unsigned int gateId;
    // ******* set t0 & t1 pattern on PI/PPI*********
    val_d = &val_d[tex1Dfetch(cirInfo_t,9)*4*blockIdx.x]; // val_d position for block
    for(int m = 0 ; m < loop; ++m){
        if(thId + m*LS_thNum < tex1Dfetch(cirInfo_t,3) + tex1Dfetch(cirInfo_t,4)){
            val_d[(thId + m*LS_thNum) * 4 + 0] = pat_t0_d[(patLoop%2)*patSize*LS_bkNum + blockIdx.x*patSize + (thId + m*LS_thNum) * 2 + 0];
            val_d[(thId + m*LS_thNum) * 4 + 1] = pat_t0_d[(patLoop%2)*patSize*LS_bkNum + blockIdx.x*patSize + (thId + m*LS_thNum) * 2 + 1];
            val_d[(thId + m*LS_thNum) * 4 + 2] = pat_t1_d[(patLoop%2)*patSize*LS_bkNum + blockIdx.x*patSize + (thId + m*LS_thNum) * 2 + 0];
            val_d[(thId + m*LS_thNum) * 4 + 3] = pat_t1_d[(patLoop%2)*patSize*LS_bkNum + blockIdx.x*patSize + (thId + m*LS_thNum) * 2 + 1];
        }
    }
    __syncthreads();
    // hold or capture of m patterns in time frame one is saved in the last index of pat_t1_d
    // that is, the h_ of last PPI in time frame one
    // each bit of hold_capture represent hold(0) or capture(1) of m pattern
    // this will affect how PPI perform evaluation
    PatValue hold_capture = val_d[(tex1Dfetch(cirInfo_t,3)+tex1Dfetch(cirInfo_t,4))*4-1];
    // ********** Eval t0 Val **************
    // from lvel 1 to last level
    for(int i = 1 ; i < tex1Dfetch(cirInfo_t,6); ++i){
        unsigned int currLvlStr = gStrOnLvlOri_d[i];
        unsigned int nextLvlStr = gStrOnLvlOri_d[i+1];
        unsigned int nGatesInLvl = nextLvlStr - currLvlStr;
        loop = (nGatesInLvl-1)/LS_thNum + 1;
        for(int m = 0; m < loop; ++m){
            if(m*LS_thNum  < nGatesInLvl){
                int startGateId = currLvlStr + m*LS_thNum;
                // ****** coalesing read gType & gFi to share memory ******
                int gateDataSize;
                if((nextLvlStr - startGateId) < LS_thNum)
                    gateDataSize = (nextLvlStr- startGateId)*4;
                else
                    gateDataSize = LS_thNum*4;
                for(int n = 0 ; n < (gateDataSize-1)/LS_thNum + 1; ++n){
                    if( n*LS_thNum + thId < gateDataSize )
                        gFiOri_s[thId+n*LS_thNum] = gFiOri_d[startGateId*4 + thId+n*LS_thNum ];
                }
                if(thId < gateDataSize/4){
                    gTypeOri_s[thId] = gTypeOri_d[startGateId + thId];
                }
            }
            __syncthreads();
            // ******** Eval time frame 0****************
            if(thId + m*LS_thNum < nGatesInLvl){
                gateId = currLvlStr +  m*LS_thNum + thId;
                evalGate(gateId,gTypeOri_s[thId],gFiOri_s[thId*4+0],gFiOri_s[thId*4+1],gFiOri_s[thId*4+2],gFiOri_s[thId*4+3],val_d,hold_capture,0); // eval time frame 0
            }
        }
        __syncthreads();
    }
    // ********** Eval t1 Val **************
    // use level 0 Eval to set PPI t1 pattern
    for(int i = 0 ; i < tex1Dfetch(cirInfo_t,6); ++i){
        __syncthreads();
        unsigned int currLvlStr = gStrOnLvlOri_d[i];
        unsigned int nextLvlStr = gStrOnLvlOri_d[i+1];
        unsigned int nGatesInLvl = nextLvlStr - currLvlStr;
        loop = (nGatesInLvl-1)/LS_thNum + 1;
        for(int m = 0; m < loop; ++m){
            if(m*LS_thNum < nGatesInLvl){
                int startGateId = currLvlStr + m*LS_thNum;
                // ****** coalesing read gType & gFi to share memory ******
                int gateDataSize;
                if((nextLvlStr - startGateId) < LS_thNum)
                    gateDataSize = (nextLvlStr - startGateId)*4;
                else
                    gateDataSize = LS_thNum*4;
                for(int n = 0 ; n < (gateDataSize-1)/LS_thNum + 1; ++n){
                    if( n*LS_thNum + thId < gateDataSize )
                        gFiOri_s[thId+n*LS_thNum] = gFiOri_d[startGateId*4 + thId+n*LS_thNum ];
                }
                if(thId < gateDataSize/4){
                    gTypeOri_s[thId] = gTypeOri_d[startGateId + thId];
                }
            }
            __syncthreads();
            // ******** Eval time frame 1 ****************
            if(thId + m*LS_thNum < nGatesInLvl){
                gateId = currLvlStr +  m*LS_thNum + thId;
                evalGate(gateId,gTypeOri_s[thId],gFiOri_s[thId*4+0],gFiOri_s[thId*4+1],gFiOri_s[thId*4+2],gFiOri_s[thId*4+3],val_d,hold_capture,1); // eval time frame 1
            }
        }
    }
}
//}}}
//{{{ __global__ void dynamicBoundAnalysis()
__global__ void dynamicBoundAnalysis(unsigned int* gTypeOri_d,unsigned int* gFiOri_d,float* at_d,unsigned int* gStrOnLvlOri_d,PatValue* val_d,float* dList_d,int currLvl){
    __shared__ PatValue   fival_s[DBA_thNum/paraPatNum*4*4];        // fanin val may have 4 fanin in maximum; 4 logic value= t0_l_ t0_h_ t1_l_ t1_h_
    __shared__ PatValue   currval_s[DBA_thNum/paraPatNum*4];        // fanin val may have 4 fanin in maximum
    __shared__ unsigned int gTypeOri_s[DBA_thNum/paraPatNum];
    __shared__ float fiat_s[(DBA_thNum/paraPatNum)*paraPatNum*4];   // fanin arrival time may have 4 fanin in maximum
    __shared__ float dList_s[DBA_thNum/paraPatNum*4*2];             // every gate may  have 4 fanin and each have rising/falling
    int thIdonG = threadIdx.x % paraPatNum;  // thread index on gate
    int gIdonB  = threadIdx.x / paraPatNum;  // gate Id on Block

    // DBA_thNum/paraPatNum = number of gates a block can handle
    int gNumInbk = (DBA_thNum/paraPatNum);      // Number of gates processed by a block
    int gOffset = gNumInbk*DBA_bkNum;           // Number of gates jump after current gate
    unsigned int gStrCurrLvl = gStrOnLvlOri_d[currLvl];
    unsigned int gStrNextLvl = gStrOnLvlOri_d[currLvl + 1];
    int bkLoop = (gStrNextLvl - gStrCurrLvl-1)/gOffset + 1; // block loop
    for(int i = 0 ; i < bkLoop;++i){
        __syncthreads();     // sync all threads
        if(blockIdx.x*gNumInbk + i*gOffset + gStrCurrLvl < gStrNextLvl){
            unsigned int startGateId = gStrCurrLvl + blockIdx.x*gNumInbk + i*gOffset;
            // ********** coalesing read curr gate Val, gType, Delay into share mem ***************
            // using all threads in a block
            int gateDataSize;
            if(gStrNextLvl - startGateId > gNumInbk)
                gateDataSize = gNumInbk;
            else
                gateDataSize = (gStrNextLvl-startGateId);

            if(threadIdx.x < gateDataSize){
                gTypeOri_s[threadIdx.x] = gTypeOri_d[startGateId + threadIdx.x];
            }
            if(threadIdx.x < gateDataSize*5 && threadIdx.x >= gateDataSize){
                currval_s[threadIdx.x-gateDataSize] = val_d[startGateId*4+threadIdx.x-gateDataSize];
            }
            if(threadIdx.x < gateDataSize*13 && threadIdx.x >= gateDataSize*5 ){
                dList_s[threadIdx.x-gateDataSize*5] = dList_d[startGateId*4*2+threadIdx.x-gateDataSize*5];
            }
            // ********** coalescing read Fi AT and Fi val of current gate ************
            // Every 32/64 threads will be reponsible for its gates
            unsigned int gateId = gStrCurrLvl + i*gOffset + blockIdx.x*gNumInbk + gIdonB;
            if(gateId < gStrNextLvl){
                for(int j = 0 ; j < 4; ++j){
                    unsigned int figateId = gFiOri_d[gateId*4 + j];
                    if(figateId != ~0){
                       if(currLvl == 0)
                            fiat_s[gIdonB*paraPatNum*4 + thIdonG*4 + j] = 0.0;
                        else
                            fiat_s[gIdonB*paraPatNum*4 + thIdonG*4 + j] = at_d[figateId*paraPatNum + thIdonG];
                        if(thIdonG < 4){
                            fival_s[gIdonB*4*4 + j*4 + thIdonG] = val_d[figateId*4 + thIdonG];
                        }
                    }
                }
                __syncthreads();

                // ********* Eval AT according to transition *********
                // trans is the transition of gate in claculation
                char trans; // 0 = Rise; 1 = Fall; 2 = static
                trans = getTrans(currval_s[gIdonB*4 + 0],currval_s[gIdonB*4 + 1],currval_s[gIdonB*4 + 2],currval_s[gIdonB*4 + 3],thIdonG);
                if(trans != 2){ // has transition
                    char fiNum = getFiNum(gTypeOri_s[gIdonB]);
                    bool isCtrl = getCtrl(gTypeOri_s[gIdonB],trans); // is trans ctrl value?
                    // trType is the transition we want to find in fanin gate
                    char trType = transType(gTypeOri_s[gIdonB],trans); // 0 = fin rising ; 1 = fin falling ; 2 = fin any transition
                    char fitr[4] = {3,3,3,3};
                    for(char fi = 0 ; fi < fiNum ; ++fi){
                        fitr[fi] = getTrans(fival_s[gIdonB*4*4 + fi*4 + 0],fival_s[gIdonB*4*4 + fi*4 + 1],fival_s[gIdonB*4*4 + fi*4 + 2],fival_s[gIdonB*4*4 + fi*4 + 3],thIdonG);
                    }
                    float gAT;
                    if(isCtrl){ // find ctrl value arrived fastest
                        gAT = 999999.9;
                        for(char fi = 0 ; fi < fiNum ; ++fi){
                            if((trType == fitr[fi] || trType == 2) && gAT > fiat_s[gIdonB*paraPatNum*4 + thIdonG*4 + fi] + dList_s[gIdonB*4*2 + fi*2 + trans] ){
                                gAT = fiat_s[gIdonB*paraPatNum*4 + thIdonG*4 + fi] + dList_s[gIdonB*4*2 + fi*2 + trans];
                            }
                        }
                    }
                    else{   // find non ctrl value arrived latest
                        gAT = 0.0;
                        for(char fi = 0 ; fi < fiNum ; ++fi){
                            if((trType == fitr[fi] || trType == 2) && gAT < fiat_s[gIdonB*paraPatNum*4 + thIdonG*4 + fi] + dList_s[gIdonB*4*2 + fi*2 + trans]){
                                gAT = fiat_s[gIdonB*paraPatNum*4 + thIdonG*4 + fi] + dList_s[gIdonB*4*2 + fi*2 + trans];
                            }
                        }
                    }
                    at_d[gateId*paraPatNum + thIdonG]  = gAT;
                }
                else{   // no transition
                    at_d[gateId*paraPatNum + thIdonG]  = -1.0;
                }

                //**********************************************
            }
        }
    }
}
//}}}
//{{{ __global__ void evalLBCriteria()
__global__ void evalLBCriteria(unsigned int* gTypeOri_d,unsigned int* gFiOri_d,float* at_d,float* ATUB_d,float* PT_UBLB_d,float* Ttc_d,float* dList_d,unsigned int* gStrOnLvlOri_d,
                              PatValue* val_d,unsigned int* fList_d,char* partialDict_d,unsigned int* Rmnfault_d,unsigned int* RmnfNum_d,bool* fRdn_d,unsigned int iterNum,float delta){
    // every thread response for eval a fault's criteria
    unsigned int RmnfNum = RmnfNum_d[0];
    int loop = (RmnfNum - 1) / (EVB_thNum*EVB_bkNum) + 1;
    __shared__ int fList_s[EVB_thNum*3];
    __shared__ PatValue fval_s[EVB_thNum*4];
    for(int i = 0 ; i < loop ; ++i){
        // sync to prevent data in shared mem was refleshed
        __syncthreads();
        // ************  coalesing fGate Data to shared *************
        unsigned int fId;
        for(int j = 0 ; j < 4 ;++j){
            // using 4 threads to move 1 gate's value
            if(i*EVB_thNum*EVB_bkNum + blockIdx.x*(EVB_thNum) + j*(EVB_thNum>>2) + (threadIdx.x>>2) < RmnfNum &&
                    j*(EVB_thNum>>2) + (threadIdx.x>>2) < EVB_thNum){
                fId = Rmnfault_d[i*EVB_thNum*EVB_bkNum + blockIdx.x*(EVB_thNum) + j*(EVB_thNum>>2) + (threadIdx.x>>2)];
                int fGate = tex1Dfetch(gDum2Ori_t,fList_d[fId*3 + 2]);
                int fLine = fList_d[fId*3 + 1] -1;
                int fType = fList_d[fId*3 + 0];
                if(((threadIdx.x)&0x03) == 0){
                    fList_s[(j*(EVB_thNum>>2) + (threadIdx.x>>2))*3+2] = fGate;
                    fList_s[(j*(EVB_thNum>>2) + (threadIdx.x>>2))*3+1] = fLine;
                    fList_s[(j*(EVB_thNum>>2) + (threadIdx.x>>2))*3+0] = fType;
                    //printf("j:%d blk:%d thId:%d idx:%d\n",j,blockIdx.x,threadIdx.x,(j*(EVB_thNum>>2) + (threadIdx.x>>2)));
                }
                if(fLine == -1){ // output fault
                    fval_s[(j*(EVB_thNum>>2) + (threadIdx.x>>2))*4 + (threadIdx.x&0x03)] = val_d[fGate*4 + (threadIdx.x&0x03)];
                }
                else{
                    fGate = gFiOri_d[fGate*4 + fLine];
                    fval_s[(j*(EVB_thNum>>2) + (threadIdx.x>>2))*4 + (threadIdx.x&0x03)] = val_d[fGate*4 + (threadIdx.x&0x03)];
                }
            }
        }
        __syncthreads();

        if(i*EVB_thNum*EVB_bkNum + blockIdx.x*EVB_thNum + threadIdx.x < RmnfNum){
            // every thread get one fault to evaluate
            fId = Rmnfault_d[i*EVB_thNum*EVB_bkNum + blockIdx.x*EVB_thNum + threadIdx.x];
            int fGate = fList_s[threadIdx.x*3 + 2];
            int fLine = fList_s[threadIdx.x*3 + 1];
            int fType = fList_s[threadIdx.x*3 + 0];
            float Ttc = Ttc_d[0]*1.1;
            if(fLine == -1){ // output fault
                // loop for m patterns
                for(int patIdx = 0 ; patIdx < paraPatNum; ++patIdx){
                    if(getTrans(fval_s[threadIdx.x*4 + 0],fval_s[threadIdx.x*4 + 1],fval_s[threadIdx.x*4 + 2],fval_s[threadIdx.x*4 + 3],patIdx) == fType ){ // fault active
                        if(iterNum > 0 && fRdn_d[fId]){
                            partialDict_d[fId*paraPatNum + patIdx] = 2; // possible TRF detected
                        }
                        else{
                            if((ATUB_d[fGate*2 + fType] - at_d[fGate*paraPatNum + patIdx])/
                                    (Ttc - at_d[fGate*paraPatNum + patIdx] - PT_UBLB_d[fGate*4 + fType*2]) < delta){
                                partialDict_d[fId*paraPatNum + patIdx] = 1; // possible to bo Q
                            }
                            else{
                                partialDict_d[fId*paraPatNum + patIdx] = 0; //  not possible to bo Q
                            }
                        }
                    }
                    else{
                        partialDict_d[fId*paraPatNum + patIdx] = 0;
                    }
                }
            }
            else{ // input fault
                int finGate = gFiOri_d[fGate*4 + fLine];
                unsigned int gType = gTypeOri_d[fGate];
                int inv;
                if((gType >= 10 && gType <= 13) ||
                        (gType >= 18 && gType <= 21) ||
                        (gType >= 25 && gType <= 27) ||
                        (gType == 29 )){
                    inv = 1;
                }
                else{
                    inv = 0;
                }
                // loop for m patterns
                for(int patIdx = 0 ; patIdx < paraPatNum; ++patIdx){
                    if(getTrans(fval_s[threadIdx.x*4 + 0],fval_s[threadIdx.x*4 + 1],fval_s[threadIdx.x*4 + 2],fval_s[threadIdx.x*4 + 3],patIdx) == fType){ // fault active
                        // dict: 0: not long enough; 1: maybe long enough; 2: long enough
                        if(iterNum != 0 && fRdn_d[fId]){   // fault has been drop in previous pattern group
                            partialDict_d[fId*paraPatNum + patIdx] = 2; // possible to be detected
                        }
                        else{
                            if((ATUB_d[finGate*2 + fType] - at_d[finGate*paraPatNum + patIdx])/
                                    (Ttc - at_d[finGate*paraPatNum + patIdx] - dList_d[fGate*8 + fLine*2 + (fType^inv) ] - PT_UBLB_d[fGate*4 + (fType^inv)*2])
                                    < delta){
                                partialDict_d[fId*paraPatNum + patIdx] = 1;
                            }
                            else{
                                partialDict_d[fId*paraPatNum + patIdx] = 0;
                            }
                        }
                    }
                    else{
                        partialDict_d[fId*paraPatNum + patIdx] = 0;
                    }

                }
            }
        }
    }
}
//}}}
//{{{ __global__ void fCompact1()
__global__ void fCompact1(unsigned int* fMask_d,unsigned int* fSum_d, unsigned int* bSum_d,unsigned int* RmnfNum_d){
    // Use Stream Compaction Algorithm to perform fault dropping
    // first level of compaction
    __shared__ unsigned int fSum_s[FC_thNum*2]; // fault sum in every block
    unsigned int fOffset = blockIdx.x*FC_thNum*2; // fault Offset
    unsigned int RmnfNum = RmnfNum_d[0];
    unsigned int eleNum = FC_thNum*2;   // number of element
    int offset = 1;
    char lstMsk = 0;
    for(int i = 0 ; i < 2; ++i){
        if(fOffset + i*FC_thNum + threadIdx.x < RmnfNum){
            fSum_s[i*FC_thNum + threadIdx.x] = fMask_d[fOffset + i*FC_thNum + threadIdx.x] != ~0 ? 1 : 0;
        }
        else{
            fSum_s[i*FC_thNum + threadIdx.x] = 0;
        }
    }
    __syncthreads();
    lstMsk = fSum_s[FC_thNum*2-1];
    for(int d = eleNum >> 1; d > 0; d >>= 1){
        __syncthreads();
        if(threadIdx.x < d){
            fSum_s[offset*(2*threadIdx.x+2)-1] += fSum_s[offset*(2*threadIdx.x+1)-1];
        }
        offset *= 2;
    }
    if(threadIdx.x == 0){
        fSum_s[eleNum - 1] = 0;
    }
    for(int d = 1; d < eleNum; d*=2){
        offset >>= 1;
        __syncthreads();
        if(threadIdx.x < d){
            unsigned int tmp = fSum_s[offset*(2*threadIdx.x+1)-1];
            fSum_s[offset*(2*threadIdx.x+1)-1] = fSum_s[offset*(2*threadIdx.x+2)-1];
            fSum_s[offset*(2*threadIdx.x+2)-1] += tmp;
        }
    }
    __syncthreads();
    if(threadIdx.x == 0){
        bSum_d[blockIdx.x] = fSum_s[eleNum - 1] + lstMsk;
        //printf("bSum_d[%d]=%d\n",blockIdx.x,bSum_d[blockIdx.x]);
    }
    for(int i = 0 ; i < 2; ++i){
        if(fOffset + i*FC_thNum + threadIdx.x < RmnfNum){
            fSum_d[fOffset + i*FC_thNum + threadIdx.x] = fSum_s[i*FC_thNum + threadIdx.x];
        }
    }
}
//}}}
//{{{ __global__ void fCompact2()
__global__ void fCompact2(unsigned int* fSum_d, unsigned int* bSum_d,unsigned int FC_bkNum){
    // Second level of compaction
    __shared__ unsigned int bSum_s[FC_thNum*2]; // block sum
    unsigned int eleNum = powf(2,ceilf(log2f(FC_bkNum)));
    int offset = 1;
    for(int i = 0 ; i < 2; ++i){
        if(i*FC_thNum + threadIdx.x < FC_bkNum){
            bSum_s[i*FC_thNum + threadIdx.x] = bSum_d[i*FC_thNum + threadIdx.x];
        }
        else
            bSum_s[i*FC_thNum + threadIdx.x] = 0;
    }
    __syncthreads();
    for(int d = eleNum >> 1; d > 0; d >>= 1){
        __syncthreads();
        if(threadIdx.x < d){
            bSum_s[offset*(2*threadIdx.x+2)-1] += bSum_s[offset*(2*threadIdx.x+1)-1];
        }
        offset *= 2;
    }
    if(threadIdx.x == 0)
        bSum_s[eleNum - 1] = 0;
    for(int d = 1; d < eleNum; d*=2){
        offset >>= 1;
        __syncthreads();
        if(threadIdx.x < d){
            unsigned int tmp = bSum_s[offset*(2*threadIdx.x+1)-1];
            bSum_s[offset*(2*threadIdx.x+1)-1] = bSum_s[offset*(2*threadIdx.x+2)-1];
            bSum_s[offset*(2*threadIdx.x+2)-1] += tmp;
        }
    }
    __syncthreads();
    for(int i = 0 ; i < 2; ++i){
        if(i*FC_thNum + threadIdx.x < FC_bkNum){
            bSum_d[i*FC_thNum + threadIdx.x] = bSum_s[i*FC_thNum + threadIdx.x];
        }
    }
}
//}}}
//{{{ __global__ void uniformAdd()
__global__ void uniformAdd(unsigned int* fMask_d,unsigned int* fSum_d, unsigned int* bSum_d,
                          unsigned int* Rmnfault_d,unsigned int* RmnfNum_d,unsigned int* LBRmnfault_d,unsigned int* LBRmnfNum_d,bool afterLB){
    __shared__ unsigned int fSum_s[FC_thNum*2];
    __shared__ unsigned int fMask_s[FC_thNum*2];
    unsigned int foffset = blockIdx.x*FC_thNum*2; // fault Offset
    unsigned int RmnfNum = RmnfNum_d[0];
    unsigned int accSum = bSum_d[blockIdx.x]; // accumulate Sum
    for(int i = 0 ; i < 2; ++i){
        if(foffset + i*FC_thNum + threadIdx.x < RmnfNum){
            fSum_s[i*FC_thNum + threadIdx.x] = fSum_d[foffset + i*FC_thNum + threadIdx.x] + accSum;
            fMask_s[i*FC_thNum + threadIdx.x] = fMask_d[foffset + i*FC_thNum + threadIdx.x];
            if(fMask_s[i*FC_thNum+threadIdx.x] != ~0){
                Rmnfault_d[fSum_s[i*FC_thNum + threadIdx.x]] = fMask_s[i*FC_thNum + threadIdx.x];
                if(afterLB)
                    LBRmnfault_d[fSum_s[i*FC_thNum+threadIdx.x]] = fMask_s[i*FC_thNum + threadIdx.x];
            }
            if(foffset + i*FC_thNum + threadIdx.x == RmnfNum - 1){
                RmnfNum_d[1] = fSum_s[i*FC_thNum + threadIdx.x] + (fMask_s[i*FC_thNum + threadIdx.x] != ~0 ? 1:0);
                if(afterLB)
                    LBRmnfNum_d[0] = fSum_s[i*FC_thNum + threadIdx.x] + (fMask_s[i*FC_thNum + threadIdx.x] != ~0 ? 1:0);
            }
        }
    }
    __syncthreads();

}
//}}}
//{{{ __golbal__ void untimedFaultSim()
// every thread block have multiple thread group, each thread group handle a fault
__global__ void untimedFaultSim(unsigned int* gTypeDum_d,unsigned int* gFiDum_d,unsigned int* foArrayDum_d,unsigned int* foIdxArrayDum_d,unsigned int* foOffsetDum_d,
                      unsigned int* gStrOnLvlDum_d,PatValue* val_d,unsigned int* fList_d,char* partialDict_d,unsigned int* Rmnfault_d,unsigned int* RmnfNum_d,
                      PatValue* twoLvlval_d,unsigned int* twoLvlfG_d,unsigned int* EventList_d,unsigned int* fLvl_d){
    extern __shared__ unsigned int sharedMem[];
    unsigned int* foStr_s     = sharedMem;  // save where fo should str & faulty gateId
    unsigned int* foSize_s    = &foStr_s[blockDim.x]; // save foSize & faulty pin of faulty gateId
    unsigned int* eventSize_s = &foSize_s[blockDim.x]; // number of event gate in Event List
    PatValue* detect_s        = (PatValue*)&eventSize_s[2];
    PatValue* fival_s         = &detect_s[blockDim.x];

    unsigned int RmnfNum = RmnfNum_d[1];
    // update New Remain fault Number
    if(blockIdx.x == 0 && threadIdx.x == 0)
        RmnfNum_d[0] = RmnfNum;

    int floop = (RmnfNum - 1)/gridDim.x*TFS_thGroup + 1;
    unsigned int cirLvl = tex1Dfetch(cirInfo_t,6);
    unsigned int gatesPerLvl = tex1Dfetch(cirInfo_t,7);
    unsigned int thIdonF   = threadIdx.x % blockDim.x;
    unsigned int fIdonB    = threadIdx.x / blockDim.x;
    unsigned int fNumInB   = TFS_thGroup;
    twoLvlval_d  = &twoLvlval_d[(blockIdx.x*fNumInB + fIdonB)*gatesPerLvl*2*2];
    twoLvlfG_d   =  &twoLvlfG_d[(blockIdx.x*fNumInB + fIdonB)*gatesPerLvl];
    EventList_d  = &EventList_d[(blockIdx.x*fNumInB + fIdonB)*gatesPerLvl*2];
    for(int i = 0 ; i < floop ; ++i){ // loop for pick fault
        __syncthreads();
        bool dataflag = false;            // represent which data array we current on
        if(i*gridDim.x*fNumInB + blockIdx.x*fNumInB + fIdonB < RmnfNum){
            unsigned int fId = Rmnfault_d[i*gridDim.x*fNumInB + blockIdx.x*fNumInB + fIdonB];
            unsigned int fLvl = fLvl_d[fId];   // fault level
            unsigned int fType = fList_d[fId*3 + 0];
            int          fLine = fList_d[fId*3 + 1] - 1;
            unsigned int fGate = fList_d[fId*3 + 2];
            // fault injection
            unsigned gStrCurrLvl = gStrOnLvlDum_d[fLvl];
            unsigned gStrNextLvl = gStrOnLvlDum_d[fLvl+1];
            unsigned int loop = (gatesPerLvl - 1)/(blockDim.x) + 1;

            for(int j = 0 ; j < loop; ++j){
                if(thIdonF + j*blockDim.x < gatesPerLvl)
                    twoLvlfG_d[thIdonF + j*blockDim.x] = 0;
            }
            loop = (paraPatNum - 1)/ blockDim.x + 1;
            detect_s[fIdonB*blockDim.x + thIdonF] = 0;
            __syncthreads();
            // ********** Set twoLvl value for faulty gate *************
            if(thIdonF == 0){
                if(fLine == -1){ // output fault
                    if(fType == 0){ // rising fault
                        twoLvlval_d[dataflag*gatesPerLvl*2 + (fGate-gStrCurrLvl)*2 + 0] = ~0;
                        twoLvlval_d[dataflag*gatesPerLvl*2 + (fGate-gStrCurrLvl)*2 + 1] = 0;
                    }
                    else{   // falling fault
                        twoLvlval_d[dataflag*gatesPerLvl*2 + (fGate-gStrCurrLvl)*2 + 0] = 0;
                        twoLvlval_d[dataflag*gatesPerLvl*2 + (fGate-gStrCurrLvl)*2 + 1] = ~0;
                    }
                }
                else{ // intput fault
                    char fiNum = getFiNum(gTypeDum_d[fGate]);
                    for(char fi = 0 ; fi < fiNum ;++fi){
                        unsigned int figateId = tex1Dfetch(gDum2Ori_t,gFiDum_d[fGate*4+fi]);
                        if(fLine == fi){
                            if(fType == 0){ // slow to  rise fault
                                fival_s[fIdonB*blockDim.x*4*2 + fi*2+0] = ~0;
                                fival_s[fIdonB*blockDim.x*4*2 + fi*2+1] = 0;
                            }
                            else{ // slow to fall fault
                                fival_s[fIdonB*blockDim.x*4*2 + fi*2+0] = 0;
                                fival_s[fIdonB*blockDim.x*4*2 + fi*2+1] = ~0;
                            }
                        }
                        else{
                            fival_s[fIdonB*blockDim.x*4*2 + fi*2+0] = val_d[figateId*4 + 2];
                            fival_s[fIdonB*blockDim.x*4*2 + fi*2+1] = val_d[figateId*4 + 3];
                        }
                    }
                    evalGate(gTypeDum_d[fGate],fival_s[fIdonB*blockDim.x*4*2 + 0],fival_s[fIdonB*blockDim.x*4*2 + 1],
                                               fival_s[fIdonB*blockDim.x*4*2 + 2],fival_s[fIdonB*blockDim.x*4*2 + 3],
                                               fival_s[fIdonB*blockDim.x*4*2 + 4],fival_s[fIdonB*blockDim.x*4*2 + 5],
                                               fival_s[fIdonB*blockDim.x*4*2 + 6],fival_s[fIdonB*blockDim.x*4*2 + 7],
                                               &twoLvlval_d[dataflag*gatesPerLvl*2 + (fGate-gStrCurrLvl)*2]);
                }
                if((twoLvlval_d[dataflag*gatesPerLvl*2 + (fGate-gStrCurrLvl)*2 + 0] != val_d[tex1Dfetch(gDum2Ori_t,fGate)*4 + 2] ||
                            twoLvlval_d[dataflag*gatesPerLvl*2 + (fGate-gStrCurrLvl)*2 + 1] != val_d[tex1Dfetch(gDum2Ori_t,fGate)*4 + 3])){
                    // fault can propogate =>  set foGate effected pin
                    unsigned int foStr = foOffsetDum_d[fGate];
                    unsigned int foEnd = foOffsetDum_d[fGate+1];
                    for(int fo = foStr ; fo < foEnd; ++fo){
                        EventList_d[fo - foStr] = (foArrayDum_d[fo]<<4) | (0x01<<foIdxArrayDum_d[fo]);
                    }
                    eventSize_s[fIdonB] = foEnd - foStr;
                }
                else{
                    eventSize_s[fIdonB] = 0;
                }
                if(eventSize_s[fIdonB] == 0){   // fGate is PO or PPO
                    detect_s[fIdonB*blockDim.x + thIdonF] |= (twoLvlval_d[dataflag*gatesPerLvl*2 + (fGate-gStrCurrLvl)*2 + 0] ^
                            val_d[tex1Dfetch(gDum2Ori_t,fGate)*4 + 2]) |
                        (twoLvlval_d[dataflag*gatesPerLvl*2 + (fGate-gStrCurrLvl)*2 + 1] ^
                         val_d[tex1Dfetch(gDum2Ori_t,fGate)*4 + 3]);
                }
            }
            __syncthreads();
            // finish fault Injection

            for(int currlvl = fLvl + 1; currlvl < cirLvl; ++currlvl){
                // loop for Lvl on circuit
                __syncthreads();
                if(eventSize_s[fIdonB] == 0){
                    break;
                }
                dataflag = !dataflag;
                unsigned int gStrPrevLvl = gStrOnLvlDum_d[currlvl - 1];
                gStrCurrLvl = gStrOnLvlDum_d[currlvl];
                gStrNextLvl = gStrOnLvlDum_d[currlvl+1];
                // simulate val fiSFD
                loop = (eventSize_s[fIdonB] - 1)/blockDim.x + 1; // loop for event ;
                // ********** Set twoLvl value for gate *************
                for(int j = 0 ; j < loop; ++j){
                    if(j*blockDim.x + thIdonF < eventSize_s[fIdonB]){
                        // use 4-bits to save which pin is faulty pin
                        // use other bits to save faulty gate Id
                        unsigned int gateId = (EventList_d[(!dataflag)*gatesPerLvl + j*blockDim.x + thIdonF]>>4);
                        unsigned int fPin   = (EventList_d[(!dataflag)*gatesPerLvl + j*blockDim.x + thIdonF] & 0x0F);
                        char fiNum = getFiNum(gTypeDum_d[gateId]);
                        for(char fi = 0 ; fi < fiNum; ++fi){
                            unsigned int figateId = gFiDum_d[gateId*4 + fi];
                            // if fanin gate is faulty read value from twoLvlval of the block
                            // else read faulty free value from logic sim
                            if((fPin&(0x01<<fi)) != 0){
                                fival_s[fIdonB*blockDim.x*4*2 + thIdonF*8 + fi*2+0] = twoLvlval_d[(!dataflag)*gatesPerLvl*2 + (figateId-gStrPrevLvl)*2 + 0];
                                fival_s[fIdonB*blockDim.x*4*2 + thIdonF*8 + fi*2+1] = twoLvlval_d[(!dataflag)*gatesPerLvl*2 + (figateId-gStrPrevLvl)*2 + 1];
                            }
                            else{
                                fival_s[fIdonB*blockDim.x*4*2 + thIdonF*8 + fi*2+0] = val_d[tex1Dfetch(gDum2Ori_t,figateId)*4 + 2];
                                fival_s[fIdonB*blockDim.x*4*2 + thIdonF*8 + fi*2+1] = val_d[tex1Dfetch(gDum2Ori_t,figateId)*4 + 3];
                           }
                        }
                        evalGate(gTypeDum_d[gateId],fival_s[fIdonB*blockDim.x*4*2 + thIdonF*8 + 0],fival_s[fIdonB*blockDim.x*4*2 + thIdonF*8 + 1],
                                 fival_s[fIdonB*blockDim.x*4*2 + thIdonF*8 + 2],fival_s[fIdonB*blockDim.x*4*2 + thIdonF*8 + 3],
                                 fival_s[fIdonB*blockDim.x*4*2 + thIdonF*8 + 4],fival_s[fIdonB*blockDim.x*4*2 + thIdonF*8 + 5],
                                 fival_s[fIdonB*blockDim.x*4*2 + thIdonF*8 + 6],fival_s[fIdonB*blockDim.x*4*2 + thIdonF*8 + 7],
                                 &twoLvlval_d[dataflag*gatesPerLvl*2 + (gateId-gStrCurrLvl)*2]);
                        unsigned int foStr = foOffsetDum_d[gateId];
                        unsigned int foEnd = foOffsetDum_d[gateId+1];
                        if(foEnd - foStr != 0){
                            if((twoLvlval_d[dataflag*gatesPerLvl*2 + (gateId-gStrCurrLvl)*2 + 0] !=
                                val_d[tex1Dfetch(gDum2Ori_t,gateId)*4 + 2] ||
                                twoLvlval_d[dataflag*gatesPerLvl*2 + (gateId-gStrCurrLvl)*2 + 1] !=
                                val_d[tex1Dfetch(gDum2Ori_t,gateId)*4 + 3])){
                                // fault can propogate =>  set foGate effected pin
                                for(int fo = foStr ; fo < foEnd; ++fo){
                                    atomicAdd(&twoLvlfG_d[foArrayDum_d[fo]-gStrNextLvl],(0x01<<foIdxArrayDum_d[fo]));
                                }
                            }
                        }
                        else{
                            detect_s[fIdonB*blockDim.x + thIdonF] |= (twoLvlval_d[dataflag*gatesPerLvl*2 + (gateId-gStrCurrLvl)*2 + 0] ^
                                    val_d[tex1Dfetch(gDum2Ori_t,gateId)*4 + 2]) |
                                (twoLvlval_d[dataflag*gatesPerLvl*2 + (gateId-gStrCurrLvl)*2 + 1] ^
                                 val_d[tex1Dfetch(gDum2Ori_t,gateId)*4 + 3]);
                        }
                    }
                }
                __syncthreads();
                // add event List
                if(currlvl+1 < cirLvl){
                    // evaluate a set of gates
                    loop = (eventSize_s[fIdonB] - 1)/blockDim.x + 1;
                    unsigned int eventSize = 0;
                    for(int j = 0 ; j < loop; ++j){
                        __syncthreads();
                        unsigned int Idx = j*blockDim.x + thIdonF;
                        if(Idx < eventSize_s[fIdonB]){
                            unsigned int eventGate = EventList_d[(!dataflag)*gatesPerLvl + Idx]>>4;
                            foStr_s[fIdonB*blockDim.x + thIdonF]  = foOffsetDum_d[eventGate];
                            foSize_s[fIdonB*blockDim.x + thIdonF] = foOffsetDum_d[eventGate+1] - foOffsetDum_d[eventGate];
                        }
                        __syncthreads();
                        if(thIdonF == 0){
                            for(int m = 0 ; m < blockDim.x && j*blockDim.x + m < eventSize_s[fIdonB]; ++m){
                                for(int fo = 0; fo < foSize_s[fIdonB*blockDim.x + m]; ++fo){
                                    if((twoLvlfG_d[foArrayDum_d[foStr_s[fIdonB*blockDim.x + m]+fo]-gStrNextLvl]) != 0){
                                        EventList_d[dataflag*gatesPerLvl + eventSize] = (foArrayDum_d[foStr_s[fIdonB*blockDim.x + m]+fo]<<4) +
                                                                                      (twoLvlfG_d[foArrayDum_d[foStr_s[fIdonB*blockDim.x + m]+fo]-gStrNextLvl]);
                                        twoLvlfG_d[foArrayDum_d[foStr_s[fIdonB*blockDim.x + m]+fo]-gStrNextLvl] = 0;
                                        eventSize++;
                                    }
                                }
                            }
                        }
                        __syncthreads();
                    }
                    if(thIdonF == 0){
                        eventSize_s[fIdonB] = eventSize;
                    }
                }
            }
            // after simulation Check Result
            int offset = 1;
            for(int j = blockDim.x >> 1; j > 0 ; j>>=1){
                if(thIdonF < j){
                    detect_s[fIdonB*blockDim.x + offset*(2*thIdonF+2)-1] |= detect_s[fIdonB*blockDim.x + offset*(2*thIdonF+1)-1];
                }
                offset *= 2;
                __syncthreads();
            }
            loop = (paraPatNum -1)/blockDim.x + 1;
            for(int j = 0 ; j < loop ; ++j){
                if(j*blockDim.x + thIdonF < paraPatNum){
                    if((detect_s[fIdonB*blockDim.x + blockDim.x - 1] & ((PatValue)0x01<<(j*blockDim.x+thIdonF))) == 0){
                        partialDict_d[fId*paraPatNum + j*blockDim.x + thIdonF] = 0;
                    }
                }
            }
        }
    }
}
//}}}
//{{{ __global__ void evalUBCriteria()
__global__ void evalUBCriteria(unsigned int* gTypeOri_d,unsigned int* gFiOri_d,float* at_d,float* ATUB_d,float* PT_UBLB_d,float* Ttc_d,float* dList_d,
                         unsigned int* gStrOnLvlOri_d,unsigned int* fList_d,char* partialDict_d,unsigned int* Rmnfault_d,unsigned int* RmnfNum_d,float delta){
    __shared__ unsigned int fList_s[EVB_thNum*3];
    unsigned int RmnfNum = RmnfNum_d[0];
    int loop = (RmnfNum - 1) / (EVB_thNum*EVB_bkNum) + 1;
    unsigned int fId;
    unsigned int thIdonF = threadIdx.x%3;
    unsigned int fIdonB  = threadIdx.x/3;
    unsigned int fNumInB = EVB_thNum/3;
    for(int i = 0 ; i < loop ; ++i){
        __syncthreads();
        // coalesing read fList_d to fList_s share mem first
        int floop = (EVB_thNum -1)/(fNumInB) + 1;
        for(int j = 0 ; j < floop; ++j){
            if(i*EVB_bkNum*EVB_thNum + blockIdx.x*EVB_thNum + j*fNumInB + fIdonB < RmnfNum && j*fNumInB + fIdonB < EVB_thNum){
                fId = Rmnfault_d[i*EVB_bkNum*EVB_thNum + blockIdx.x*EVB_thNum + j*fNumInB + fIdonB];
                fList_s[(j*fNumInB + fIdonB)*3 + thIdonF] = fList_d[fId*3 + thIdonF];
            }
        }
        __syncthreads();
        if(i*EVB_thNum*EVB_bkNum + blockIdx.x*EVB_thNum + threadIdx.x < RmnfNum){
            fId = Rmnfault_d[i*EVB_thNum*EVB_bkNum + blockIdx.x*EVB_thNum + threadIdx.x];
            int fGate = tex1Dfetch(gDum2Ori_t,fList_s[threadIdx.x*3 + 2]);
            int fType = fList_s[threadIdx.x*3 + 0];
            float Ttc = Ttc_d[0]*1.1;
            if(fList_s[threadIdx.x*3 + 1] == 0){ // output fault
                for(int patIdx = 0 ; patIdx < paraPatNum; ++patIdx){
                    if(partialDict_d[fId*paraPatNum + patIdx] == 1 ){  // fault is detected
                        if((ATUB_d[fGate*2 + fType] + PT_UBLB_d[fGate*4 + fType*2] - at_d[fGate*paraPatNum + patIdx] - PT_UBLB_d[fGate*4 + fType*2 + 1])/
                                (Ttc - at_d[fGate*paraPatNum + patIdx] - PT_UBLB_d[fGate*4 + fType*2 + 1]) < delta){
                            partialDict_d[fId*paraPatNum + patIdx] = 2;
                        }
                        else{
                        }
                    }
                    else{
                    }
                }
            }
            else{ // input fault
                int fLine = fList_s[threadIdx.x*3 + 1] -1;
                int finGate = gFiOri_d[fGate*4 + fLine];
                unsigned int gType = gTypeOri_d[fGate];
                int inv;
                if((gType >= 10 && gType <= 13) ||
                        (gType >= 18 && gType <= 21) ||
                        (gType >= 25 && gType <= 27) ||
                        (gType == 29 )){
                    inv = 1;
                }
                else{
                    inv = 0;
                }
                for(int patIdx = 0 ; patIdx < paraPatNum; ++patIdx){
                    if(partialDict_d[fId*paraPatNum + patIdx] == 1){ // Check DSMUB (Use PDLB)
                        if((ATUB_d[finGate*2 + fType] + PT_UBLB_d[fGate*4 + (fType^inv)*2] -
                                    at_d[finGate*paraPatNum + patIdx] - PT_UBLB_d[fGate*4 + (fType^inv)*2 + 1] )/
                                (Ttc - at_d[finGate*paraPatNum + patIdx] - dList_d[fGate*8 + fLine*2 + (fType^inv) ] - PT_UBLB_d[fGate*4 + (fType^inv)*2 + 1])
                                < delta){
                            partialDict_d[fId*paraPatNum + patIdx] = 2;
                        }
                        else{
                        }
                    }
                    else{
                    }

                }
            }
        }
    }
}
//}}}
//{{{ __global__ void actualPathDelayCal()
__global__ void actualPathDelayCal(unsigned int* gTypeDum_d,unsigned int* gFiDum_d,unsigned int* foArrayDum_d,unsigned int* foIdxArrayDum_d,unsigned int* foOffsetDum_d,
                                   float* at_d,float* Ttc_d,float* dList_d,unsigned int* gStrOnLvlDum_d,PatValue* val_d,unsigned int* fList_d,unsigned int* Rmnfault_d,
                                   unsigned int* RmnfNum_d,char* partialDict_d,PatValue* twoLvlval_d,float* twoLvlat_d,unsigned int* twoLvlfG_d,unsigned int* EventList_d,
                                   unsigned int* fLvl_d,float* ATUB_d,float* PT_UBLB_d,float delta){
    // Dynamic shared Mememory malloc
    extern __shared__ PatValue      sharedPatValue[];
    extern __shared__ unsigned int  sharedInt[];
    extern __shared__ char          sharedChar[];
    PatValue*       fival_s     = sharedPatValue;
    float*          maxPD_s     = (float*)&sharedInt[blockDim.x*16]; // only for debug not used
    float*          PDs_s       = &maxPD_s[paraPatNum*blockDim.x]; // Structual longest path
    unsigned int*   eventSize_s = (unsigned int*)&PDs_s[1];
    unsigned int*   detPatNum_s = &eventSize_s[1];
    unsigned int*   foStr_s     = &detPatNum_s[1];
    unsigned int*   foSize_s    = &foStr_s[blockDim.x];
    char*           dict_s      = &sharedChar[blockDim.x*(64+paraPatNum*4+8)+12];
    char*           dictTmp_s   = &dict_s[paraPatNum];
    char*           dictScn_s   = &dictTmp_s[paraPatNum];
    bool*           patDone_s   = (bool*)&dictScn_s[paraPatNum];
    unsigned int RmnfNum = RmnfNum_d[1];
    // update remain fault number
    if(blockIdx.x == 0 && threadIdx.x == 0)
        RmnfNum_d[0] = RmnfNum;
    if(RmnfNum == 0)
        return;
    int floop = (RmnfNum - 1)/gridDim.x + 1;
    float Ttc = Ttc_d[0]*1.1;
    unsigned int cirLvl = tex1Dfetch(cirInfo_t,6);
    unsigned int gatesPerLvl = tex1Dfetch(cirInfo_t,7);
    twoLvlval_d  = &twoLvlval_d[blockIdx.x*gatesPerLvl*2*2];
    twoLvlat_d   = &twoLvlat_d[blockIdx.x*gatesPerLvl*paraPatNum*2];
    twoLvlfG_d   = &twoLvlfG_d[blockIdx.x*gatesPerLvl];
    EventList_d  = &EventList_d[blockIdx.x*gatesPerLvl*2];
    for(int i = 0 ; i < floop ; ++i){ // loop for pick fault
        __syncthreads();
        bool dataflag = false;            // represent which data arry we current on
        if(blockIdx.x + i*gridDim.x < RmnfNum){
            unsigned int fId = Rmnfault_d[blockIdx.x + i*gridDim.x];
            unsigned int fLvl = fLvl_d[fId];   // fault level
            unsigned int fType = fList_d[fId*3 + 0];
            int          fLine = fList_d[fId*3 + 1] - 1;
            unsigned int fGate = fList_d[fId*3 + 2];
            // fault injection
            unsigned gStrCurrLvl = gStrOnLvlDum_d[fLvl];
            unsigned gStrNextLvl = gStrOnLvlDum_d[fLvl+1];
            unsigned int loop = (gatesPerLvl - 1)/(blockDim.x) + 1;
            for(unsigned int j = 0 ; j < loop; ++j){
                if(threadIdx.x + j*blockDim.x < gatesPerLvl)
                    twoLvlfG_d[threadIdx.x + j*blockDim.x] = 0;
            }
            // **************** Compact partial dictionary *****************
            // Compact patId that may detect this fault into the front of the
            // array: dict_s  using stream compaction
            loop = (paraPatNum - 1)/blockDim.x + 1;
            for(unsigned int j = 0 ; j < loop ; ++j){
                if(j*blockDim.x + threadIdx.x < paraPatNum){
                    dictScn_s[j*blockDim.x + threadIdx.x] = partialDict_d[fId*paraPatNum + j*blockDim.x + threadIdx.x] == 1 ? 1 : 0;
                    dictTmp_s[j*blockDim.x + threadIdx.x] = dictScn_s[j*blockDim.x + threadIdx.x];
                }
            }
            __syncthreads();
            int offset = 1;
            for(int j = paraPatNum >> 1; j > 0 ; j>>=1){
                __syncthreads();
                if(threadIdx.x < j){
                    dictScn_s[offset*(2*threadIdx.x+2)-1] += dictScn_s[offset*(2*threadIdx.x+1)-1];
                }
                offset *= 2;
            }
            if(threadIdx.x == 0)
                dictScn_s[paraPatNum-1] = 0;
            for(int j = 1; j < paraPatNum; j*=2){
                offset >>= 1;
                __syncthreads();
                if(threadIdx.x < j){
                    char tmp  = dictScn_s[offset*(2*threadIdx.x+1)-1];
                    dictScn_s[offset*(2*threadIdx.x+1)-1] = dictScn_s[offset*(2*threadIdx.x+2)-1];
                    dictScn_s[offset*(2*threadIdx.x+2)-1] += tmp;
                }
            }
            __syncthreads();
            for(unsigned int j = 0 ; j < loop ; ++j){
                if(j*blockDim.x + threadIdx.x < paraPatNum){
                    if(dictTmp_s[j*blockDim.x + threadIdx.x] == 1)
                        dict_s[dictScn_s[j*blockDim.x + threadIdx.x]] = j*blockDim.x + threadIdx.x;
                    patDone_s[j*blockDim.x + threadIdx.x] = false;
                }
            }
            __syncthreads();
            // ********** End of Compact dictionary ****************

            // ********** Set Two Level Array value for faulty gate *************
            if(threadIdx.x == blockDim.x - 1){ // Use one thread to inject fault
                detPatNum_s[0] = dictScn_s[paraPatNum - 1] + dictTmp_s[paraPatNum - 1]; // number of pattern needs Actual Path delay Cal
                if(fLine == -1){ // output fault
                    if(fType == 0){ // rising fault
                        twoLvlval_d[dataflag*gatesPerLvl*2 + (fGate-gStrCurrLvl)*2 + 0] = ~0;
                        twoLvlval_d[dataflag*gatesPerLvl*2 + (fGate-gStrCurrLvl)*2 + 1] = 0;
                    }
                    else{   // falling fault
                        twoLvlval_d[dataflag*gatesPerLvl*2 + (fGate-gStrCurrLvl)*2 + 0] = 0;
                        twoLvlval_d[dataflag*gatesPerLvl*2 + (fGate-gStrCurrLvl)*2 + 1] = ~0;
                    }
                }
                else{ // intput fault
                    char fiNum = getFiNum(gTypeDum_d[fGate]);
                    for(char fi = 0 ; fi < fiNum ;++fi){
                        unsigned int figateId = tex1Dfetch(gDum2Ori_t,gFiDum_d[fGate*4+fi]);
                        if(fLine == fi){
                            if(fType == 0){ // slow to  rise fault
                                fival_s[fi*2+0] = ~0;
                                fival_s[fi*2+1] = 0;
                            }
                            else{ // slow to fall fault
                                fival_s[fi*2+0] = 0;
                                fival_s[fi*2+1] = ~0;
                            }
                        }
                        else{
                            fival_s[fi*2+0] = val_d[figateId*4 + 2];
                            fival_s[fi*2+1] = val_d[figateId*4 + 3];
                        }
                    }
                    evalGate(gTypeDum_d[fGate],fival_s[0],fival_s[1],fival_s[2],fival_s[3],
                             fival_s[4],fival_s[5],fival_s[6],fival_s[7],&twoLvlval_d[dataflag*gatesPerLvl*2 + (fGate-gStrCurrLvl)*2]);
                }
            }
            else if(threadIdx.x == 0){   //  Use one thread to Save Event List
                unsigned int foStr = foOffsetDum_d[fGate];
                unsigned int foEnd = foOffsetDum_d[fGate+1];
                for(int j = foStr ; j < foEnd; ++j){
                    // using last 4 bits to indicate faulty pin
                    EventList_d[j - foStr] = (foArrayDum_d[j]<<4) | (0x01<<foIdxArrayDum_d[j]);
                }
                eventSize_s[0] = foEnd - foStr;
            }
            __syncthreads();
            // *********** Set faulty Lvl Arrival Time ***************
            for(int j = 0 ; j < loop ; ++j){
                if(j*blockDim.x + threadIdx.x < detPatNum_s[0]){
                    char patIdx = dict_s[j*blockDim.x + threadIdx.x];
                    if(fLine != -1 ){ // input fault
                        unsigned int fifGate = tex1Dfetch(gDum2Ori_t,gFiDum_d[fGate*4+fLine]);
                        unsigned int gType = gTypeDum_d[fGate];
                        int inv;
                        if((gType >= 10 && gType <= 13) ||
                                (gType >= 18 && gType <= 21) ||
                                (gType >= 25 && gType <= 27) ||
                                (gType == 29 )){
                            inv = 1;
                        }
                        else{
                            inv = 0;
                        }
                        twoLvlat_d[dataflag*gatesPerLvl*paraPatNum + (fGate-gStrCurrLvl)*paraPatNum + patIdx] = at_d[fifGate*paraPatNum + patIdx] +
                                   dList_d[tex1Dfetch(gDum2Ori_t,fGate)*8 + fLine*2 + (fType^inv)];
                        PDs_s[0] = ATUB_d[fifGate*2 + fType]+ dList_d[tex1Dfetch(gDum2Ori_t,fGate)*8 + fLine*2 + (fType^inv)] +
                                          PT_UBLB_d[tex1Dfetch(gDum2Ori_t,fGate)*4 + (fType^inv)*2];
                    }
                    else{
                        twoLvlat_d[dataflag*gatesPerLvl*paraPatNum + (fGate-gStrCurrLvl)*paraPatNum + patIdx] =
                            at_d[tex1Dfetch(gDum2Ori_t,fGate)*paraPatNum + patIdx];
                        PDs_s[0] = ATUB_d[tex1Dfetch(gDum2Ori_t,fGate)*2 + fType] + PT_UBLB_d[tex1Dfetch(gDum2Ori_t,fGate)*4 + fType*2];
                    }
                }
            }
            __syncthreads();
            // *************** finish fault injection ******************

            float maxPD = 0.0;  // maximum path delay of each thread
            for(int currlvl = fLvl + 1; currlvl < cirLvl; ++currlvl){  // loop for Lvl on circuit
            //for(int currlvl = fLvl + 1; currlvl < fLvl+3; ++currlvl){  // loop for Lvl on circuit
                if(eventSize_s[0] == 0)
                    break;
                __syncthreads();
                dataflag = !dataflag; // switch between two array
                unsigned int gStrPrevLvl = gStrOnLvlDum_d[currlvl - 1];
                gStrCurrLvl = gStrOnLvlDum_d[currlvl];
                gStrNextLvl = gStrOnLvlDum_d[currlvl+1];
                // simulate event gate value
                loop = (eventSize_s[0] - 1)/(blockDim.x) + 1; // loop for event
                for(int j = 0 ; j < loop; ++j){
                    __syncthreads();
                    // ********** Set Two Lvl Array value for gate *************
                    if(j*blockDim.x + threadIdx.x < eventSize_s[0]){
                        unsigned int gateId = (EventList_d[(!dataflag)*gatesPerLvl + j*blockDim.x + threadIdx.x]>>4);
                        // Use 4-bits to represent faulty pin. ex: 0010 means second pi is faulty
                        unsigned int fPin   = (EventList_d[(!dataflag)*gatesPerLvl + j*blockDim.x + threadIdx.x] & 0x0F); // faulty pin
                        foStr_s[threadIdx.x] = gateId;
                        foSize_s[threadIdx.x] = fPin;
                        char fiNum = getFiNum(gTypeDum_d[gateId]);
                        for(char fi = 0 ; fi < fiNum; ++fi){
                            unsigned int figateId = gFiDum_d[gateId*4 + fi];
                            if((fPin&(0x01<<fi)) != 0){ // get fi value from last level array
                                fival_s[threadIdx.x*8 + fi*2+0] = twoLvlval_d[(!dataflag)*gatesPerLvl*2 + (figateId-gStrPrevLvl)*2 + 0];
                                fival_s[threadIdx.x*8 + fi*2+1] = twoLvlval_d[(!dataflag)*gatesPerLvl*2 + (figateId-gStrPrevLvl)*2 + 1];
                            }
                            else{ // get fi value from logic sim
                                fival_s[threadIdx.x*8 + fi*2+0] = val_d[tex1Dfetch(gDum2Ori_t,figateId)*4 + 2];
                                fival_s[threadIdx.x*8 + fi*2+1] = val_d[tex1Dfetch(gDum2Ori_t,figateId)*4 + 3];
                           }
                        }
                        evalGate(gTypeDum_d[gateId],fival_s[threadIdx.x*8 + 0],fival_s[threadIdx.x*8 + 1],fival_s[threadIdx.x*8 + 2],fival_s[threadIdx.x*8 + 3],
                                 fival_s[threadIdx.x*8 + 4],fival_s[threadIdx.x*8 + 5],fival_s[threadIdx.x*8 + 6],fival_s[threadIdx.x*8 + 7],
                                 &twoLvlval_d[dataflag*gatesPerLvl*2 + (gateId-gStrCurrLvl)*2]);
                        unsigned int foStr = foOffsetDum_d[gateId];
                        unsigned int foEnd = foOffsetDum_d[gateId+1];
                        if(foEnd - foStr != 0){
                            if((twoLvlval_d[dataflag*gatesPerLvl*2 + (gateId-gStrCurrLvl)*2 + 0] !=
                                val_d[tex1Dfetch(gDum2Ori_t,gateId)*4 + 2] ||
                                twoLvlval_d[dataflag*gatesPerLvl*2 + (gateId-gStrCurrLvl)*2 + 1] !=
                                val_d[tex1Dfetch(gDum2Ori_t,gateId)*4 + 3])){
                                // fault can propogate =>  set foGate effected pin
                                for(int fo = foStr ; fo < foEnd; ++fo){
                                    atomicAdd(&twoLvlfG_d[foArrayDum_d[fo]-gStrNextLvl],(0x01<<foIdxArrayDum_d[fo]));
                                }
                            }
                        }
                        // evaluate AT after every blockDim.x faulty gates have been evaluated
                        for(int k = 0 ; k < detPatNum_s[0]; ++k){
                            char patIdx = dict_s[k];
                            if(patDone_s[patIdx]){ // this pattern has met DSM criteria
                                continue;
                            }
                            char outV = getV(twoLvlval_d[dataflag*gatesPerLvl*2 + (gateId-gStrCurrLvl)*2 + 0],twoLvlval_d[dataflag*gatesPerLvl*2 + (gateId-gStrCurrLvl)*2 + 1],
                                             val_d[tex1Dfetch(gDum2Ori_t,gateId)*4 + 2],val_d[tex1Dfetch(gDum2Ori_t,gateId)*4 + 3],patIdx);// faulty value
                            if(outV < 2){  // D' = STF or D = STR faulty value can propogate
                                unsigned int fiNum = getFiNum(gTypeDum_d[gateId]);
                                float PDa;  // actual path delay
                                bool isCtrl = getCtrl(gTypeDum_d[gateId],outV);
                                char trType = transType(gTypeDum_d[gateId],outV); // 0 = fin D ; 1 = D' ; 2 = fin any transition
                                int transPin = -1;
                                // isCtrl is the same as previous
                                // ex: when AND gate output is D(STR) we want to find the latest D(STR) on it output
                                // => isCtrl = false (find latest input D) trType = D(0)
                                // trType is the input transition type we looking for
                                if(isCtrl){ // output  control value
                                    PDa = 1000.0;
                                    for(char fi = 0 ; fi < fiNum ; ++fi){
                                        unsigned int figateId = gFiDum_d[gateId*4 + fi]; // gateId start from 0
                                        float delay = gTypeDum_d[gateId] == 33 ? 0.0 : dList_d[tex1Dfetch(gDum2Ori_t,gateId)*4*2 + fi*2 + outV];
                                        char fitr;
                                        if((fPin&(0x01<<fi)) != 0){
                                            fitr = getV(twoLvlval_d[(!dataflag)*gatesPerLvl*2 + (figateId-gStrPrevLvl)*2 + 0],twoLvlval_d[(!dataflag)*gatesPerLvl*2 + (figateId-gStrPrevLvl)*2 + 1],
                                                    val_d[tex1Dfetch(gDum2Ori_t,figateId)*4 + 2],val_d[tex1Dfetch(gDum2Ori_t,figateId)*4 + 3],patIdx);
                                            if((trType == fitr || trType == 2) && PDa >=
                                                twoLvlat_d[(!dataflag)*gatesPerLvl*paraPatNum + (figateId-gStrPrevLvl)*paraPatNum + patIdx] + delay){
                                                PDa = twoLvlat_d[(!dataflag)*gatesPerLvl*paraPatNum + (figateId-gStrPrevLvl)*paraPatNum + patIdx] + delay;
                                                transPin = fi;
                                            }
                                        }
                                    }
                                }
                                else{
                                    PDa = -1000.0;
                                    for(char fi = 0 ; fi < fiNum ; ++fi){
                                        unsigned int figateId = gFiDum_d[gateId*4 + fi]; // gateId start from 0
                                        float delay = gTypeDum_d[gateId] == 33 ? 0.0 : dList_d[tex1Dfetch(gDum2Ori_t,gateId)*4*2 + fi*2 + outV] ;
                                        char fitr;
                                        if((fPin&(0x01<<fi)) != 0){
                                            fitr = getV(twoLvlval_d[(!dataflag)*gatesPerLvl*2 + (figateId-gStrPrevLvl)*2 + 0],twoLvlval_d[(!dataflag)*gatesPerLvl*2 + (figateId-gStrPrevLvl)*2 + 1],
                                                    val_d[tex1Dfetch(gDum2Ori_t,figateId)*4 + 2],val_d[tex1Dfetch(gDum2Ori_t,figateId)*4 + 3],patIdx);
                                            if((trType == fitr || trType == 2) && PDa <=
                                                twoLvlat_d[(!dataflag)*gatesPerLvl*paraPatNum + (figateId-gStrPrevLvl)*paraPatNum + patIdx] + delay){
                                                PDa = twoLvlat_d[(!dataflag)*gatesPerLvl*paraPatNum + (figateId - gStrPrevLvl)*paraPatNum + patIdx] + delay;
                                                transPin = fi;
                                            }
                                        }
                                    }
                                }
                                twoLvlat_d[(dataflag)*gatesPerLvl*paraPatNum + (gateId-gStrCurrLvl)*paraPatNum + patIdx] = PDa;
                                if(gTypeDum_d[gateId] == 1 || gTypeDum_d[gateId] == 3){ // PO or PPO, check DSM (Use Atual path delay)
                                    if((PDs_s[0] - PDa)/(Ttc - PDa) < delta){
                                        partialDict_d[fId*paraPatNum + patIdx] = 2;
                                        patDone_s[patIdx] = true;
                                    }
                                }
                            }
                        }
                    }
                }
                __syncthreads();
                // add event List
                if(currlvl+1 < cirLvl){
                    // evaluate a set of gates
                    loop = (eventSize_s[0] - 1)/blockDim.x + 1;
                    unsigned int eventSize = 0;
                    for(int j = 0 ; j < loop; ++j){
                        __syncthreads();
                        unsigned int Idx = j*blockDim.x + threadIdx.x;
                        if(Idx < eventSize_s[0]){
                            unsigned int eventGate = EventList_d[(!dataflag)*gatesPerLvl + Idx]>>4;
                            foStr_s[threadIdx.x]  = foOffsetDum_d[eventGate];
                            foSize_s[threadIdx.x] = foOffsetDum_d[eventGate+1] - foOffsetDum_d[eventGate];
                        }
                        __syncthreads();
                        if(threadIdx.x == 0){
                            for(int m = 0 ; m < blockDim.x && j*blockDim.x + m < eventSize_s[0]; ++m){
                                for(int fo = 0; fo < foSize_s[m]; ++fo){
                                    if((twoLvlfG_d[foArrayDum_d[foStr_s[m]+fo]-gStrNextLvl]) != 0){
                                        EventList_d[dataflag*gatesPerLvl + eventSize] = (foArrayDum_d[foStr_s[m]+fo]<<4) + (twoLvlfG_d[foArrayDum_d[foStr_s[m]+fo]-gStrNextLvl]);
                                        twoLvlfG_d[foArrayDum_d[foStr_s[m]+fo]-gStrNextLvl] = 0;
                                        eventSize++;
                                    }
                                }
                            }
                        }
                        __syncthreads();
                    }
                    if(threadIdx.x == 0){
                        eventSize_s[0] = eventSize;
                    }
                }
            }
        }
    }

}
//}}}
//{{{ __global__ void SFDAnalysis()
__global__ void SFDAnalysis(char* partialDict_d,int* SFD_d,int* SFD_z,unsigned int* fMask_d,unsigned int* Rmnfault_d,unsigned int* RmnfNum_d,
                            bool* pRdn_d,unsigned int RmnpNum,int patLoop,int iterNum,int mode){
    unsigned int RmnfNum = RmnfNum_d[0];
    int loop;
    unsigned fId;
    if(iterNum == 0){ // Iteration
        if(mode == 0){  // mode 0: after PDLB analysis
            loop = (RmnfNum - 1) / (SFD_thNum*SFD_bkNum) + 1;
            for(int i = 0 ; i < loop; ++i){
                // no coleasing read
                if(i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x < RmnfNum){
                    fId = Rmnfault_d[i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x];
                    bool meetCond = false;  // to see if meet two condition
                    for(int j = patLoop*paraPatNum; j < (patLoop+1)*paraPatNum && j < RmnpNum; ++j){
                        if(partialDict_d[fId*paraPatNum + (j -patLoop*paraPatNum)] != 0){  // dict == 1 or 2
                            meetCond = true;
                            break;
                        }
                    }
                    if(patLoop == 0){
                        SFD_d[fId] = -4;
                    }
                    if(meetCond){
                        fMask_d[i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x] = fId;
                    }
                    else{
                        fMask_d[i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x] = ~0;
                    }
                }

            }
        }
        else if(mode == 1){
            loop = (RmnfNum - 1) / (SFD_thNum*SFD_bkNum) + 1;
            for(int i = 0 ; i < loop ; ++i){
                if(i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x < RmnfNum){
                    fId = Rmnfault_d[i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x];
                    int SFD = SFD_d[fId];
                    bool detect = false;  // to see if fault is detected int this pattern Loop
                    for(int j = patLoop*paraPatNum; j < (patLoop+1)*paraPatNum && j < RmnpNum; ++j){
                        unsigned int patIdx = j-patLoop*paraPatNum;
                        if(partialDict_d[fId*paraPatNum + patIdx] == 1){
                            SFD |= 0x01;
                        }
                        else if(partialDict_d[fId*paraPatNum + patIdx] == 2){
                            detect = true;
                            if(SFD == -4){
                                SFD = (patIdx<<2);
                            }
                            else if((SFD>>2) >= 0){     // have already detect once
                                SFD = 2;
                                break;
                            }
                            else{
                                SFD = (patIdx<<2) | (SFD&0x01);
                            }
                        }
                    }
                    // extract remain fault with Q
                    if((SFD&0x01) == 1){
                        fMask_d[i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x] = fId;
                    }
                    else{
                        fMask_d[i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x] = ~0;
                        if(detect){
                            SFD_z[fId] = SFD | 0x01;
                        }
                        else{
                            SFD_z[fId] = SFD&((~0)<<1);
                        }
                    }
                    SFD_d[fId] = SFD&((~0)<<1);
                }
            }
        }
        else if(mode == 2){
            __shared__ bool pRdn_s[paraPatNum];
            loop = (paraPatNum - 1) / (SFD_thNum) + 1;
            for(int i = 0 ; i < loop; ++i){
                if(i*SFD_thNum +  threadIdx.x < paraPatNum){
                    pRdn_s[i*SFD_thNum +  threadIdx.x] = pRdn_d[i*SFD_thNum +  threadIdx.x];
                }
            }
            __syncthreads();
            loop = (RmnfNum - 1) / (SFD_thNum*SFD_bkNum) + 1;
            for(int i = 0 ; i < loop ; ++i){
                if(i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x < RmnfNum){
                    fId = Rmnfault_d[i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x];
                    int SFD = SFD_d[fId] & ((~0)<<1);
                    bool detect = false;
                    for(int j = patLoop*paraPatNum; j < (patLoop+1)*paraPatNum && j < RmnpNum; ++j){
                        unsigned int patIdx = j-patLoop*paraPatNum;
                        if(partialDict_d[fId*paraPatNum + patIdx] == 2){
                            detect = true;
                            if((SFD>>2) >= 0){
                                SFD = 2;
                                break;
                            }
                            else if((SFD>>2) < 0){
                                SFD = (patIdx<<2);
                            }
                        }
                    }
                    SFD_d[fId] = SFD;
                    if(detect)
                        SFD_z[fId] = SFD | 0x01;
                    else
                        SFD_z[fId] = SFD;
                }
            }
        }
    }
    else{
        if(mode == 0){
            loop = (paraPatNum - 1) / (SFD_thNum*SFD_bkNum) + 1;
            for(int i = 0 ; i < loop; ++i){
                if(i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x < paraPatNum){
                    pRdn_d[i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x] = true;
                }
            }
            loop = (RmnfNum - 1) / (SFD_thNum*SFD_bkNum) + 1;
            for(int i = 0 ; i < loop; ++i){
                // no coleasing read
                // Initialize pRdn
                if(i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x < RmnfNum){
                    fId = Rmnfault_d[i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x];
                    bool meetCond = false;  // to see if meet two condition
                    for(int j = patLoop*paraPatNum; j < (patLoop+1)*paraPatNum && j < RmnpNum; ++j){
                        if(partialDict_d[fId*paraPatNum + (j -patLoop*paraPatNum)] != 0){  // dict == 1 or 2
                            meetCond = true;
                            break;
                        }
                    }
                    if(patLoop == 0){
                        SFD_d[fId] = -4;
                    }
                    if(meetCond){
                        fMask_d[i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x] = fId;
                    }
                    else{
                        fMask_d[i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x] = ~0;
                    }
                }

            }
        }
        else if(mode == 1){
            loop = (RmnfNum - 1) / (SFD_thNum*SFD_bkNum) + 1;
            for(int i = 0 ; i < loop ; ++i){
                if(i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x < RmnfNum){
                    fId = Rmnfault_d[i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x];
                    int SFD = SFD_d[fId];
                    for(int j = patLoop*paraPatNum; j < (patLoop+1)*paraPatNum && j < RmnpNum; ++j){
                        unsigned int patIdx = j-patLoop*paraPatNum;
                        if(partialDict_d[fId*paraPatNum + patIdx] == 1 ){
                            if((SFD>>2) < 0){
                                pRdn_d[patIdx] = false;
                            }
                        }
                        if(partialDict_d[fId*paraPatNum + patIdx] == 2){
                            if((SFD>>2) < 0){
                                pRdn_d[patIdx] = false;
                            }
                            break;
                        }
                    }
                }
            }
        }
        else if(mode == 2){
            loop = (paraPatNum - 1) / (SFD_thNum*SFD_bkNum) + 1;
            for(int i = 0 ; i < loop; ++i){
                if(i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x < paraPatNum){
                    pRdn_d[i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x] = true;
                }
            }
            loop = (RmnfNum - 1) / (SFD_thNum*SFD_bkNum) + 1;
            for(int i = 0 ; i < loop ; ++i){
                if(i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x < RmnfNum){
                    fId = Rmnfault_d[i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x];
                    int SFD = SFD_d[fId];
                    for(int j = patLoop*paraPatNum; j < (patLoop+1)*paraPatNum && j < RmnpNum; ++j){
                        unsigned int patIdx = j-patLoop*paraPatNum;
                        if(partialDict_d[fId*paraPatNum + patIdx] == 2){
                            if((SFD>>2) < 0){
                                pRdn_d[patIdx] = false;
                            }
                            break;
                        }
                    }
                }
            }
        }
    }
}
//}}}
//{{{ __global__ void SFDRdnPatBuild()
__global__ void SFDRdnPatBuild(char* partialDict_d,int* SFD_d,int* SFD_z,unsigned int* LBRmnfault_d,unsigned int* LBRmnfNum_d,
                                    bool* pRdn_d,unsigned int RmnpNum,unsigned int* fMask_d,int patLoop,int mode){
    __shared__ bool pRdn_s[paraPatNum];
    int loop = (paraPatNum - 1) / (SFD_thNum) + 1;
    for(int i = 0 ; i < loop; ++i){
        if(i*SFD_thNum + threadIdx.x < paraPatNum)
            pRdn_s[i*SFD_thNum + threadIdx.x] = pRdn_d[i*SFD_thNum + threadIdx.x];
    }
    unsigned int LBRmnfNum = LBRmnfNum_d[0];
    unsigned fId;
    __syncthreads();
    if(mode == 1){
        loop = (LBRmnfNum - 1) / (SFD_thNum*SFD_bkNum) + 1;
        for(int i = 0 ; i < loop ; ++i){
            if(i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x < LBRmnfNum){
                fId = LBRmnfault_d[i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x];
                int SFD = SFD_d[fId];
                for(int j = patLoop*paraPatNum; j < (patLoop+1)*paraPatNum && j < RmnpNum; ++j){
                    unsigned int patIdx = j-patLoop*paraPatNum;
                    if(!pRdn_s[patIdx]){
                        if(partialDict_d[fId*paraPatNum + patIdx] == 1){
                            SFD |= 0x01;
                        }
                        else if(partialDict_d[fId*paraPatNum + patIdx] == 2){
                            if(SFD == -4){
                                SFD = (patIdx<<2);
                            }
                            else if((SFD>>2) >= 0){     // have already detect once
                                SFD = 2 | (SFD&0x01);
                                break;
                            }
                            else{
                                SFD = (patIdx<<2) | (SFD&0x01);
                            }
                        }
                    }
                }
                // extract remain fault with Q
                if((SFD&0x01) == 1){
                    fMask_d[i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x] = fId;
                }
                else{
                    fMask_d[i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x] = ~0;
                }
            }
        }
    }
    else{
        loop = (LBRmnfNum - 1) / (SFD_thNum*SFD_bkNum) + 1;
        for(int i = 0 ; i < loop ; ++i){
            if(i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x < LBRmnfNum){
                fId = LBRmnfault_d[i*SFD_thNum*SFD_bkNum + blockIdx.x*SFD_thNum + threadIdx.x];
                int SFD = SFD_d[fId];
                bool detect = false;
                for(int j = patLoop*paraPatNum; j < (patLoop+1)*paraPatNum && j < RmnpNum; ++j){
                    unsigned int patIdx = j-patLoop*paraPatNum;
                    //if(fId == 0){
                    //    printf("fId:%d patIdx:%d partialDict_d:%d SFD:%d\n",fId,patIdx,partialDict_d[fId*paraPatNum + j],SFD);
                    //}
                    if(!pRdn_s[patIdx]){ // pattern is not redundant
                        if(partialDict_d[fId*paraPatNum + patIdx] == 2){
                            detect = true;
                            if(SFD == -4){
                                SFD = (patIdx<<2);
                            }
                            else if((SFD>>2) >= 0){
                                SFD = 2;
                                break;
                            }
                            else{
                                SFD = (patIdx<<2);
                            }
                        }
                    }
                }
                SFD_d[fId] = SFD;
                if(detect)
                    SFD_z[fId] = SFD | 0x01;
                else
                    SFD_z[fId] = SFD & ((~0)<<1);
            }
        }
    }
}
//}}}
//{{{ __device__ bool isInv()
__device__ bool isInv(unsigned int gType){
    if((gType >= 10 && gType <= 13) ||
       (gType >= 18 && gType <= 21) ||
       (gType >= 25 && gType <= 27) ||
       (gType == 29 )){
        return true;
    }
    return false;
}
//}}}
//{{{ __device__ char getTrans()
__device__ char getTrans(PatValue t0_l_,PatValue t0_h_,PatValue t1_l_,PatValue t1_h_,int bitIdx){
    PatValue mask = 0x01;
    char t0,t1;

    char v0 = (t0_l_ & (mask<<bitIdx)) == 0 ? 0 : 1;
    char v1 = (t0_h_ & (mask<<bitIdx)) == 0 ? 0 : 1;
    if(v0 == 0 && v1 == 1){
        t0 = 1;
    }
    else if(v0 == 1 && v1 == 0){
        t0 = 0;
    }
    else if(v0 == 0 && v1 == 0){
        t0 = 2; // 2 = X
    }
    else{
        return 2;
    }

    v0 = (t1_l_ & (mask<<bitIdx)) == 0 ? 0 : 1;
    v1 = (t1_h_ & (mask<<bitIdx)) == 0 ? 0 : 1;
    if(v0 == 0 && v1 == 1){
        t1 = 1;
    }
    else if(v0 == 1 && v1 == 0){
        t1 = 0;
    }
    else if(v0 == 0 && v1 == 0){
        t1 = 2; // 2 = X
    }
    else{
        return 2;
    }

    if(t0 != t1){
        if(t0 == 0)
            return 0;   // rising
        if(t0 == 1)
            return 1;   // falling
    }
    else{
        return 2;
    }
    return 2;
}
//}}}
//{{{ __device__ char getFiNum()
__device__ char  getFiNum(unsigned int gType){
    char fiNum;
    switch(gType){
        case  0: fiNum = 0; break;
        case  1: fiNum = 1; break;
        case  2: fiNum = 1; break;
        case  3: fiNum = 1; break;
        case  4: fiNum = 0; break;
        case  5: fiNum = 0; break;
        case  6: fiNum = 0; break;
        case  7: fiNum = 2; break;
        case  8: fiNum = 3; break;
        case  9: fiNum = 4; break;
        case 10: fiNum = 0; break;
        case 11: fiNum = 2; break;
        case 12: fiNum = 3; break;
        case 13: fiNum = 4; break;
        case 14: fiNum = 0; break;
        case 15: fiNum = 2; break;
        case 16: fiNum = 3; break;
        case 17: fiNum = 4; break;
        case 18: fiNum = 0; break;
        case 19: fiNum = 2; break;
        case 20: fiNum = 3; break;
        case 21: fiNum = 4; break;
        case 22: fiNum = 0; break;
        case 23: fiNum = 2; break;
        case 24: fiNum = 3; break;
        case 25: fiNum = 0; break;
        case 26: fiNum = 2; break;
        case 27: fiNum = 3; break;
        case 28: fiNum = 1; break;
        case 29: fiNum = 1; break;
        case 30: fiNum = 0; break;
        case 31: fiNum = 0; break;
        case 32: fiNum = 0; break;
        case 33: fiNum = 1; break;
        default:
                 fiNum =0;
    }
    return fiNum;
}
//}}}

//{{{ __device__ bool getCtrl()
__device__ bool getCtrl(unsigned int gType, char trans){
    // Given gate type and gate output transition
    // Return we need to find the controlling value or noncontrolling value in fanin of the gate
    bool isCtrl = false;
    switch(gType){
        case  0: isCtrl = false; break; // PI
        case  1: isCtrl = false; break; // PO
        case  2: isCtrl = false; break; // PPI
        case  3: isCtrl = false; break; // PPO
        case  4: isCtrl = false; break; // TIEHI
        case  5: isCtrl = false; break; // TIELO
        case  6: // AND
                 if(trans == 0){
                     isCtrl = false;
                 }
                 else{
                     isCtrl = true;
                 }
                 break;
        case  7: // AND2
                 if(trans == 0){
                     isCtrl = false;
                 }
                 else{
                     isCtrl = true;
                 }
                 break;
        case  8:
                 if(trans == 0){
                     isCtrl = false;
                 }
                 else{
                     isCtrl = true;
                 }
                 break;
        case  9:
                 if(trans == 0){
                     isCtrl = false;
                 }
                 else{
                     isCtrl = true;
                 }
                 break;
        case  10: // NAND
                 if(trans == 0){
                     isCtrl = true;
                 }
                 else{
                     isCtrl = false;
                 }
                 break;
        case  11:
                 if(trans == 0){
                     isCtrl = true;
                 }
                 else{
                     isCtrl = false;
                 }
                 break;
        case  12:
                 if(trans == 0){
                     isCtrl = true;
                 }
                 else{
                     isCtrl = false;
                 }
                 break;
        case  13:
                 if(trans == 0){
                     isCtrl = true;
                 }
                 else{
                     isCtrl = false;
                 }
                 break;
        case  14: // OR
                 if(trans == 0){
                     isCtrl = true;
                 }
                 else{
                     isCtrl = false;
                 }
                 break;
        case  15:
                 if(trans == 0){
                     isCtrl = true;
                 }
                 else{
                     isCtrl = false;
                 }
                 break;
        case  16:
                 if(trans == 0){
                     isCtrl = true;
                 }
                 else{
                     isCtrl = false;
                 }
                 break;
        case  17:
                 if(trans == 0){
                     isCtrl = true;
                 }
                 else{
                     isCtrl = false;
                 }
                 break;
        case  18: // NOR
                 if(trans == 0){
                     isCtrl = false;
                 }
                 else{
                     isCtrl = true;
                 }
                 break;
        case  19:
                 if(trans == 0){
                     isCtrl = false;
                 }
                 else{
                     isCtrl = true;
                 }
                 break;
        case  20:
                 if(trans == 0){
                     isCtrl = false;
                 }
                 else{
                     isCtrl = true;
                 }
                 break;
        case  21:
                 if(trans == 0){
                     isCtrl = false;
                 }
                 else{
                     isCtrl = true;
                 }
                 break;
        case 22: isCtrl = false; break; // XOR
        case 23: isCtrl = false; break;
        case 24: isCtrl = false; break;
        case 25: isCtrl = false; break; // XNOR
        case 26: isCtrl = false; break;
        case 27: isCtrl = false; break;
        case 28: isCtrl = false; break; // BUF
        case 29: isCtrl = false; break; // INV
        case 30: isCtrl = false; break;
        case 31: isCtrl = false; break;
        case 32: isCtrl = false; break;
        case 33: isCtrl = false; break; // DUMMY
    }
    return isCtrl;
}
//}}}
//{{{ __device__ char transType()
__device__ char transType(unsigned int gType, char trans){  // 0 =  fin rising ; 1 = fin falling ; 2 = fin any transition
    // Given gate type and gate output transition
    // Find the transition we need to find in fanin of the gate
    char trType = 0;
    switch(gType){
        case  0: trType = 2;break; // PI
        case  1: trType = 2;break; // PO
        case  2: trType = 2;break; // PPI
        case  3: trType = 2;break; // PPO
        case  4: trType = 2;break; // TIEHI
        case  5: trType = 2;break; // TIELO
        case  6: // AND
                 if(trans == 0){
                     trType = 0;
                 }
                 else{
                     trType = 1;
                 }
                 break;
        case  7: // AND2
                 if(trans == 0){
                     trType = 0;
                 }
                 else{
                     trType = 1;
                 }
                 break;
        case  8:
                 if(trans == 0){
                     trType = 0;
                 }
                 else{
                     trType = 1;
                 }
                 break;
        case  9:
                 if(trans == 0){
                     trType = 0;
                 }
                 else{
                     trType = 1;
                 }
                 break;
        case  10: // NAND
                 if(trans == 0){
                     trType = 1;
                 }
                 else{
                     trType = 0;
                 }
                 break;
        case  11:
                 if(trans == 0){
                     trType = 1;
                 }
                 else{
                     trType = 0;
                 }
                 break;
        case  12:
                 if(trans == 0){
                     trType = 1;
                 }
                 else{
                     trType = 0;
                 }
                 break;
        case  13:
                 if(trans == 0){
                     trType = 1;
                 }
                 else{
                     trType = 0;
                 }
                 break;
        case  14: // OR
                 if(trans == 0){
                     trType = 0;
                 }
                 else{
                     trType = 1;
                 }
                 break;
        case  15:
                 if(trans == 0){
                     trType = 0;
                 }
                 else{
                     trType = 1;
                 }
                 break;
        case  16:
                 if(trans == 0){
                     trType = 0;
                 }
                 else{
                     trType = 1;
                 }
                 break;
        case  17:
                 if(trans == 0){
                     trType = 0;
                 }
                 else{
                     trType = 1;
                 }
                 break;
        case  18: // NOR
                 if(trans == 0){
                     trType = 1;
                 }
                 else{
                     trType = 0;
                 }
                 break;
        case  19:
                 if(trans == 0){
                     trType = 1;
                 }
                 else{
                     trType = 0;
                 }
                 break;
        case  20:
                 if(trans == 0){
                     trType = 1;
                 }
                 else{
                     trType = 0;
                 }
                 break;
        case  21:
                 if(trans == 0){
                     trType = 1;
                 }
                 else{
                     trType = 0;
                 }
                 break;
        case 22: trType = 2;  break; // XOR
        case 23: trType = 2;  break;
        case 24: trType = 2;  break;
        case 25: trType = 2;  break; // XNOR
        case 26: trType = 2;  break;
        case 27: trType = 2;  break;
        case 28: trType = 2;  break; // BUF
        case 29: trType = 2;  break; // INV
        case 30: trType = 2;  break;
        case 31: trType = 2;  break;
        case 32: trType = 2;  break;
        case 33: trType = 2;  break; // DUMMY
    }
    return trType;
}
//}}}
//{{{ __device__ char getBitValue()
__device__ char getBV(PatValue pv,int bitIdx){
    return (pv & ((PatValue)0x01 << bitIdx)) == 0 ? 0 : 1;
}
//}}}
//{{{ __device__ void getV()
__device__ char getV(PatValue fl_,PatValue fh_,PatValue gl_, PatValue gh_,unsigned int bitIdx){
    // Given gate fauly low/high value, faulty free low/high value and bit index
    // Return this pattern is D/D' or faulty free
    PatValue mask = 0x01;
    char fv,gv;   // fault value good value
    char vl = (fl_ &(mask<<bitIdx)) == 0 ? 0 : 1;
    char vh = (fh_ &(mask<<bitIdx)) == 0 ? 0 : 1;
    if(vl == 0 && vh == 1)
        fv = 1;
    else if(vl == 1 && vh == 0)
        fv = 0;
    else if(vl == 0 && vh ==0)
        fv = 2; // X
    else
        fv =3;  // dont care
    vl = (gl_ &(mask<<bitIdx)) == 0 ? 0 : 1;
    vh = (gh_ &(mask<<bitIdx)) == 0 ? 0 : 1;
    if(vl == 0 && vh == 1)
        gv = 1;
    else if(vl == 1 && vh == 0)
        gv = 0;
    else if(vl == 0 && vh ==0)
        gv = 2; // X
    else
        gv = 3;  // dont care
    if(gv == 0 && fv == 1)
        return 1;   // D' = good0/faulty1 is equal to falling
    else if(gv == 1 && fv == 0)
        return 0;   // D = good1/faulty0 is equal to rising
    else if(gv == 2 && fv == 1)
        return 1;   // D'
    else if(gv == 2 && fv == 0)
        return 0;   // D
    else if(gv == fv)
        return 2;   // faulty free
    else
        return 3;
}
//}}}
//{{{ __device__ void evalGate()
__device__ void evalGate(unsigned int gateId,unsigned int gType,unsigned int gFiOri0,unsigned int gFiOri1,unsigned int gFiOri2,unsigned int gFiOri3,
                         PatValue* val_d,PatValue hold_capture,int timeframe){
    // Evaluate function for logicSim. Evaluate the faulty free value of the gate
    // The faulty free value will be written back to val_d
    if(gType ==0){
        return;
    }
    else if(gType == 2){ // PPI
        // hold_capture indicate that which patterns will capture in time frame 1 in m patterns
        // modBit Indicate that which bit should be changed from time frame 0 to time frame 1
        // modified l_ of PPI
        PatValue modBit = (val_d[gFiOri0*4 + 2*(timeframe-1) + 0] ^ val_d[gateId*4 + 2*(timeframe-1) + 0]) & hold_capture;
        val_d[gateId*4 + 2*timeframe + 0] = val_d[gateId*4 + 2*(timeframe-1) + 0] ^ modBit;
        // modified h_ of PPI
        modBit = (val_d[gFiOri0*4 + 2*(timeframe-1) + 1] ^ val_d[gateId*4 + 2*(timeframe-1) + 1]) & hold_capture;
        val_d[gateId*4 + 2*timeframe + 1] = val_d[gateId*4 + 2*(timeframe-1) + 1] ^ modBit;
    }
    if(gType == 1 || gType == 3 || gType == 28){
        val_d[gateId*4 + 2*timeframe + 0] = val_d[gFiOri0*4 + 2*timeframe + 0];
        val_d[gateId*4 + 2*timeframe + 1] = val_d[gFiOri0*4 + 2*timeframe + 1];
    }
    else if(gType == 4){
        val_d[gateId*4 + 2*timeframe + 0] = 0;
        val_d[gateId*4 + 2*timeframe + 1] = ~0;
    }
    else if(gType == 5){
        val_d[gateId*4 + 2*timeframe + 0] = ~0;
        val_d[gateId*4 + 2*timeframe + 1] = 0;
    }
    else if(gType == 6){ // AND
    }
    else if(gType == 7){
        val_d[gateId*4 + 2*timeframe + 0] =  val_d[gFiOri0*4 + 2*timeframe + 0] | val_d[gFiOri1*4 + 2*timeframe + 0];
        val_d[gateId*4 + 2*timeframe + 1] =  val_d[gFiOri0*4 + 2*timeframe + 1] & val_d[gFiOri1*4 + 2*timeframe + 1];
    }
    else if(gType == 8){
        val_d[gateId*4 + 2*timeframe + 0] =  val_d[gFiOri0*4 + 2*timeframe + 0] | val_d[gFiOri1*4 + 2*timeframe + 0] | val_d[gFiOri2*4 + 2*timeframe + 0];
        val_d[gateId*4 + 2*timeframe + 1] =  val_d[gFiOri0*4 + 2*timeframe + 1] & val_d[gFiOri1*4 + 2*timeframe + 1] & val_d[gFiOri2*4 + 2*timeframe + 1];
    }
    else if(gType == 9){
        val_d[gateId*4 + 2*timeframe + 0] =  val_d[gFiOri0*4 + 2*timeframe + 0] | val_d[gFiOri1*4 + 2*timeframe + 0] | val_d[gFiOri2*4 + 2*timeframe + 0] | val_d[gFiOri3*4 + 2*timeframe + 0];
        val_d[gateId*4 + 2*timeframe + 1] =  val_d[gFiOri0*4 + 2*timeframe + 1] & val_d[gFiOri1*4 + 2*timeframe + 1] & val_d[gFiOri2*4 + 2*timeframe + 1] & val_d[gFiOri3*4 + 2*timeframe + 1];
    }
    else if(gType == 10){ // NAND
    }
    else if(gType == 11){
        val_d[gateId*4 + 2*timeframe + 0] =  val_d[gFiOri0*4 + 2*timeframe + 1] & val_d[gFiOri1*4 + 2*timeframe + 1];
        val_d[gateId*4 + 2*timeframe + 1] =  val_d[gFiOri0*4 + 2*timeframe + 0] | val_d[gFiOri1*4 + 2*timeframe + 0];
    }
    else if(gType == 12){
        val_d[gateId*4 + 2*timeframe + 0] =  val_d[gFiOri0*4 + 2*timeframe + 1] & val_d[gFiOri1*4 + 2*timeframe + 1] & val_d[gFiOri2*4 + 2*timeframe + 1];
        val_d[gateId*4 + 2*timeframe + 1] =  val_d[gFiOri0*4 + 2*timeframe + 0] | val_d[gFiOri1*4 + 2*timeframe + 0] | val_d[gFiOri2*4 + 2*timeframe + 0];
    }
    else if(gType == 13){
        val_d[gateId*4 + 2*timeframe + 0] =  val_d[gFiOri0*4 + 2*timeframe + 1] & val_d[gFiOri1*4 + 2*timeframe + 1] & val_d[gFiOri2*4 + 2*timeframe + 1] & val_d[gFiOri3*4 + 2*timeframe + 1];
        val_d[gateId*4 + 2*timeframe + 1] =  val_d[gFiOri0*4 + 2*timeframe + 0] | val_d[gFiOri1*4 + 2*timeframe + 0] | val_d[gFiOri2*4 + 2*timeframe + 0] | val_d[gFiOri3*4 + 2*timeframe + 0];
    }
    else if(gType == 14){ // OR
    }
    else if(gType == 15){
        val_d[gateId*4 + 2*timeframe + 0] =  val_d[gFiOri0*4 + 2*timeframe + 0] & val_d[gFiOri1*4 + 2*timeframe + 0];
        val_d[gateId*4 + 2*timeframe + 1] =  val_d[gFiOri0*4 + 2*timeframe + 1] | val_d[gFiOri1*4 + 2*timeframe + 1];
    }
    else if(gType == 16){
        val_d[gateId*4 + 2*timeframe + 0] =  val_d[gFiOri0*4 + 2*timeframe + 0] & val_d[gFiOri1*4 + 2*timeframe + 0] & val_d[gFiOri2*4 + 2*timeframe + 0];
        val_d[gateId*4 + 2*timeframe + 1] =  val_d[gFiOri0*4 + 2*timeframe + 1] | val_d[gFiOri1*4 + 2*timeframe + 1] | val_d[gFiOri2*4 + 2*timeframe + 1];
    }
    else if(gType == 17){
        val_d[gateId*4 + 2*timeframe + 0] =  val_d[gFiOri0*4 + 2*timeframe + 0] & val_d[gFiOri1*4 + 2*timeframe + 0] & val_d[gFiOri2*4 + 2*timeframe + 0] & val_d[gFiOri3*4 + 2*timeframe + 0];
        val_d[gateId*4 + 2*timeframe + 1] =  val_d[gFiOri0*4 + 2*timeframe + 1] | val_d[gFiOri1*4 + 2*timeframe + 1] | val_d[gFiOri2*4 + 2*timeframe + 1] | val_d[gFiOri3*4 + 2*timeframe + 1];
    }
    else if(gType == 18){ // NOR
    }
    else if(gType == 19){
        val_d[gateId*4 + 2*timeframe + 0] =  val_d[gFiOri0*4 + 2*timeframe + 1] | val_d[gFiOri1*4 + 2*timeframe + 1];
        val_d[gateId*4 + 2*timeframe + 1] =  val_d[gFiOri0*4 + 2*timeframe + 0] & val_d[gFiOri1*4 + 2*timeframe + 0];
    }
    else if(gType == 20){
        val_d[gateId*4 + 2*timeframe + 0] =  val_d[gFiOri0*4 + 2*timeframe + 1] | val_d[gFiOri1*4 + 2*timeframe + 1] | val_d[gFiOri2*4 + 2*timeframe + 1];
        val_d[gateId*4 + 2*timeframe + 1] =  val_d[gFiOri0*4 + 2*timeframe + 0] & val_d[gFiOri1*4 + 2*timeframe + 0] & val_d[gFiOri2*4 + 2*timeframe + 0];
    }
    else if(gType == 21){
        val_d[gateId*4 + 2*timeframe + 0] =  val_d[gFiOri0*4 + 2*timeframe + 1] | val_d[gFiOri1*4 + 2*timeframe + 1] | val_d[gFiOri2*4 + 2*timeframe + 1] | val_d[gFiOri3*4 + 2*timeframe + 1];
        val_d[gateId*4 + 2*timeframe + 1] =  val_d[gFiOri0*4 + 2*timeframe + 0] & val_d[gFiOri1*4 + 2*timeframe + 0] & val_d[gFiOri2*4 + 2*timeframe + 0] & val_d[gFiOri3*4 + 2*timeframe + 0];
    }
    else if(gType == 22){ // XOR
    }
    else if(gType == 23){
        val_d[gateId*4 + 2*timeframe + 0] = (val_d[gFiOri0*4 + 2*timeframe + 0] & val_d[gFiOri1*4 + 2*timeframe + 0]) | (val_d[gFiOri0*4 + 2*timeframe + 1] & val_d[gFiOri1*4 + 2*timeframe + 1]);
        val_d[gateId*4 + 2*timeframe + 1] = (val_d[gFiOri0*4 + 2*timeframe + 0] & val_d[gFiOri1*4 + 2*timeframe + 1]) | (val_d[gFiOri0*4 + 2*timeframe + 1] & val_d[gFiOri1*4 + 2*timeframe + 0]);
    }
    else if(gType == 24){
        val_d[gateId*4 + 2*timeframe + 0] = (val_d[gFiOri0*4 + 2*timeframe + 0] & val_d[gFiOri1*4 + 2*timeframe + 0]) | (val_d[gFiOri0*4 + 2*timeframe + 1] & val_d[gFiOri1*4 + 2*timeframe + 1]);
        val_d[gateId*4 + 2*timeframe + 1] = (val_d[gFiOri0*4 + 2*timeframe + 0] & val_d[gFiOri1*4 + 2*timeframe + 1]) | (val_d[gFiOri0*4 + 2*timeframe + 1] & val_d[gFiOri1*4 + 2*timeframe + 0]);
        val_d[gateId*4 + 2*timeframe + 0] = (val_d[gateId*4 + 2*timeframe + 0] & val_d[gFiOri2*4 + 2*timeframe + 0]) | (val_d[gateId*4 + 2*timeframe + 1] & val_d[gFiOri2*4 + 2*timeframe + 1]);
        val_d[gateId*4 + 2*timeframe + 1] = (val_d[gateId*4 + 2*timeframe + 0] & val_d[gFiOri2*4 + 2*timeframe + 1]) | (val_d[gateId*4 + 2*timeframe + 1] & val_d[gFiOri2*4 + 2*timeframe + 0]);
    }
    else if(gType == 25){ // XNOR
    }
    else if(gType == 26){
        val_d[gateId*4 + 2*timeframe + 0] = (val_d[gFiOri0*4 + 2*timeframe + 0] & val_d[gFiOri1*4 + 2*timeframe + 1]) | (val_d[gFiOri0*4 + 2*timeframe + 1] & val_d[gFiOri1*4 + 2*timeframe + 0]);
        val_d[gateId*4 + 2*timeframe + 1] = (val_d[gFiOri0*4 + 2*timeframe + 0] & val_d[gFiOri1*4 + 2*timeframe + 0]) | (val_d[gFiOri0*4 + 2*timeframe + 1] & val_d[gFiOri1*4 + 2*timeframe + 1]);
    }
    else if(gType == 27){
        val_d[gateId*4 + 2*timeframe + 0] = (val_d[gFiOri0*4 + 2*timeframe + 0] & val_d[gFiOri1*4 + 2*timeframe + 0]) | (val_d[gFiOri0*4 + 2*timeframe + 1] & val_d[gFiOri1*4 + 2*timeframe + 1]);
        val_d[gateId*4 + 2*timeframe + 1] = (val_d[gFiOri0*4 + 2*timeframe + 0] & val_d[gFiOri1*4 + 2*timeframe + 1]) | (val_d[gFiOri0*4 + 2*timeframe + 1] & val_d[gFiOri1*4 + 2*timeframe + 0]);
        val_d[gateId*4 + 2*timeframe + 0] = (val_d[gateId*4 + 2*timeframe + 0] & val_d[gFiOri2*4 + 2*timeframe + 0]) | (val_d[gateId*4 + 2*timeframe + 1] & val_d[gFiOri2*4 + 2*timeframe + 1]);
        val_d[gateId*4 + 2*timeframe + 1] = (val_d[gateId*4 + 2*timeframe + 0] & val_d[gFiOri2*4 + 2*timeframe + 1]) | (val_d[gateId*4 + 2*timeframe + 1] & val_d[gFiOri2*4 + 2*timeframe + 0]);
        PatValue temp = val_d[gateId*4 + 2*timeframe + 0];
        val_d[gateId*4 + 2*timeframe + 0] = val_d[gFiOri0*4 + 2*timeframe + 0];
        val_d[gateId*4 + 2*timeframe + 1] = temp;
    }
    else if(gType == 29){
        val_d[gateId*4 + 2*timeframe + 0] = val_d[gFiOri0*4 + 2*timeframe + 1];
        val_d[gateId*4 + 2*timeframe + 1] = val_d[gFiOri0*4 + 2*timeframe + 0];
    }
    else if(gType == 30){
    }

}
__device__ void evalGate(unsigned int gType,PatValue fi0l_,PatValue fi0h_,PatValue fi1l_,PatValue fi1h_,
        PatValue fi2l_,PatValue fi2h_,PatValue fi3l_,PatValue fi3h_,PatValue* twoLvlval_d){
    // Evaluate function for faultSim. Evaluate the faulty value of the gate
    // The faulty value will be written back to twoLvlval_d
    if(gType == 0){
        return;
    }
    else if(gType == 1){
        twoLvlval_d[0] = fi0l_;
        twoLvlval_d[1] = fi0h_;
    }
    else if(gType == 3){
        twoLvlval_d[0] = fi0l_;
        twoLvlval_d[1] = fi0h_;
    }
    else if(gType == 2){
        twoLvlval_d[0] = fi0l_;
        twoLvlval_d[1] = fi0h_;
    }
    else if(gType == 28){
        twoLvlval_d[0] = fi0l_;
        twoLvlval_d[1] = fi0h_;
    }
    else if(gType == 33){
        twoLvlval_d[0] = fi0l_;
        twoLvlval_d[1] = fi0h_;
    }
    else if(gType == 29){ // INV
        twoLvlval_d[0] = fi0h_;
        twoLvlval_d[1] = fi0l_;
    }
    else if(gType == 4){
        twoLvlval_d[0] = 0;
        twoLvlval_d[1] = ~0;
    }
    else if(gType == 5){
        twoLvlval_d[0] = ~0;
        twoLvlval_d[1] = 0;
    }
    else if(gType == 6){ // AND
    }
    else if(gType == 7){
        twoLvlval_d[0] =  fi0l_ | fi1l_;
        twoLvlval_d[1] =  fi0h_ & fi1h_;
    }
    else if(gType == 8){
        twoLvlval_d[0] = fi0l_ | fi1l_ | fi2l_;
        twoLvlval_d[1] = fi0h_ & fi1h_ & fi2h_;
    }
    else if(gType == 9){
        twoLvlval_d[0] = fi0l_ | fi1l_ | fi2l_ | fi3l_;
        twoLvlval_d[1] = fi0h_ & fi1h_ & fi2h_ & fi3h_;
    }
    else if(gType == 10){ // NAND
    }
    else if(gType == 11){
        twoLvlval_d[0] =  fi0h_ & fi1h_;
        twoLvlval_d[1] =  fi0l_ | fi1l_;
    }
    else if(gType == 12){
        twoLvlval_d[0] =  fi0h_ & fi1h_ & fi2h_;
        twoLvlval_d[1] =  fi0l_ | fi1l_ | fi2l_;
    }
    else if(gType == 13){
        twoLvlval_d[0] =  fi0h_ & fi1h_ & fi2h_ & fi3h_;
        twoLvlval_d[1] =  fi0l_ | fi1l_ | fi2l_ | fi3l_;
    }
    else if(gType == 14){ // OR
    }
    else if(gType == 15){
        twoLvlval_d[0] =  fi0l_ & fi1l_;
        twoLvlval_d[1] =  fi0h_ | fi1h_;
    }
    else if(gType == 16){
        twoLvlval_d[0] =  fi0l_ & fi1l_ & fi2l_;
        twoLvlval_d[1] =  fi0h_ | fi1h_ | fi2h_;
    }
    else if(gType == 17){
        twoLvlval_d[0] =  fi0l_ & fi1l_ & fi2l_ & fi3l_;
        twoLvlval_d[1] =  fi0h_ | fi1h_ | fi2h_ | fi3h_;
    }
    else if(gType == 18){ // NOR
    }
    else if(gType == 19){
        twoLvlval_d[0] =  fi0h_ | fi1h_;
        twoLvlval_d[1] =  fi0l_ & fi1l_;
    }
    else if(gType == 20){
        twoLvlval_d[0] =  fi0h_ | fi1h_ | fi2h_;
        twoLvlval_d[1] =  fi0l_ & fi1l_ & fi2l_;
    }
    else if(gType == 21){
        twoLvlval_d[0] =  fi0h_ | fi1h_ | fi2h_ | fi3h_;
        twoLvlval_d[1] =  fi0l_ & fi1l_ & fi2l_ & fi3l_;
    }
    else if(gType == 22){ // XOR
    }
    else if(gType == 23){
        twoLvlval_d[0] = (fi0l_ & fi1l_)| (fi0h_ & fi1h_);
        twoLvlval_d[1] = (fi0l_ & fi1h_)| (fi0h_ & fi1l_);
    }
    else if(gType == 24){
        twoLvlval_d[0] = ((fi0l_ & fi1l_)| (fi0h_ & fi1h_) & fi2l_) | ((fi0l_ & fi1h_)| (fi0h_ & fi1l_) & fi2h_);
        twoLvlval_d[1] = ((fi0l_ & fi1l_)| (fi0h_ & fi1h_) & fi2h_) | ((fi0l_ & fi1h_)| (fi0h_ & fi1l_) & fi2l_);
    }
    else if(gType == 25){ // XNOR
    }
    else if(gType == 26){
        twoLvlval_d[0] = (fi0l_ & fi1h_)| (fi0h_ & fi1l_);
        twoLvlval_d[1] = (fi0l_ & fi1l_)| (fi0h_ & fi1h_);
    }
    else if(gType == 27){
        twoLvlval_d[0] = ((fi0l_ & fi1l_)| (fi0h_ & fi1h_) & fi2h_) | ((fi0l_ & fi1h_)| (fi0h_ & fi1l_) & fi2l_);
        twoLvlval_d[1] = ((fi0l_ & fi1l_)| (fi0h_ & fi1h_) & fi2l_) | ((fi0l_ & fi1h_)| (fi0h_ & fi1l_) & fi2h_);
    }
    else if(gType == 30){
    }
}
//}}}
//{{{ void SddCudaSim::CompactRedundantFault()
void SddCudaSim::CompactRedundantFault(int* SFD,bool* fRdn_d,vector<unsigned int>& faultList,unsigned int iterNum,unsigned int& DSMfNum){
    // Iteration 0 identify U fault which are not detect by DSM
    // Iteration 1 identify fault which arre not detect by TFS & DSM
    if(iterNum == 0){
        ofstream udf;
        if(arg->DSM_only == true)
            udf.open(arg->UDfName.c_str());
        // after iterNum 0 identify Redundant fault
        // these faults don't have to use UB/LB in next iteration
        bool*   fRdn = (bool*)malloc(sizeof(bool)*faultList.size());
        for(int i = 0 ; i < faultList.size(); ++i){
            if((SFD[i]>>2) < 0){ // redundant fault
                fRdn[i] = true;
                if(arg->DSM_only){
                    udf<<i<<endl;
                }
            }
            else{
                fRdn[i] = false;
                DSMfNum++;
            }
        }
        hipMemcpy(fRdn_d,fRdn,sizeof(bool)*faultList.size(),hipMemcpyHostToDevice);
        free(fRdn);
        if(arg->DSM_only)
            udf.close();
    }
    else if(iterNum == 1){
        // after iterNum 1 if fault still redundant
        // these fault can't be detect by both TRF and timing
        // remove these pattern from remain fault list
        vector<unsigned int> faultList_tmp = faultList;
        faultList.clear();
        for(int i = 0 ; i < faultList_tmp.size(); ++i){
            if((SFD[i]>>2) >= 0){ // not redundant fault
                faultList.push_back(i);
            }
        }
    }

}
//}}}
//{{{ void SddCudaSim::SetRedundantPat()
void SddCudaSim::SetRedundantPat(bool* pRdn_d,vector<Pat*>& patList,int patLoop){
    hipDeviceSynchronize();
    bool* pRdn = (bool*)malloc(sizeof(bool)*paraPatNum);
    hipMemcpy(pRdn,pRdn_d,sizeof(bool)*paraPatNum ,hipMemcpyDeviceToHost);
    for(unsigned int i = patLoop*paraPatNum; i < (patLoop+1)*paraPatNum && i < patList.size(); ++i){
        unsigned int patIdx = i - patLoop*paraPatNum;
        if(pRdn[patIdx]){ //redundant pattern
            patList[i]->redundant = true;
            //printf("X");
        }
        else{
            //printf("O");
        }
    }
    free(pRdn);
}
//}}}
//{{{ void Check Function()
void SddCudaSim::CheckLogicVal(PatValue* val_d,int patLoop,int patNum, CircuitInfo* cirInfo){
    PatValue* val = (PatValue*)malloc(sizeof(PatValue)*(cirInfo->OrigateNum)*4*LS_bkNum);
    hipMemcpy(val        ,val_d       ,sizeof(PatValue)*(cirInfo->OrigateNum)*4*LS_bkNum   ,hipMemcpyDeviceToHost);
    for(int m = 0 ; m < LS_bkNum && patLoop+m < (patNum-1)/paraPatNum + 1; ++m){
        cout<<"********* Pattern: "<<m<<" *********"<<endl;
        int offset = cirInfo->OrigateNum*4*m;
        for(int j = 0 ; j < cirInfo->OrigateNum;++j){
            printf("gate %8d_t0:",j);PrintBinaryValue(val[offset + 4*j+0],val[offset + 4*j+1]);
            printf("gate %8d_t1:",j);PrintBinaryValue(val[offset + 4*j+2],val[offset + 4*j+3]);
            cout<<"------------------------------------------------------"<<endl;
        }
    }
    free(val);
}
void SddCudaSim::CheckDict(char* dictCPU,char* partialDict_d,vector<unsigned int>& RmnfaultList,int patLoop,int patNum, CircuitInfo* cirInfo){
    char*   dictGPU         = (char*)malloc(sizeof(char)*cirInfo->fNum*paraPatNum);
    hipMemcpy(dictGPU          ,partialDict_d       ,sizeof(char)*(cirInfo->fNum)*paraPatNum ,hipMemcpyDeviceToHost);
    printf("----------- Check Dict -----------\n");
    bool passed = true;
    unsigned int errNum = 0;
    unsigned int fId;
    unsigned int patIdx;
    for(int m = 0 ; m < RmnfaultList.size(); ++m){
        fId = RmnfaultList[m];
        for(int j = patLoop*paraPatNum; j < (patLoop+1)*paraPatNum && j < patNum; ++j){
            patIdx = j - patLoop*paraPatNum;
            if(dictCPU[fId*paraPatNum + patIdx] != dictGPU[fId*paraPatNum + patIdx]){
                printf("[Error]: fault:%d pat:%d CPU:%d GPU:%d\n",m,j,dictCPU[fId*paraPatNum + patIdx],dictGPU[fId*paraPatNum + patIdx]);
                passed = false;
                errNum++;
                if(errNum > 64)
                    return;
            }
        }
    }
    if(passed)
        printf("[Correct] TFS\n");
    free(dictGPU);
}
void SddCudaSim::CheckDynamicAT(PatValue* val_d,float* at_d,int i, CircuitInfo* cirInfo){
    PatValue* val = (PatValue*)malloc(sizeof(PatValue)*(cirInfo->OrigateNum)*4*LS_bkNum);
    float* at = (float*)malloc(sizeof(float)*(cirInfo->OrigateNum)*paraPatNum);
    hipMemcpy(val        ,val_d       ,sizeof(PatValue)*(cirInfo->OrigateNum)*4*LS_bkNum   ,hipMemcpyDeviceToHost);
    hipMemcpy(at         ,at_d       ,sizeof(float)*(cirInfo->OrigateNum)*paraPatNum ,hipMemcpyDeviceToHost);
    for(int j = 0; j < paraPatNum; ++j){
        if(i*paraPatNum + j == cirInfo->patNum)
            break;
        cout<<"> Simulate Patterns: "<< i*paraPatNum + j<<endl;
        for(int m = 0 ; m < cirInfo->OrigateNum; ++m){
            char t0 = getBitValue(val[m*4+0],val[m*4+1],j);
            char t1 = getBitValue(val[m*4+2],val[m*4+3],j);
            if(t0 != 3 && t1 !=3 && t0 != t1){
                printf("gate:%4d  val:%d->%d  at:%f \n",m,t0,t1,at[m*paraPatNum+j]);
            }
        }
    }
    free(val);
    free(at);
}
void SddCudaSim::CheckFComp(int* SFD,char* partialDict_d,unsigned int* Rmnfault_d,unsigned int* RmnfNum_d,vector<unsigned int>& RmnfaultList,
        int patLoop,int patNum, CircuitInfo* cirInfo, char mode){ // Check fault Compaction
    hipDeviceSynchronize();
    char*         dictGPU      = (char*)malloc(sizeof(char)*cirInfo->fNum*paraPatNum);
    unsigned int* Rmnfault_ptr = (unsigned int*)malloc(sizeof(unsigned int)*(cirInfo->fNum));
    unsigned int* RmnfNumGPU   = (unsigned int*)malloc(sizeof(unsigned int)*2);
    vector<unsigned int > Rmnfault_tmp;
    hipMemcpy(dictGPU       ,partialDict_d       ,sizeof(char)*(cirInfo->fNum)*paraPatNum ,hipMemcpyDeviceToHost);
    hipMemcpy(Rmnfault_ptr  ,Rmnfault_d   ,sizeof(unsigned int)*(cirInfo->fNum)    ,hipMemcpyDeviceToHost);
    hipMemcpy(RmnfNumGPU    ,RmnfNum_d    ,sizeof(unsigned int)*2                  ,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    unsigned errNum = 0;
    unsigned int fId;
    unsigned int patIdx;
    if(mode == 0){
        for(int i = 0 ; i < RmnfaultList.size(); ++i){
            fId = RmnfaultList[i];
            for(int j = patLoop*paraPatNum ; j < (patLoop+1)*paraPatNum && j < patNum; ++j){
                patIdx = j - patLoop*paraPatNum;
                if(dictGPU[fId*paraPatNum + patIdx] > 0){
                    Rmnfault_tmp.push_back(fId);
                    break;
                }
            }
        }
        if(Rmnfault_tmp.size() != RmnfNumGPU[1]){
            printf("[Error] RmnfNum CPU:%d GPU:%d\n",Rmnfault_tmp.size(),RmnfNumGPU[1]);
        }
        else{
            printf("[Correct] RmnfNum:%d\n",RmnfNumGPU[1]);
        }
        for(int i = 0 ; i < Rmnfault_tmp.size() && i < RmnfNumGPU[1];++i){
            if(Rmnfault_tmp[i] != Rmnfault_ptr[i]){
                printf("[Error] MOde 0 Rmnfault[%d] CPU:%d GPU:%d\n",i,Rmnfault_tmp[i],Rmnfault_ptr[i]);
                errNum++;
            }
            if(errNum > 10)
                break;
        }
        if(errNum == 0)
            printf("[Correct] Fault Compaction Mode 0\n");
    }
    else if(mode == 1){
        for(int i = 0 ; i < RmnfaultList.size();++i){
            unsigned int QNum = 0;
            unsigned int DNum = 0;
            fId = RmnfaultList[i];
            if((SFD[fId]>>2) >= 0)
                DNum++;
            for(int j = patLoop*paraPatNum ; j < (patLoop+1)*paraPatNum && j < patNum; ++j){
                patIdx = j - patLoop*paraPatNum;
                if(dictGPU[fId*paraPatNum + patIdx] == 1){
                    QNum++;
                }
                else if(dictGPU[fId*paraPatNum + patIdx] == 2){
                    DNum++;
                    if(DNum == 2){
                        break;
                    }
                }
            }
            if(QNum > 0 && DNum < 2)
                Rmnfault_tmp.push_back(fId);
        }
        if(Rmnfault_tmp.size() != RmnfNumGPU[1]){
            printf("[Error] Mode 1 RmnfNum CPU:%d GPU:%d\n",Rmnfault_tmp.size(),RmnfNumGPU[1]);
        }
        else{
            printf("[Correct] RmnfNum:%d\n",RmnfNumGPU[1]);
        }
        for(int i = 0 ; i < Rmnfault_tmp.size() || i < RmnfNumGPU[1]; ++i){
            if(i < Rmnfault_tmp.size() && i < RmnfNumGPU[1]){
                if(Rmnfault_tmp[i] != Rmnfault_ptr[i]){
                    printf("[Error]: Rmnfault[%d] CPU:%d GPU:%d\n",i,Rmnfault_tmp[i],Rmnfault_ptr[i]);
                    errNum++;
                }
            }
            else if(i < Rmnfault_tmp.size() && i >= RmnfNumGPU[1]){
                printf("[Error]: Rmnfault[%d] CPU:%d GPU: X\n",i,Rmnfault_tmp[i]);
                errNum++;
            }
            else if(i >= Rmnfault_tmp.size() && i < RmnfNumGPU[1]){
                printf("[Error]: Rmnfault[%d] CPU: X GPU:%d\n",i,Rmnfault_ptr[i]);
                errNum++;
            }

            if(errNum > 10)
                break;
        }
        if(errNum == 0)
            printf("[Correct] Fault Compaction Mode 1\n");
    }
    free(dictGPU);
    free(Rmnfault_ptr);
    free(RmnfNumGPU);
}
void SddCudaSim::CheckRedundantPat(char* partialDict_d,int* SFD_d,vector<unsigned int>& RmnfaultList,bool* pRdn_d,vector<Pat*>& patList,int patLoop,CircuitInfo* cirInfo){
    bool* pRdn = (bool*)malloc(sizeof(bool)*paraPatNum);
    hipMemcpy(pRdn,pRdn_d,sizeof(bool)*paraPatNum ,hipMemcpyDeviceToHost);
    char* dict = (char*)malloc(sizeof(char)*cirInfo->fNum*paraPatNum);
    hipMemcpy(dict          ,partialDict_d       ,sizeof(char)*(cirInfo->fNum)*paraPatNum ,hipMemcpyDeviceToHost);
    int* SFDGPU = (int*)malloc(sizeof(int)*cirInfo->fNum);
    hipMemcpy(SFDGPU        ,SFD_d       ,sizeof(int)*(cirInfo->fNum) ,hipMemcpyDeviceToHost);
    bool* pRdnCPU = (bool*)malloc(sizeof(bool)*paraPatNum);
    memset(pRdnCPU,true,sizeof(bool)*paraPatNum);
    for(int m = 0 ; m < RmnfaultList.size(); ++m){
        unsigned int fId = RmnfaultList[m];
        int SFD = SFDGPU[fId];
        for(int j = patLoop*paraPatNum; j < (patLoop+1)*paraPatNum && j < patList.size(); ++j){
            int patIdx = j - patLoop*paraPatNum;
            if(dict[fId*paraPatNum + patIdx] == 2){
                if((SFD>>2) < 0){
                    pRdnCPU[patIdx] = true;
                    break;
                }
            }
        }
    }
    bool error = false;
    for(int i = 0 ; i < paraPatNum && i+patLoop*paraPatNum < patList.size(); ++i){
        if(pRdn[i] != pRdnCPU[i]){
            printf("[Error]: Redunpant Pattern[%d] GPU:%d CPU:%d\n",patList[i+patLoop*paraPatNum]->id,pRdn[i],pRdnCPU[i]);
            error = true;
        }
    }
    if(error == false)
        printf("[Correct] Redundant Pattern\n");
    free(SFDGPU);
    free(pRdn);
    free(pRdnCPU);
    free(dict);
}
//}}}
//{{{ void Print Function()
void PrintDict(char* partialDict_d,int i,CircuitInfo* cirInfo){
    hipDeviceSynchronize();
    printf("-----------------------------------------------\n");
    char* dict = (char*)malloc(sizeof(char)*cirInfo->fNum*paraPatNum);
    hipMemcpy(dict          ,partialDict_d       ,sizeof(char)*(cirInfo->fNum)*paraPatNum ,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    for(int m = 0 ; m < cirInfo->fNum; ++m){
        printf("f %8d:",m);
        for(int j = i*paraPatNum; j < (i+1)*paraPatNum && j < cirInfo->patNum; ++j){
            printf("%d",dict[m*paraPatNum + j-i*paraPatNum]);
        }
        printf("\n");
    }
    free(dict);
}
void PrintSFD(char* partialDict_d,int* SFD,int* SFD_d,vector<unsigned int>& RmnfaultList,int patLoop,int patNum, CircuitInfo* cirInfo){
    hipDeviceSynchronize();
    printf("-----------------------------------------------\n");
    char* dict = (char*)malloc(sizeof(char)*cirInfo->fNum*paraPatNum);
    hipMemcpy(dict          ,partialDict_d       ,sizeof(char)*(cirInfo->fNum)*paraPatNum ,hipMemcpyDeviceToHost);
    int* SFDGPU = (int*)malloc(sizeof(int)*cirInfo->fNum);
    hipMemcpy(SFDGPU        ,SFD_d       ,sizeof(int)*(cirInfo->fNum) ,hipMemcpyDeviceToHost);
    /*
    for(int m = 0 ; m < RmnfaultList.size(); ++m){
        unsigned int fId = RmnfaultList[m];
        printf("f %8d:",fId);
        for(int j = patLoop*paraPatNum; j < (patLoop+1)*paraPatNum && j < patNum; ++j){
            printf("%d",dict[fId*paraPatNum + j-patLoop*paraPatNum]);
        }
        if((SFD[fId]&0x02) != 0){
            printf("\t SFD_CPU: DD");
        }
        else if((SFD[fId]>>2) < 0){
            printf("\t SFD_CPU:  R");
        }
        else{
            printf("\t SFD_CPU: %10d | %2d",(SFD[fId]>>2),(SFD[fId]&0x03));
        }
        if((SFDGPU[fId]&0x02) != 0){
            printf("\t SFD_GPU: DD");
        }
        else if((SFDGPU[fId]>>2) < 0){
            printf("\t SFD_GPU:  R");
        }
        else{
            printf("\t SFD_GPU: %10d | %2d",(SFD[fId]>>2),(SFD[fId]&0x03));
        }
        printf("\n");
    }
    */
    for(int m = 0 ; m < cirInfo->fNum; ++m){
        unsigned int fId = m;
        if(fId == 0){
        printf("f %8d:",fId);
        for(int j = patLoop*paraPatNum; j < (patLoop+1)*paraPatNum && j < patNum; ++j){
            printf("%d",dict[fId*paraPatNum + j-patLoop*paraPatNum]);
        }
        if((SFD[fId]&0x02) != 0){
            printf("\t SFD_CPU: DD");
        }
        else if((SFD[fId]>>2) < 0){
            printf("\t SFD_CPU:  R");
        }
        else{
            printf("\t SFD_CPU: %10d | %2d",(SFD[fId]>>2),(SFD[fId]&0x03));
        }
        if((SFDGPU[fId]&0x02) != 0){
            printf("\t SFD_GPU: DD");
        }
        else if((SFDGPU[fId]>>2) < 0){
            printf("\t SFD_GPU:  R");
        }
        else{
            printf("\t SFD_GPU: %10d | %2d",(SFDGPU[fId]>>2),(SFDGPU[fId]&0x03));
        }
        printf("\n");
        }
    }
    free(dict);
    free(SFDGPU);
}
//}}}
