#include <iostream>
#include <fstream>
#include <vector>
#include <stdlib.h>
#include <stdio.h>
#include <algorithm>
#include <set>
#include <list>
#include <hip/hip_runtime.h>

#include <iomanip>
using namespace std;
__global__ void temp1(int *x,int *y,int *z){
    int thId = threadIdx.x;
    z[thId]=x[thId]+y[thId];
    z[thId]=100;
    __syncthreads();
}
int main()
{
    hipSetDevice(0);
    hipDeviceReset();

int *x_d,*y_d,*z_d,*x,*y,*z;
x=(int*)malloc(sizeof(int));
y=(int*)malloc(sizeof(int));
z=(int*)malloc(sizeof(int));
*x=1;
*y=2;
hipMalloc((void**)&x_d, sizeof(int));
hipMalloc((void**)&y_d, sizeof(int));
hipMalloc((void**)&z_d, sizeof(int));
hipMemcpy(x_d, x, sizeof(int),  hipMemcpyHostToDevice);
hipMemcpy(y_d, y, sizeof(int),  hipMemcpyHostToDevice);
temp1<<<1,1>>>(x_d,y_d,z_d);
cout<<"result="<<*z<<endl;
hipMemcpy(z, z_d, sizeof(int),  hipMemcpyDeviceToHost);
hipDeviceSynchronize();
cout<<"result="<<*z<<endl;
}
